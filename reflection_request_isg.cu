#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "macro.h"
using namespace optix;

#ifdef NOGEOMETRY
rtBuffer<float4, 2>         posBuffer;
rtBuffer<float4, 2>         normalBuffer;;
rtBuffer<float4, 2>         colorBuffer;

rtDeclareVariable(float2,   bbmin,,);
rtDeclareVariable(float2,   bbmax,,);
rtDeclareVariable(optix::Matrix4x4, optixModelView_Inv, , );
__device__ float3 getImagePos(float2 tc)
{
	float2 xy = bbmin + (bbmax-bbmin)*tc;
	xy = xy;
	//printf("xyz:(%f,%f)\n",xy.x,xy.y,z);	
	float4 temp =  make_float4(xy.x,xy.y,-1,1)*optixModelView_Inv;
	temp = temp/temp.w;
	return make_float3(temp.x,temp.y,temp.z);
}

#endif

rtDeclareVariable(float3, eye_pos, , );
rtBuffer<float4, 2>         reflection_buffer;
rtBuffer<float4, 2>         addition_buffer;;
rtBuffer<uint,1>          Pixels_Buffer;
//rtBuffer<float4,2>          LastReflection_buffer;


rtTextureSampler<float4, 2> normal_texture;
rtTextureSampler<float4, 2> request_texture;
rtTextureSampler<float4,2> Last_reflection;

rtDeclareVariable(uint, radiance_ray_type, , );
rtDeclareVariable(uint, shadow_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, reflectors, , );

rtDeclareVariable(float3,   lightPos, , );

rtDeclareVariable(int,   FrameCount, , );

rtDeclareVariable(int,   PixelNum, , );
rtDeclareVariable(int,   PixelWidth, , );
rtDeclareVariable(int,   hasGlossy, , );



rtDeclareVariable(int2,   rasterSize, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );


rtDeclareVariable(uint, launch_index1D, rtLaunchIndex, );
struct PerRayData_radiance
{
 
  float3 result;
  float  importance;
  int   objectId;
  int depth;
  float t_hit;
  float reflectValue;
  float3 shadingNormal;
  int isReflectRay;
};
struct PerRayData_shadow
{
	float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );



__device__ __host__ float random( float2 p )
{
   // We need irrationals for pseudo randomness.
   // Most (all?) known transcendental numbers will (generally) work.
   const float2 r = make_float2(
  23.1406926327792690,  // e^pi (Gelfond's constant)
   2.6651441426902251); // 2^sqrt(2) (Gelfond�CSchneider constant)
   float temp =  cos( fmod( 123456789., 1e-7 + 256. * dot(p,r) ) ) ;
   return temp - floor(temp);
}

__device__ __host__ void ConcentricSampleDisk(float u1, float u2, float *dx,float*dy)
{
    float r, theta;
    // Map uniform random numbers to $[-1,1]^2$
    float sx = 2 * u1 - 1;
    float sy = 2 * u2 - 1;
	
    // Map square to $(r,\theta)$

    // Handle degeneracy at the origin
    if (sx == 0.0 && sy == 0.0) {
        *dx = 0.0;
        *dy = 0.0;
        return;
    }
    if (sx >= -sy) {
        if (sx > sy) {
            // Handle first region of disk
            r = sx;
            if (sy > 0.0) theta = sy/r;
            else          theta = 8.0f + sy/r;
        }
        else {
            // Handle second region of disk
            r = sy;
            theta = 2.0f - sx/r;
        }
    }
    else {
        if (sx <= sy) {
            // Handle third region of disk
            r = -sx;
            theta = 4.0f - sy/r;
        }
        else {
            // Handle fourth region of disk
            r = -sy;
            theta = 6.0f + sx/r;
        }
    }
    theta *= M_PI / 4.f;
     *dx = r * cosf(theta);
     *dy = r * sinf(theta);

}
static
__host__ __device__ __inline__ optix::float3 sample_phong_lobe( const optix::float2 &sample, float exponent, 
                                                                const optix::float3 &U, const optix::float3 &V, const optix::float3 &W, 
                                                                float &pdf, float &bdf_val )
{
  const float cos_theta = powf(sample.y, 1.0f/(exponent+1.0f) );

  const float phi = sample.x * 2.0f * M_PIf;
  const float sin_theta = sqrtf(1.0f - cos_theta*cos_theta);
  
  const float x = cosf(phi)*sin_theta;
  const float y = sinf(phi)*sin_theta;
  const float z = cos_theta;

  const float powered_cos = powf( cos_theta, exponent );
  pdf = (exponent+1.0f) / (2.0f*M_PIf) * powered_cos;
  bdf_val = (exponent+2.0f) / (2.0f*M_PIf) * powered_cos;  

  return x*U + y*V + z*W;
}
static
__device__ __inline__ void createONB( const optix::float3& n,
                                      optix::float3& U,
                                      optix::float3& V)
{
  using namespace optix;

  U = cross( n, make_float3( 0.0f, 1.0f, 0.0f ) );
  if ( dot(U, U) < 1.e-3f )
    U = cross( n, make_float3( 1.0f, 0.0f, 0.0f ) );
  U = normalize( U );
  V = cross( n, U );
}
#define N 128

RT_PROGRAM void addition_request()
{
	int index =launch_index1D;
	if(index >= PixelNum)
 			return;
 	uint x,y;
	uint PixPos = Pixels_Buffer[index];
    x = PixPos%rasterSize.x;
	y = PixPos/rasterSize.x;
	uint2 FinalPixelPos = make_uint2(x,y);
	 float3 ray_origin = make_float3(tex2D(request_texture, x, y));
	float reflectValue = tex2D(request_texture, x, y).w;
  PerRayData_radiance prd;
  PerRayData_shadow prd_s;
  prd_s.attenuation = make_float3(0);
  prd.result = make_float3(0);
  prd.importance = 1.f;
  prd.depth = 0;
  prd.t_hit = -1.f;
	
  // PerRayData_radiance prd2 = prd;
 //    PerRayData_radiance prd3 = prd;
 
 // PerRayData_radiance prd4 = prd;
 //	   PerRayData_radiance prd5 = prd;
  const int stepNum = 7;
  if( !isnan(ray_origin.x) ) 
  {
	  /*
	  float3 pivotArray[] =
	{
		make_float3(28.4509,10.7690, -36.0526),
		make_float3(28.4509, 10.7690 ,-14.2598),
		make_float3(28.4509 ,10.7690 ,26.6379),
		make_float3(12.0442 ,10.7690 ,26.6379),
		make_float3(13.9080, 32.9664 ,27.9780),
		make_float3(-26.8018, 32.9664, 7.1906 ),
		make_float3(-45.4884, 25.8473 ,7.1906),
	};
	*/
    if(!hasGlossy)
	{
		/*float3 minPos = pivotArray[0];
		float minD = length(ray_origin-minPos);
		for(int i = 1;i<stepNum;i++)
		{
			
			float3 pos = pivotArray[i];
			float currentD = length(ray_origin-pos);
			if(currentD<minD)
			{
				minD = currentD;
				minPos = pos;
			}
					
		}
		*/
		float3 V = normalize(ray_origin-eye_pos);
		float3 normal = make_float3(tex2D(normal_texture, x, y));
		float3 ray_direction = reflect(V, normal);
		optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(reflectors, ray, prd);
		prd.result = make_float3(1,0,0);
		reflection_buffer[FinalPixelPos] = make_float4( prd.result,1);
		return;	 
	}
	float3 V = normalize(ray_origin-eye_pos);
    float3 normal = make_float3(tex2D(normal_texture, x, y));
	float3 ray_direction = normalize(reflect(V, normal));
	float3 xo, yo;
    createONB(ray_direction, xo, yo);

	float2 randomArray;
	float3 glossy_direcion;
	 optix::Ray ray;
	 PerRayData_radiance prdArray;
	 float3 sumColor = make_float3(0,0,0);
	 float seedx = normal.x+ray_origin.y;
	 float seedy = normal.z+ray_origin.x;
	 float exponent = 30;
	 float bsdf_val,bsdf_pdf,costheta; 
	 float depthSum = 0;
	 float3 color ;
	 int usefulSample = 0;
	for(int i =0;i<N;i++)
	{
			prdArray = prd;
			randomArray.x =  random(make_float2(i*1.0/N*seedx,(i+0.5)/N)*seedy);
			randomArray.y =  random(make_float2((i+0.5)/N*seedy,i*1.0/N*seedx));
			glossy_direcion = sample_phong_lobe( randomArray, exponent, xo, yo, ray_direction, bsdf_pdf, bsdf_val );
			costheta = dot(glossy_direcion, normal);
			costheta = 1;
			bsdf_val = 1;
			bsdf_pdf = 1;
			if(bsdf_pdf > 0.0f&&costheta>0)
			{
				ray = optix::make_Ray(ray_origin, glossy_direcion, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
				rtTrace(reflectors, ray, prdArray);
				sumColor += prdArray.result*costheta*bsdf_val /bsdf_pdf;;
				depthSum += prdArray.t_hit*costheta*bsdf_val /bsdf_pdf;
				usefulSample++;
			}
	}
	
	
	float avgDepth;
	color = (sumColor)/usefulSample; 
	avgDepth = depthSum/usefulSample;
	reflection_buffer[FinalPixelPos] = make_float4(color, avgDepth);
  }
}
#ifdef NOGEOMETRY
RT_PROGRAM void gBufferAndRequest()
{
	   
	 float3 ray_origin  = eye_pos;
	 float2 tc = make_float2(launch_index.x+0.5, launch_index.y+0.5)/make_float2(rasterSize.x,rasterSize.y);
	 float3 imageSpot = getImagePos(tc);
	  PerRayData_radiance prd;
	  prd.result = make_float3(0);
	  prd.importance = 1.f;
	  prd.depth = 0;
	  prd.t_hit = -1.f;
	  prd.isReflectRay = 0;
	  float3 V = (imageSpot)-ray_origin;
	  float dist = sqrtf(dot(V,V));
	  V =  V/dist;
	
	  optix::Ray ray = optix::make_Ray(ray_origin,  V, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
	  rtTrace(reflectors, ray, prd);
	  float r_dis = prd.t_hit;
	  float3 worldPos = ray_origin + r_dis* V;
	  float objectId = prd.objectId;
	  float reflectValue = prd.reflectValue;
	  float3 normal = prd.shadingNormal;
	  posBuffer[launch_index] = make_float4(prd.result,objectId);
	  normalBuffer[launch_index] = make_float4(normal.x,normal.y,normal.z,reflectValue);
	  colorBuffer[launch_index] = make_float4(prd.result,reflectValue);

	  if(reflectValue>0.01)
	  {
	    prd.isReflectRay = 1;
	 
	   ray_origin = worldPos;
	   V = normalize(ray_origin-eye_pos);
		
	   float3 ray_direction = normalize(reflect(V, normal));

	    ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
	    rtTrace(reflectors, ray, prd);
	   r_dis = prd.t_hit;
		reflection_buffer[launch_index] = make_float4(prd.result,r_dis);
		addition_buffer[launch_index] = make_float4(prd.objectId,0,0,1);
	  }	//rtPrintf("object id:%d",prd.objectId);
	 return;	 
}

RT_PROGRAM void gBufferAndRequest_addition()
{
	int index =launch_index1D;
	if(index >= PixelNum)
 			return;
 	uint x,y;
	uint PixPos = Pixels_Buffer[index];
    x = PixPos%rasterSize.x;
	y = PixPos/rasterSize.x;
	uint2 FinalPixelPos = make_uint2(x,y);


	float3 ray_origin  = eye_pos;
	 float2 tc = make_float2(x+0.5, y+0.5)/make_float2(rasterSize.x,rasterSize.y);
	 float3 imageSpot = getImagePos(tc);
	  PerRayData_radiance prd;
	  prd.result = make_float3(0);
	  prd.importance = 1.f;
	  prd.depth = 0;
	  prd.t_hit = -1.f;
	  prd.isReflectRay = 0;
	  float3 V = (imageSpot)-ray_origin;
	  float dist = sqrtf(dot(V,V));
	  V =  V/dist;
	
	  optix::Ray ray = optix::make_Ray(ray_origin,  V, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
	  rtTrace(reflectors, ray, prd);
	  float r_dis = prd.t_hit;
	  float3 worldPos = ray_origin + r_dis* V;
	  float objectId = prd.objectId;
	  float reflectValue = prd.reflectValue;
	  float3 normal = prd.shadingNormal;
	  float3 diffuseColor = prd.result;
	    prd.isReflectRay = 1;
	 
	   ray_origin = worldPos;
	   V = normalize(ray_origin-eye_pos);
		
	   float3 ray_direction = normalize(reflect(V, normal));

	  ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
	  rtTrace(reflectors, ray, prd);
	  r_dis = prd.t_hit;
	  float3 outputColor;
	  if(reflectValue>0.001)
	  {
		  outputColor = diffuseColor*(1-reflectValue)+prd.result*reflectValue;
		 // outputColor.x += (1-outputColor.x)*3/4;
	  }
	  else
	     outputColor = diffuseColor;
	 
		//rtPrintf("object id:%d",prd.objectId);
	  reflection_buffer[FinalPixelPos] = make_float4(outputColor.x,outputColor.y,outputColor.z,r_dis);
		
	 return;	
}
#endif
RT_PROGRAM void reflection_request()
{
	//return;
	if(tex2D(normal_texture, launch_index.x, launch_index.y).w<0.01)
		return;
  float3 ray_origin = make_float3(tex2D(request_texture, launch_index.x, launch_index.y));
  // rtPrintf("x,y %d,%d\n",launch_index.x, launch_index.y);
 /*
  if(launch_index.y<=91||launch_index.y>=94)
	  return;*/
  float reflectValue = tex2D(request_texture, launch_index.x, launch_index.y).w;
  PerRayData_radiance prd;
  PerRayData_shadow prd_s;
  prd_s.attenuation = make_float3(0);
  prd.result = make_float3(0);
  prd.importance = 1.f;
  prd.depth = 0;
  prd.t_hit = -1.f;
  prd.isReflectRay = 1;
		
  // PerRayData_radiance prd2 = prd;
 //    PerRayData_radiance prd3 = prd;
 
 // PerRayData_radiance prd4 = prd;
 //	   PerRayData_radiance prd5 = prd;
 
  if( !isnan(ray_origin.x) ) 
  {
    if(!hasGlossy)
	{
		float3 V = normalize(ray_origin-eye_pos);
		float3 normal = make_float3(tex2D(normal_texture, launch_index.x, launch_index.y));
   
		float3 ray_direction = normalize(reflect(V, normal));

		/*float3 L = lightPos-ray_origin;
		float dist = sqrtf(dot(L,L));
		float3 ray_direction_s = L/dist;
		optix::Ray ray_s = optix::make_Ray(ray_origin, 
		ray_direction_s, 
		shadow_ray_type, 
		scene_epsilon, 
		dist);
		rtTrace(reflectors, ray_s, prd_s);
		float shadow = (prd_s.attenuation.x>0)?1:0;*/
		optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(reflectors, ray, prd);
	//shadow = 0;
		float r_dis = prd.t_hit;
		float3 reflectPos = ray_origin+ray_direction*r_dis;
	/*rtPrintf("eye_pos:(%f,%f��%f)\n",eye_pos.x,eye_pos.y,eye_pos.z);
		rtPrintf("wordldPos:(%f,%f��%f)\n",ray_origin.x,ray_origin.y,ray_origin.z);
		rtPrintf("reflectPos:(%f,%f��%f)\n",reflectPos.x,reflectPos.y,reflectPos.z);*/
		
		reflection_buffer[launch_index] = make_float4(prd.result,r_dis);
		addition_buffer[launch_index] = make_float4(prd.objectId,0,0,1);
		//rtPrintf("object id:%d",prd.objectId);
		return;	 
	}
	float3 V = normalize(ray_origin-eye_pos);
    float3 normal = make_float3(tex2D(normal_texture, launch_index.x, launch_index.y));
	float3 ray_direction = normalize(reflect(V, normal));
	float3 xo, yo;
    createONB(ray_direction, xo, yo);

	float2 randomArray;
	float3 glossy_direcion;
	 optix::Ray ray;
	 PerRayData_radiance prdArray;
	 float3 sumColor = make_float3(0,0,0);
	 float seedx = normal.x+ray_origin.y;
	 float seedy = normal.z+ray_origin.x;
	 float exponent = 30;
	 float bsdf_val,bsdf_pdf,costheta; 
	 float depthSum = 0;
	 float3 color ;
	 int usefulSample = 0;
	 float reflectDis;
	{
		float3 V = normalize(ray_origin-eye_pos);
		float3 normal = make_float3(tex2D(normal_texture, launch_index.x, launch_index.y));
   
		float3 ray_direction = normalize(reflect(V, normal));

		/*float3 L = lightPos-ray_origin;
		float dist = sqrtf(dot(L,L));
		float3 ray_direction_s = L/dist;
		optix::Ray ray_s = optix::make_Ray(ray_origin, 
		ray_direction_s, 
		shadow_ray_type, 
		scene_epsilon, 
		dist);
		rtTrace(reflectors, ray_s, prd_s);
		float shadow = (prd_s.attenuation.x>0)?1:0;*/
		optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(reflectors, ray, prd);
	//shadow = 0;
		reflectDis= prd.t_hit;
	
		//rtPrintf("object id:%d",prd.objectId);
	 }

	for(int i =0;i<N;i++)
	{
			prdArray = prd;
			randomArray.x =  random(make_float2(i*1.0/N*seedx,(i+0.5)/N)*seedy);
			randomArray.y =  random(make_float2((i+0.5)/N*seedy,i*1.0/N*seedx));
			glossy_direcion = sample_phong_lobe( randomArray, exponent, xo, yo, ray_direction, bsdf_pdf, bsdf_val );
			costheta = dot(glossy_direcion, normal);
			costheta = 1;
			bsdf_val = 1;
			bsdf_pdf = 1;
			if(bsdf_pdf > 0.0f&&costheta>0)
			{
				ray = optix::make_Ray(ray_origin, glossy_direcion, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
				rtTrace(reflectors, ray, prdArray);
				sumColor += prdArray.result*costheta*bsdf_val /bsdf_pdf;;
				depthSum += prdArray.t_hit*costheta*bsdf_val /bsdf_pdf;
				usefulSample++;
			}
	}
	
	
	float avgDepth;
	color = (sumColor)/usefulSample; 
	avgDepth = depthSum/usefulSample;
	

	reflection_buffer[launch_index] = make_float4(color, reflectDis);
  }
}
RT_PROGRAM void reflection_exception()
{
  reflection_buffer[launch_index] = make_float4(0.f,0.f,1.f,1.f);
}

RT_PROGRAM void reflection_miss()
{
  prd_radiance.t_hit = RT_DEFAULT_MAX;
  prd_radiance.result = make_float3(1.f, 1.f, 1.f);
}