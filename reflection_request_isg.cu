#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtBuffer<float4, 2>         reflection_buffer;
rtBuffer<float4, 2>         addition_buffer;;
rtBuffer<uint,1>          Pixels_Buffer;
//rtBuffer<float4,2>          LastReflection_buffer;

rtTextureSampler<float4, 2> normal_texture;
rtTextureSampler<float4, 2> request_texture;
rtTextureSampler<float4,2> Last_reflection;

rtDeclareVariable(uint, radiance_ray_type, , );
rtDeclareVariable(uint, shadow_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, reflectors, , );
rtDeclareVariable(float3, eye_pos, , );
rtDeclareVariable(float3,   lightPos, , );

rtDeclareVariable(int,   FrameCount, , );

rtDeclareVariable(int,   PixelNum, , );
rtDeclareVariable(int,   PixelWidth, , );
rtDeclareVariable(int,   hasGlossy, , );


rtDeclareVariable(int2,   rasterSize, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );


rtDeclareVariable(uint, launch_index1D, rtLaunchIndex, );
struct PerRayData_radiance
{
 
  float3 result;
  float  importance;
  int   objectId;
  int depth;
  float t_hit;
};
struct PerRayData_shadow
{
	float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );



__device__ __host__ float random( float2 p )
{
   // We need irrationals for pseudo randomness.
   // Most (all?) known transcendental numbers will (generally) work.
   const float2 r = make_float2(
  23.1406926327792690,  // e^pi (Gelfond's constant)
   2.6651441426902251); // 2^sqrt(2) (Gelfond�CSchneider constant)
   float temp =  cos( fmod( 123456789., 1e-7 + 256. * dot(p,r) ) ) ;
   return temp - floor(temp);
}

__device__ __host__ void ConcentricSampleDisk(float u1, float u2, float *dx,float*dy)
{
    float r, theta;
    // Map uniform random numbers to $[-1,1]^2$
    float sx = 2 * u1 - 1;
    float sy = 2 * u2 - 1;
	
    // Map square to $(r,\theta)$

    // Handle degeneracy at the origin
    if (sx == 0.0 && sy == 0.0) {
        *dx = 0.0;
        *dy = 0.0;
        return;
    }
    if (sx >= -sy) {
        if (sx > sy) {
            // Handle first region of disk
            r = sx;
            if (sy > 0.0) theta = sy/r;
            else          theta = 8.0f + sy/r;
        }
        else {
            // Handle second region of disk
            r = sy;
            theta = 2.0f - sx/r;
        }
    }
    else {
        if (sx <= sy) {
            // Handle third region of disk
            r = -sx;
            theta = 4.0f - sy/r;
        }
        else {
            // Handle fourth region of disk
            r = -sy;
            theta = 6.0f + sx/r;
        }
    }
    theta *= M_PI / 4.f;
     *dx = r * cosf(theta);
     *dy = r * sinf(theta);

}
static
__host__ __device__ __inline__ optix::float3 sample_phong_lobe( const optix::float2 &sample, float exponent, 
                                                                const optix::float3 &U, const optix::float3 &V, const optix::float3 &W, 
                                                                float &pdf, float &bdf_val )
{
  const float cos_theta = powf(sample.y, 1.0f/(exponent+1.0f) );

  const float phi = sample.x * 2.0f * M_PIf;
  const float sin_theta = sqrtf(1.0f - cos_theta*cos_theta);
  
  const float x = cosf(phi)*sin_theta;
  const float y = sinf(phi)*sin_theta;
  const float z = cos_theta;

  const float powered_cos = powf( cos_theta, exponent );
  pdf = (exponent+1.0f) / (2.0f*M_PIf) * powered_cos;
  bdf_val = (exponent+2.0f) / (2.0f*M_PIf) * powered_cos;  

  return x*U + y*V + z*W;
}
static
__device__ __inline__ void createONB( const optix::float3& n,
                                      optix::float3& U,
                                      optix::float3& V)
{
  using namespace optix;

  U = cross( n, make_float3( 0.0f, 1.0f, 0.0f ) );
  if ( dot(U, U) < 1.e-3f )
    U = cross( n, make_float3( 1.0f, 0.0f, 0.0f ) );
  U = normalize( U );
  V = cross( n, U );
}
#define N 1024
RT_PROGRAM void addition_request()
{
	int index =launch_index1D;
	if(index >= PixelNum)
 			return;
 	uint x,y;
	uint PixPos = Pixels_Buffer[index];
    x = PixPos%rasterSize.x;
	y = PixPos/rasterSize.x;
	uint2 FinalPixelPos = make_uint2(x,y);
	 float3 ray_origin = make_float3(tex2D(request_texture, x, y));
	float reflectValue = tex2D(request_texture, x, y).w;
  PerRayData_radiance prd;
  PerRayData_shadow prd_s;
  prd_s.attenuation = make_float3(0);
  prd.result = make_float3(0);
  prd.importance = 1.f;
  prd.depth = 0;
  prd.t_hit = -1.f;
	
  // PerRayData_radiance prd2 = prd;
 //    PerRayData_radiance prd3 = prd;
 
 // PerRayData_radiance prd4 = prd;
 //	   PerRayData_radiance prd5 = prd;
 
  if( !isnan(ray_origin.x) ) 
  {
    if(!hasGlossy)
	{
		
		float3 V = normalize(ray_origin-eye_pos);
		float3 normal = make_float3(tex2D(normal_texture, x, y));
		float3 ray_direction = reflect(V, normal);
		float shadow = (prd_s.attenuation.x>0)?1:0;
		optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(reflectors, ray, prd);
		reflection_buffer[FinalPixelPos] = make_float4(prd.result,1);
		return;	 
	}
	float3 V = normalize(ray_origin-eye_pos);
    float3 normal = make_float3(tex2D(normal_texture, x, y));
	float3 ray_direction = normalize(reflect(V, normal));
	float3 xo, yo;
    createONB(ray_direction, xo, yo);

	float2 randomArray;
	float3 glossy_direcion;
	 optix::Ray ray;
	 PerRayData_radiance prdArray;
	 float3 sumColor = make_float3(0,0,0);
	 float seedx = normal.x+ray_origin.y;
	 float seedy = normal.z+ray_origin.x;
	 float exponent = 30;
	 float bsdf_val,bsdf_pdf,costheta; 
	 float depthSum = 0;
	 float3 color ;
	 int usefulSample = 0;
	for(int i =0;i<N;i++)
	{
			prdArray = prd;
			randomArray.x =  random(make_float2(i*1.0/N*seedx,(i+0.5)/N)*seedy);
			randomArray.y =  random(make_float2((i+0.5)/N*seedy,i*1.0/N*seedx));
			glossy_direcion = sample_phong_lobe( randomArray, exponent, xo, yo, ray_direction, bsdf_pdf, bsdf_val );
			costheta = dot(glossy_direcion, normal);
			if(bsdf_pdf > 0.0f&&costheta>0)
			{
				ray = optix::make_Ray(ray_origin, glossy_direcion, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
				rtTrace(reflectors, ray, prdArray);
				sumColor += prdArray.result*costheta*bsdf_val /bsdf_pdf;;
				depthSum += prdArray.t_hit*costheta*bsdf_val /bsdf_pdf;
				usefulSample++;
			}
	}
	
	
	float avgDepth;
	color = (sumColor)/usefulSample; 
	avgDepth = depthSum/usefulSample;
	reflection_buffer[FinalPixelPos] = make_float4(color, avgDepth);
  }
}
RT_PROGRAM void reflection_request()
{
	//return;
  float3 ray_origin = make_float3(tex2D(request_texture, launch_index.x, launch_index.y));
  //if(launch_index.x!=91||launch_index.y!=623)
	//  return;
 // rtPrintf("x,y %d,%d\n",launch_index.x, launch_index.y);
 /* if(launch_index.x<=91||launch_index.x>=94)
	  return;
  if(launch_index.y<=91||launch_index.y>=94)
	  return;*/
  float reflectValue = tex2D(request_texture, launch_index.x, launch_index.y).w;
  PerRayData_radiance prd;
  PerRayData_shadow prd_s;
  prd_s.attenuation = make_float3(0);
  prd.result = make_float3(0);
  prd.importance = 1.f;
  prd.depth = 0;
  prd.t_hit = -1.f;
	
  // PerRayData_radiance prd2 = prd;
 //    PerRayData_radiance prd3 = prd;
 
 // PerRayData_radiance prd4 = prd;
 //	   PerRayData_radiance prd5 = prd;
 
  if( !isnan(ray_origin.x) ) 
  {
    if(!hasGlossy)
	{

		float3 V = normalize(ray_origin-eye_pos);
		float3 normal = make_float3(tex2D(normal_texture, launch_index.x, launch_index.y));
   
		float3 ray_direction = normalize(reflect(V, normal));

		/*float3 L = lightPos-ray_origin;
		float dist = sqrtf(dot(L,L));
		float3 ray_direction_s = L/dist;
		optix::Ray ray_s = optix::make_Ray(ray_origin, 
		ray_direction_s, 
		shadow_ray_type, 
		scene_epsilon, 
		dist);
		rtTrace(reflectors, ray_s, prd_s);
		float shadow = (prd_s.attenuation.x>0)?1:0;*/
		optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(reflectors, ray, prd);
	//shadow = 0;
		float r_dis = prd.t_hit;
		float3 reflectPos = ray_origin+ray_direction*r_dis;
	/*rtPrintf("eye_pos:(%f,%f��%f)\n",eye_pos.x,eye_pos.y,eye_pos.z);
		rtPrintf("wordldPos:(%f,%f��%f)\n",ray_origin.x,ray_origin.y,ray_origin.z);
		rtPrintf("reflectPos:(%f,%f��%f)\n",reflectPos.x,reflectPos.y,reflectPos.z);*/
		reflection_buffer[launch_index] = make_float4(prd.result,r_dis);
		addition_buffer[launch_index] = make_float4(prd.objectId,0,0,1);
		//rtPrintf("object id:%d",prd.objectId);
		return;	 
	}
	float3 V = normalize(ray_origin-eye_pos);
    float3 normal = make_float3(tex2D(normal_texture, launch_index.x, launch_index.y));
	float3 ray_direction = normalize(reflect(V, normal));
	float3 xo, yo;
    createONB(ray_direction, xo, yo);

	float2 randomArray;
	float3 glossy_direcion;
	 optix::Ray ray;
	 PerRayData_radiance prdArray;
	 float3 sumColor = make_float3(0,0,0);
	 float seedx = normal.x+ray_origin.y;
	 float seedy = normal.z+ray_origin.x;
	 float exponent = 30;
	 float bsdf_val,bsdf_pdf,costheta; 
	 float depthSum = 0;
	 float3 color ;
	 int usefulSample = 0;
	for(int i =0;i<N;i++)
	{
			prdArray = prd;
			randomArray.x =  random(make_float2(i*1.0/N*seedx,(i+0.5)/N)*seedy);
			randomArray.y =  random(make_float2((i+0.5)/N*seedy,i*1.0/N*seedx));
			glossy_direcion = sample_phong_lobe( randomArray, exponent, xo, yo, ray_direction, bsdf_pdf, bsdf_val );
			costheta = dot(glossy_direcion, normal);
			if(bsdf_pdf > 0.0f&&costheta>0)
			{
				ray = optix::make_Ray(ray_origin, glossy_direcion, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
				rtTrace(reflectors, ray, prdArray);
				sumColor += prdArray.result*costheta*bsdf_val /bsdf_pdf;;
				depthSum += prdArray.t_hit*costheta*bsdf_val /bsdf_pdf;
				usefulSample++;
			}
	}
	
	
	float avgDepth;
	color = (sumColor)/usefulSample; 
	avgDepth = depthSum/usefulSample;
	

	reflection_buffer[launch_index] = make_float4(color, avgDepth);
  }
}
RT_PROGRAM void reflection_exception()
{
  reflection_buffer[launch_index] = make_float4(1.f,0.f,0.f,-1.f);
}

RT_PROGRAM void reflection_miss()
{
  prd_radiance.t_hit = RT_DEFAULT_MAX;
  prd_radiance.result = make_float3(1.f, 1.f, 1.f);
}