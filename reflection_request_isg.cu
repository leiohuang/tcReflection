#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtBuffer<float4, 2>         reflection_buffer;
rtBuffer<uint,1>          Pixels_Buffer;
//rtBuffer<float4,2>          LastReflection_buffer;

rtTextureSampler<float4, 2> normal_texture;
rtTextureSampler<float4, 2> request_texture;
rtTextureSampler<float4,2> Last_reflection;

rtDeclareVariable(uint, radiance_ray_type, , );
rtDeclareVariable(uint, shadow_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, reflectors, , );
rtDeclareVariable(float3, eye_pos, , );
rtDeclareVariable(float3,   lightPos, , );

rtDeclareVariable(int,   FrameCount, , );

rtDeclareVariable(int,   PixelNum, , );
rtDeclareVariable(int,   PixelWidth, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
  float t_hit;
};
struct PerRayData_shadow
{
	float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );



__device__ __host__ float random( float2 p )
{
   // We need irrationals for pseudo randomness.
   // Most (all?) known transcendental numbers will (generally) work.
   const float2 r = make_float2(
  23.1406926327792690,  // e^pi (Gelfond's constant)
   2.6651441426902251); // 2^sqrt(2) (Gelfond�CSchneider constant)
   float temp =  cos( fmod( 123456789., 1e-7 + 256. * dot(p,r) ) ) ;
   return temp - floor(temp);
}

__device__ __host__ void ConcentricSampleDisk(float u1, float u2, float *dx,float*dy)
{
    float r, theta;
    // Map uniform random numbers to $[-1,1]^2$
    float sx = 2 * u1 - 1;
    float sy = 2 * u2 - 1;
	
    // Map square to $(r,\theta)$

    // Handle degeneracy at the origin
    if (sx == 0.0 && sy == 0.0) {
        *dx = 0.0;
        *dy = 0.0;
        return;
    }
    if (sx >= -sy) {
        if (sx > sy) {
            // Handle first region of disk
            r = sx;
            if (sy > 0.0) theta = sy/r;
            else          theta = 8.0f + sy/r;
        }
        else {
            // Handle second region of disk
            r = sy;
            theta = 2.0f - sx/r;
        }
    }
    else {
        if (sx <= sy) {
            // Handle third region of disk
            r = -sx;
            theta = 4.0f - sy/r;
        }
        else {
            // Handle fourth region of disk
            r = -sy;
            theta = 6.0f + sx/r;
        }
    }
    theta *= M_PI / 4.f;
     *dx = r * cosf(theta);
     *dy = r * sinf(theta);

}
static
__host__ __device__ __inline__ optix::float3 sample_phong_lobe( const optix::float2 &sample, float exponent, 
                                                                const optix::float3 &U, const optix::float3 &V, const optix::float3 &W, 
                                                                float &pdf, float &bdf_val )
{
  const float cos_theta = powf(sample.y, 1.0f/(exponent+1.0f) );

  const float phi = sample.x * 2.0f * M_PIf;
  const float sin_theta = sqrtf(1.0f - cos_theta*cos_theta);
  
  const float x = cosf(phi)*sin_theta;
  const float y = sinf(phi)*sin_theta;
  const float z = cos_theta;

  const float powered_cos = powf( cos_theta, exponent );
  pdf = (exponent+1.0f) / (2.0f*M_PIf) * powered_cos;
  bdf_val = (exponent+2.0f) / (2.0f*M_PIf) * powered_cos;  

  return x*U + y*V + z*W;
}
static
__device__ __inline__ void createONB( const optix::float3& n,
                                      optix::float3& U,
                                      optix::float3& V)
{
  using namespace optix;

  U = cross( n, make_float3( 0.0f, 1.0f, 0.0f ) );
  if ( dot(U, U) < 1.e-3f )
    U = cross( n, make_float3( 1.0f, 0.0f, 0.0f ) );
  U = normalize( U );
  V = cross( n, U );
}
RT_PROGRAM void addition_request()
{
	int index = launch_index.y * PixelWidth  + launch_index.x;
	if(index >= PixelNum)
 			return;
 	uint x,y;
 //	uint PixPos = Pixels_Buffer[index];
	uint PixPos = Pixels_Buffer[index];
    x = PixPos%1024;
	y = PixPos/1024;
	//x =  launch_index.x;
    //y =  launch_index.y;
	uint2 FinalPixelPos = make_uint2(x,y);
	 float3 ray_origin = make_float3(tex2D(request_texture, x, y));
  float reflectValue = tex2D(request_texture, x, y).w;
  PerRayData_radiance prd;
  PerRayData_shadow prd_s;
  prd_s.attenuation = make_float3(0);
  prd.result = make_float3(0);
  prd.importance = 1.f;
  prd.depth = 0;
  prd.t_hit = -1.f;
	
  // PerRayData_radiance prd2 = prd;
 //    PerRayData_radiance prd3 = prd;
 
 // PerRayData_radiance prd4 = prd;
 //	   PerRayData_radiance prd5 = prd;
 
  if( !isnan(ray_origin.x) ) 
  {
    if(1)
	{
		float3 V = normalize(ray_origin-eye_pos);
		float3 normal = make_float3(tex2D(normal_texture, x, y));
		float3 ray_direction = reflect(V, normal);
		float3 L = lightPos-ray_origin;
		float dist = sqrtf(dot(L,L));
		float3 ray_direction_s = L/dist;
		optix::Ray ray_s = optix::make_Ray(ray_origin, 
		ray_direction_s, 
		shadow_ray_type, 
		scene_epsilon, 
		dist);
		rtTrace(reflectors, ray_s, prd_s);
		float shadow = (prd_s.attenuation.x>0)?1:0;
		optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(reflectors, ray, prd);
		reflection_buffer[FinalPixelPos] = make_float4(prd.result,1);
		return;	 
	}
  }
 	
}
RT_PROGRAM void reflection_request()
{
	//return;
  float3 ray_origin = make_float3(tex2D(request_texture, launch_index.x, launch_index.y));
  float reflectValue = tex2D(request_texture, launch_index.x, launch_index.y).w;
  PerRayData_radiance prd;
  PerRayData_shadow prd_s;
  prd_s.attenuation = make_float3(0);
  prd.result = make_float3(0);
  prd.importance = 1.f;
  prd.depth = 0;
  prd.t_hit = -1.f;
	
  // PerRayData_radiance prd2 = prd;
 //    PerRayData_radiance prd3 = prd;
 
 // PerRayData_radiance prd4 = prd;
 //	   PerRayData_radiance prd5 = prd;
 
  if( !isnan(ray_origin.x) ) 
  {
    if(0)
	{
		float3 V = normalize(ray_origin-eye_pos);
		float3 normal = make_float3(tex2D(normal_texture, launch_index.x, launch_index.y));
   
		float3 ray_direction = normalize(reflect(V, normal));

		/*float3 L = lightPos-ray_origin;
		float dist = sqrtf(dot(L,L));
		float3 ray_direction_s = L/dist;
		optix::Ray ray_s = optix::make_Ray(ray_origin, 
		ray_direction_s, 
		shadow_ray_type, 
		scene_epsilon, 
		dist);
		rtTrace(reflectors, ray_s, prd_s);
		float shadow = (prd_s.attenuation.x>0)?1:0;*/
		optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(reflectors, ray, prd);
	//shadow = 0;
		float r_dis = prd.t_hit;
		float3 reflectPos = ray_origin+ray_direction*r_dis;
		reflection_buffer[launch_index] = make_float4(prd.result,r_dis);
		return;	 
	}
	float3 V = normalize(ray_origin-eye_pos);
    float3 normal = make_float3(tex2D(normal_texture, launch_index.x, launch_index.y));
	float3 ray_direction = normalize(reflect(V, normal));
	float3 xo, yo;
    createONB(ray_direction, xo, yo);
#define N 11
	float2 randomArray[N];
	float rx[N],ry[N];
	float3 glossy_direcion[N];
	 optix::Ray ray[N];
	 PerRayData_radiance prdArray[N];
	 float3 sumColor = make_float3(0,0,0);
	 float seedx = normal.x+ray_origin.y;
	 float seedy = normal.z+ray_origin.x;
	 float exponent = 3;
	 float bsdf_val[N];
	 float bsdf_pdf[N];
	 float costheta[N];
	for(int i =0;i<N;i++)
	{
		prdArray[i] = prd;
		randomArray[i].x =  random(make_float2(i*1.0/N*seedx,(i+0.5)/N)*seedy);
		randomArray[i].y =  random(make_float2((i+0.5)/N*seedy,i*1.0/N*seedx));

		//ConcentricSampleDisk(randomArray[2*i],randomArray[2*i+1],&rx[i],&ry[i]);
		//rx[i] = randomArray[2*i];
		//ry[i] = randomArray[2*i+1];
		/*float angle = -5.5;
		rx[i] = sin(angle);
		ry[i] = cos(angle);*/
		glossy_direcion[i] = sample_phong_lobe( randomArray[i], 50, xo, yo, ray_direction, bsdf_pdf[i], bsdf_val[i] );
		costheta[i] = dot(glossy_direcion[i], normal);
		ray[i] = optix::make_Ray(ray_origin, glossy_direcion[i], radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
	}
	

	rtTrace(reflectors, ray[0], prdArray[0]);
	rtTrace(reflectors, ray[1], prdArray[1]);
 	rtTrace(reflectors, ray[2], prdArray[2]);
 	rtTrace(reflectors, ray[3], prdArray[3]);
 	rtTrace(reflectors, ray[4], prdArray[4]);
	rtTrace(reflectors, ray[5], prdArray[5]);
 	rtTrace(reflectors, ray[6], prdArray[6]);
 	rtTrace(reflectors, ray[7], prdArray[7]);
 	rtTrace(reflectors, ray[8], prdArray[8]);
 	rtTrace(reflectors, ray[9], prdArray[9]);
	rtTrace(reflectors, ray[10], prdArray[10]);
 	/*rtTrace(reflectors, ray[11], prdArray[11]);
 	rtTrace(reflectors, ray[12], prdArray[12]);
 	rtTrace(reflectors, ray[13], prdArray[13]);
 	/*rtTrace(reflectors, ray[14], prdArray[14]);
	rtTrace(reflectors, ray[15], prdArray[15]);
 	/*rtTrace(reflectors, ray[16], prdArray[16]);
 	rtTrace(reflectors, ray[17], prdArray[17]);
 	rtTrace(reflectors, ray[18], prdArray[18]);
 	rtTrace(reflectors, ray[19], prdArray[19]);
	rtTrace(reflectors, ray[20], prdArray[20]);
 	rtTrace(reflectors, ray[21], prdArray[21]);
 	rtTrace(reflectors, ray[22], prdArray[22]);
 	rtTrace(reflectors, ray[23], prdArray[23]);
 	rtTrace(reflectors, ray[24], prdArray[24]);
	rtTrace(reflectors, ray[25], prdArray[25]);
 	rtTrace(reflectors, ray[26], prdArray[26]);
 	rtTrace(reflectors, ray[27], prdArray[27]);
 	rtTrace(reflectors, ray[28], prdArray[28]);
 	rtTrace(reflectors, ray[29], prdArray[29]);
 
	/*rtTrace(reflectors, ray[30], prdArray[30]);
 	rtTrace(reflectors, ray[31], prdArray[31]);
 	rtTrace(reflectors, ray[32], prdArray[32]);
 	rtTrace(reflectors, ray[33], prdArray[33]);
 	rtTrace(reflectors, ray[34], prdArray[34]);
	rtTrace(reflectors, ray[35], prdArray[35]);
 	rtTrace(reflectors, ray[36], prdArray[36]);
 	rtTrace(reflectors, ray[37], prdArray[37]);
 	rtTrace(reflectors, ray[38], prdArray[38]);
 	rtTrace(reflectors, ray[39], prdArray[39]);
 */
 

 	float depthSum = 0;
	float depthCount = 0;
	float r ;
	float3 color ;
	sumColor = make_float3(0,0,0);
	for(int i=0;i<N;i++)
	{
	//if(prdArray[i].t_hit<8)
		float3 color = prdArray[i].result*costheta[i]*bsdf_val[i] /bsdf_pdf[i];
		sumColor +=color;
		float len = prdArray[i].t_hit;
		depthSum+=len; 

	}
	float avgDepth;
	color = (sumColor)/N; 
	avgDepth = depthSum/N;
	/*color.x = rx[0];
	color.y = ry[0];
	color.z = 0;*/
	//color = normalize(rx[0]*a*xo+ry[0]*a*yo);
	 reflection_buffer[launch_index] = make_float4(color, avgDepth);
	
/*#define  SampleCount  4
	float3 sumColor = make_float3(0,0,0);
	int count = 1;
	//int i=1;
	PerRayData_radiance prdArray[4];
	optix::Ray rayArray[4];
	for(int i =0;i<SampleCount;i++)
	{
		prdArray[i] = prd;
		float seed1= i*1.0/(SampleCount+1),seed2 = (i+1)*1.0/(SampleCount+1);
		float random1 = random(make_float2(seed1,seed2));
		float random2 = random(make_float2(seed2,seed1));
		float r1x,r1y;
		ConcentricSampleDisk(random1,random2,&r1x,&r1y);
		float3 glossy_direcion1 = ray_direction+r1x*a*xo+r1y*a*yo;
		rayArray[i] = optix::make_Ray(ray_origin, glossy_direcion1, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		//rtTrace(reflectors, ray2, prdArray[i]);
		//sumColor+=prdArray[i].result;
	}
	rtTrace(reflectors, rayArray[0], prdArray[0]);
	rtTrace(reflectors, rayArray[1], prdArray[1]);
	rtTrace(reflectors, rayArray[2], prdArray[2]);
	rtTrace(reflectors, rayArray[3], prdArray[3]);
	rtTrace(reflectors, ray_s, prd_s);
	float shadow = (prd_s.attenuation.x>0)?1:0;
	float3 color = prdArray[0].result;
    reflection_buffer[launch_index] = make_float4(color, shadow);
*/	
  }
}
RT_PROGRAM void reflection_exception()
{
  reflection_buffer[launch_index] = make_float4(1.f,0.f,0.f,-1.f);
}

RT_PROGRAM void reflection_miss()
{
  prd_radiance.t_hit = RT_DEFAULT_MAX;
  prd_radiance.result = make_float3(1.f, 1.f, 1.f);
}