#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    objectId;
  int depth;
  float t_hit;
};

struct PerRayData_shadow
{
  float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtTextureSampler<float4, 2> diffuse_texture;
rtDeclareVariable(float3,   texcoord, attribute texcoord, );
rtDeclareVariable(float3,   lightPos, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(uint, max_depth, , );
rtDeclareVariable(uint, radiance_ray_type, , );
rtDeclareVariable(uint, shadow_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, reflectors, , );
rtDeclareVariable(int, id, ,);

RT_PROGRAM void closest_hit_radiance()
{
  prd_radiance.t_hit = t_hit;
   float3 hit_point = ray.origin + t_hit * ray.direction;
  float3 L =  normalize(lightPos-hit_point);
  float3 world_shade_normal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal));
  float kd=fmax(dot(world_shade_normal,L),0);
	 float ka=0.2;
  float3 color =make_float3((kd+ka)*tex2D(diffuse_texture, texcoord.x, texcoord.y));

  float fresnel = fresnel_schlick(dot(-ray.direction, shading_normal), 5.f, 0.9f);

  PerRayData_radiance refl_prd;
  refl_prd.importance = prd_radiance.importance * fresnel * optix::luminance(color);
  refl_prd.depth = prd_radiance.depth + 1;
  float3 result;
  // if(0)
 if(refl_prd.depth <= max_depth && refl_prd.importance > 0.05) 
  {

    optix::Ray refl_ray = optix::make_Ray(hit_point, reflect(ray.direction, shading_normal), 
                                          radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(reflectors, refl_ray, refl_prd);
     result = refl_prd.result * 0.2 + 0.8*color;
  } 
  else
  {
    result = make_float3(1.f,0.f,0.f);
  }
  prd_radiance.t_hit = t_hit;
  prd_radiance.objectId = id;
 
  prd_radiance.result= color ;//	prd_radiance.result = make_float3(1,0,0);

}
RT_PROGRAM void any_hit_shadow()
{
   prd_shadow.attenuation = make_float3(1,1,1);
   rtTerminateRay();
}
