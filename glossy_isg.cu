#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "commonStructs.h"
#include "helpers.h"

using namespace optix;

rtDeclareVariable(uint, radiance_ray_type, , );
rtDeclareVariable(uint, shadow_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, reflectors, , );
rtDeclareVariable(uint, max_depth, , );	

rtBuffer<BasicLight> lights;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

struct PerRayData_radiance
{
  
  float3 result;
  float  importance;
  int    objectId;
  int depth;
  float t_hit;
  float reflectValue;
   float3 shadingNormal;
    int isReflectRay;
};

struct PerRayData_shadow
{
  float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(float3,   lightPos, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int3, index_color, attribute index_color, ); 
rtDeclareVariable(float,reflectValue, ,);
rtDeclareVariable(float3,diffuse_Color,,);
rtDeclareVariable(int, id, ,);
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
RT_PROGRAM void closest_hit_radiance()
{
  prd_radiance.t_hit = t_hit;

  float3 hit_point = ray.origin + t_hit * ray.direction;
  
   float3 L =  normalize(lightPos-hit_point);
  float3 world_shade_normal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal));
  float kd=fmax(dot(world_shade_normal,L),0);
	 float ka=0.2;
  float3 color = (kd+ka)*make_float3(diffuse_Color.x, diffuse_Color.y, diffuse_Color.z);
	
  float fresnel = fresnel_schlick(dot(-ray.direction, shading_normal), 5.f, 0.9f);

  PerRayData_radiance refl_prd;
  refl_prd.importance = prd_radiance.importance * fresnel * optix::luminance(color);
  refl_prd.depth = prd_radiance.depth + 1;
  refl_prd.objectId = id+1;
  float3 result;
 /* rtPrintf("hit_point%f,%f,%f\n",hit_point.x,hit_point.y,hit_point.z);
  rtPrintf("lightPos%f,%f,%f\n",lightPos.x,lightPos.y,lightPos.z);
  rtPrintf("L%f,%f,%f\n",L.x,L.y,L.z);
  rtPrintf("%d,%d\n",launch_index.x,launch_index.y);
 */
 // if(0)
  if((refl_prd.depth <= max_depth ) )
  {

    optix::Ray refl_ray = optix::make_Ray(hit_point, reflect(ray.direction, shading_normal), 
                                          radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(reflectors, refl_ray, refl_prd);
    result = refl_prd.result * 0.20 + 0.80*color;
  } else
  {
    result = color;
  }
  prd_radiance.t_hit = t_hit;
  prd_radiance.objectId = id;
  prd_radiance.shadingNormal = shading_normal;
  
  prd_radiance.reflectValue = reflectValue;
 // color = make_float3(index_color.x,index_color.y,index_color.z);
  prd_radiance.result = color;
  //  prd_radiance.result = L;
}

RT_PROGRAM void any_hit_shadow()
{
   prd_shadow.attenuation = make_float3(1,1,1);
  rtTerminateRay();
}
