#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "commonStructs.h"
#include "helpers.h"

using namespace optix;

rtDeclareVariable(uint, radiance_ray_type, , );
rtDeclareVariable(uint, shadow_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, reflectors, , );
rtDeclareVariable(uint, max_depth, , );	

rtBuffer<BasicLight> lights;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

struct PerRayData_radiance
{
  
  float3 result;
  float  importance;
  int    objectId;
  int depth;
  float t_hit;
};

struct PerRayData_shadow
{
  float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(float3,   lightPos, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int3, index_color, attribute index_color, ); 

rtDeclareVariable(float3,diffuse_Color,,);
rtDeclareVariable(int, id, ,);
RT_PROGRAM void closest_hit_radiance()
{
  prd_radiance.t_hit = t_hit;

  float3 hit_point = ray.origin + t_hit * ray.direction;
  
   float3 L =  normalize(lightPos-hit_point);
  float3 world_shade_normal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal));
  float kd=fmax(dot(world_shade_normal,L),0);
	 float ka=0.2;
  float3 color = (kd+ka)*make_float3(diffuse_Color.x, diffuse_Color.y, diffuse_Color.z);
	
  float fresnel = fresnel_schlick(dot(-ray.direction, shading_normal), 5.f, 0.9f);

  PerRayData_radiance refl_prd;
  refl_prd.importance = prd_radiance.importance * fresnel * optix::luminance(color);
  refl_prd.depth = prd_radiance.depth + 1;
  refl_prd.objectId = id+1;
  float3 result;
  if(fabs(diffuse_Color.z- 0.66)<0.001)
  {
	   prd_radiance.result = color;
	   return;
  }
 // if(0)
  if((refl_prd.depth <= max_depth && refl_prd.importance > 0.05) )
  {

    optix::Ray refl_ray = optix::make_Ray(hit_point, reflect(ray.direction, shading_normal), 
                                          radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(reflectors, refl_ray, refl_prd);
    result = refl_prd.result * 0.20 + 0.80*color;
  } else
  {
    result = color;
  }
  prd_radiance.t_hit = t_hit;
   prd_radiance.objectId = id;
 // color = make_float3(index_color.x,index_color.y,index_color.z);
  prd_radiance.result = color;
}

RT_PROGRAM void any_hit_shadow()
{
   prd_shadow.attenuation = make_float3(1,1,1);
  rtTerminateRay();
}
