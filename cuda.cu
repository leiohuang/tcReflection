#include "hip/hip_runtime.h"
﻿//#include<stdio.h>
#include"hip/hip_runtime.h"
#include "macro.h"
//#include "Camera.h"
//#include<cudpp.h>
#include "nvMath.h"
#include "nvMatrix.h"
#include <hip/hip_vector_types.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#define UseNine 0
//#include <cuda_gl_interop.h>
//#include <cutil_gl_inline.h>
//#include <helper_cuda_gl.h>
#define UseTrial   1
#define DISMAXTHRES 3000.0
__constant__  float Cuda_MvpMat[16];
__constant__  float Cuda_MvpMatInv[16];
__constant__  float Cuda_CameraViewMat[16]; 
__constant__  float Cuda_CameraViewMatInv[16];
__constant__ float2 d_imageBBmin;
__constant__ float2 d_imageBBmax;
__constant__ float3 d_refCameraPos;
__constant__ float3 d_newCameraPos;
__constant__ float3 d_cameraVec;
__constant__ int Cuda_FramMent;
__constant__ int d_width,d_height;


//__host__ __device__  uint rastWidth = 1024,rastHeight=1024;
#define rastWidth rasterWidth
#define rastHeight rasterHeight
unsigned int  *g_PixelState;//in device address
unsigned int  *g_PixelSums;//in device address
unsigned int  *g_PixelPos; //in device address

hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

texture<float4,2,hipReadModeElementType> cuda_WorldPos_Tex;
texture<float4,2,hipReadModeElementType> cuda_WorlNormal_Tex;
texture<float4,2,hipReadModeElementType> cuda_Reflect_Tex;
texture<float4,2,hipReadModeElementType> cuda_RePro_Pixel;

#ifdef DIFFNORMAL
   texture<float4,2,hipReadModeElementType> cuda_Diff_Tex;
#endif

float4 *cuda_PBO_Buffer;
float2 *cuda_LastVec_Buffer;
float4 *cuda_test_Buffer;
float4 *cuda_diffNormBuffer;
__device__ float4* d_cudaPboBuffer;
__device__ float4* d_cudaTestBuffer;
__device__ float2* d_cudaLastBuffer;
__device__ float4* d_cudaDiffNorlBuffer;
//float4 *cuda_RePos_Buffer;
/*CUDPPHandle theCudpp;
CUDPPHandle scanplan;
*/
thrust::device_vector<uint> thrustSumVec;
thrust::device_vector<uint> thrustStateVec;
thrust::device_vector<uint> thrustPosVec;

__device__ int* d_sum;
__device__ int* d_state;
__device__ int* d_pos;

__device__ float2 toNDC(float4 temp)
{
	temp.x = temp.x / temp.w * 0.5 + 0.5;
	temp.y = temp.y / temp.w * 0.5 + 0.5;
	return make_float2(temp.x,temp.y)* make_float2(rastWidth,rastHeight);
}
__device__ int isOutOfRange(float2 tc)
{
	return tc.x>rasterWidth-0.5||tc.y>rasterHeight-0.5||tc.x<0||tc.y<0;
}

__device__ float4 cuda_Mat_Mut3(const float Mat4[16],float3 vec)
{
	float result_F[4];

	for(int i =0;i<4;i++)
		result_F[i] = (Mat4[i] * vec.x + Mat4[i+4] * vec.y +
		Mat4[i+8] * vec.z + Mat4[i+12] );
	float4 result_V = make_float4(result_F[0],result_F[1],result_F[2],result_F[3]);
	return result_V;
}

__device__ float4 Cude_Mul_Mat4(const float Mat4[16],float TmpVec4[4]){

	float result_F[4];
	for(int i =0;i<4;i++)
		result_F[i] = (Mat4[i] * TmpVec4[0] + Mat4[i+4] * TmpVec4[1] +
		Mat4[i+8] * TmpVec4[2] + Mat4[i+12] * TmpVec4[3]);
	return make_float4(result_F[0],result_F[1],result_F[2],result_F[3]);

	

}
__device__ float4 Cude_Mul_Mat4(const float Mat4[16],float4 TmpVec4){

	float result_F[4];
	for(int i =0;i<4;i++)
		result_F[i] = (Mat4[i] * TmpVec4.x + Mat4[i+4] * TmpVec4.y +
		Mat4[i+8] * TmpVec4.z + Mat4[i+12] * TmpVec4.w);
	return make_float4(result_F[0],result_F[1],result_F[2],result_F[3]);

	

}
__device__ float3 transFormToNdc(float3 temp)
{
	float4 ProPos = cuda_Mat_Mut3(Cuda_MvpMat,temp);
	float2 ndcTc=  toNDC(ProPos);
	return make_float3(ndcTc.x,ndcTc.y,ProPos.z/ProPos.w);
}
__device__ float2 normalizeVec(float2 moveVec)
{
	if(abs(moveVec.y)>abs(moveVec.x))
	{
		return moveVec/abs(moveVec.y);
	}
	else
	{
		return moveVec/abs(moveVec.x);
	}
}
__device__ int isConcave(float2 tc)
{
	float4 Tmpfloat4;
	Tmpfloat4 = tex2D(cuda_Diff_Tex,tc.x,tc.y);
	if(Tmpfloat4.x>=0.5||Tmpfloat4.y>=0.5||Tmpfloat4.z>=0.5)
		return 1;
	else
		return 0;

}
__device__ float Cuda_Compute_Cos(float2 TmpUV,float3 ReflectPos,int orignId)
{
	float3 WorldPos,WorldNormal;
	float4 Tmpfloat4;
	Tmpfloat4 = tex2D(cuda_WorlNormal_Tex,TmpUV.x,TmpUV.y);
	int NowId = (int)Tmpfloat4.w;
	if(NowId != orignId)
		return -100;
	WorldNormal = make_float3(Tmpfloat4.x,Tmpfloat4.y,Tmpfloat4.z);

	Tmpfloat4 = tex2D(cuda_WorldPos_Tex,TmpUV.x,TmpUV.y);
	WorldPos = make_float3(Tmpfloat4.x,Tmpfloat4.y,Tmpfloat4.z);

	float3 TmpReMirrorPos,TmpCam_Mirror_Normal,TmpCamera_Vec_Normal,TmpInsertPoint;

	TmpReMirrorPos =  abs(dot(WorldPos - ReflectPos,WorldNormal)) * (-2) * WorldNormal + ReflectPos;
	TmpCam_Mirror_Normal = normalize(cross(d_cameraVec,normalize(d_refCameraPos - TmpReMirrorPos)));
	TmpCamera_Vec_Normal = normalize(cross(TmpCam_Mirror_Normal,d_cameraVec));
	TmpInsertPoint = abs(dot(TmpCamera_Vec_Normal,d_refCameraPos - TmpReMirrorPos)/dot(TmpCamera_Vec_Normal,normalize(WorldPos-TmpReMirrorPos))) * normalize(WorldPos-TmpReMirrorPos) + TmpReMirrorPos;
	//TmpDis1 = dot(TmpInsertPoint - d_refCameraPos,d_cameraVec);
	//TmpDis2 = dot(TmpInsertPoint - d_refCameraPos,TmpInsertPoint - d_refCameraPos);
	//return  TmpDis2 - TmpDis1*TmpDis1 ;
	//return  dot(d_cameraVec,TmpInsertPoint - d_refCameraPos);
	return  dot(d_cameraVec,normalize(TmpInsertPoint - d_refCameraPos));
}
__device__ float3 getIntersectPosition(float3 position, float3 normal, float2 tc)
{
	float2 ndc = tc *2.0 - 1.0;
	//printf("ndc:%f,%f\n",ndc.x,ndc.y);
	float4 temp = make_float4(ndc.x,ndc.y,0,1);
	float4 imgPoint =Cude_Mul_Mat4(Cuda_MvpMatInv ,temp);
	float3 pointOnPlane = make_float3(imgPoint.x,imgPoint.y,imgPoint.z)/imgPoint.w;
	float3 D = pointOnPlane - d_refCameraPos;
	float3 hitPoint = d_refCameraPos + dot( position - d_refCameraPos,normal)/dot(D,normal) * D;
	return hitPoint;
}
__device__ float3 to3(float4 vec)
{
	return make_float3(vec.x,vec.y,vec.z);
}

__device__  float getIntersection(float3 pos1, float3 normal1, float3 pos2,float3 normal2)
{
	float3 v = normalize(cross(normal1,normal2));
	float3 palaneNormal = normalize(cross(v,normal2));
	float3 D = pos2 - pos1;
	printf("this normal(%f,%f,%f)\n",normal1.x,normal1.y,normal1.z);
	printf("this pos(%,%f,%f)\n",pos1.x,pos1.y,pos1.z);
		
	printf("neighbor normal(%f,%f,%f)\n",normal2.x,normal2.y,normal2.z);
	printf("neighbor pos(%,%f,%f)\n",pos2.x,pos2.y,pos2.z);


	//vec3 hitPoint = pos1 + dot( pos2 - pos1,palaneNormal)/dot(normal1,palaneNormal) * normal1;
	if(abs(dot(normal1,palaneNormal)) <0.00000001)
	{
		printf("right");
		return 0;
	}
	float t = dot(pos2-pos1,palaneNormal)/dot(normal1,palaneNormal);
	printf("t:%f",t);
	return t;
}
__device__ int sign(float value)
{
	if(value>0)
		return 1;
	else if(value==0)
		return 0;
	else 
		return -1;
}
__device__ float neiboghoodConcave(float2 tex, float2 step,float4 sourcePosition,float4 sourceNormal )
{	
	float4 neighborPos = tex2D(cuda_WorldPos_Tex,tex.x+step.x,tex.y+step.y);;
	float4 neighborNormal;
	if(neighborPos.w == sourcePosition.w)  // using right value
	{
	
		neighborNormal =  tex2D(cuda_WorlNormal_Tex,tex.x+step.x,tex.y+step.y);;
		
		float concave = getIntersection(to3(neighborPos),to3(neighborNormal),to3(sourcePosition),to3(sourceNormal));
		return sign(concave);
	}
	neighborPos =  tex2D(cuda_WorldPos_Tex,tex.x-step.x,tex.y-step.y);;
	if(neighborPos.w == sourcePosition.w)
	{
		neighborNormal = tex2D(cuda_WorlNormal_Tex,tex.x-step.x,tex.y-step.y);;
		float concave = getIntersection(to3(neighborPos),to3(neighborNormal),to3(sourcePosition),to3(sourceNormal));
		return sign(concave);
	}
	return 0;

}
__device__ bool isInBox(float2 centerUv,float2 projectUv)
{
	if(fabs(centerUv.x-projectUv.x)<0.5&&fabs(centerUv.y-projectUv.y)<0.5)
		return true;
	else 
		return false;
}

__device__ float3 getWorldPos(float2 tc, float depthR)
{
	//printf("get worldPos");
	float z = 1.0/depthR;
	//printf("tc:(%f,%f)\n",tc.x,tc.y);
	float2 xy = d_imageBBmin+ (d_imageBBmax-d_imageBBmin)*tc;
	xy = xy*(-z);
	//printf("xyz:(%f,%f)\n",xy.x,xy.y,z);
	
	float4 temp = Cude_Mul_Mat4(Cuda_CameraViewMatInv, make_float4(xy.x,xy.y,z,1));
	return make_float3(temp.x,temp.y,temp.z)/temp.w;
}

__device__ float getDepthRep(float3 worldPos)
{
	float4 tmp = Cude_Mul_Mat4(Cuda_CameraViewMat,make_float4(worldPos.x,worldPos.y,worldPos.z,1));
	return 1.0/tmp.z;
}

__device__ float neiboghoodPixel(float2 tex, float2 step,float4 sourceValue)
{
    float f0 = getDepthRep(make_float3(sourceValue));
	bool minusValid = true,plusValid = true;
	float4 temp;
	float plusValue,minusValue;
	temp =  tex2D(cuda_WorldPos_Tex,tex.x+step.x,tex.y+step.y);
	if(temp.w == sourceValue.w)
	{
		printf("sample  up pos:(%f,%f,%f)\n",temp.x,temp.y,temp.z);
		plusValue = getDepthRep(make_float3(temp));
		return plusValue;
	}
	
	temp =  tex2D(cuda_WorldPos_Tex,tex.x-step.x,tex.y-step.y);
	if(temp.w == sourceValue.w)
	{
		minusValue = getDepthRep(make_float3(temp));
		return minusValue;
	}
	else
	{
		return 0;
	}

}

__device__ void nextThreeStep(float2 moveVec,float2 candicate[3],float2 currentUv)
{
	#define PI 3.141592654f

	if(moveVec.x==0)
	{
		candicate[0]= make_float2(0,copysignf(1,moveVec.y));
		candicate[1] = make_float2(-1,copysignf(1,moveVec.y));
		candicate[2] = make_float2(1,copysignf(1,moveVec.y));
		candicate[0]+=currentUv;
		candicate[1]+=currentUv;
		candicate[2]+=currentUv;
		return;
	}
	float absTangleValue = abs(moveVec.y/moveVec.x);
	if(absTangleValue<tanf(PI/4/2))
	{
		
		candicate[0]= make_float2(copysignf(1,moveVec.x),0);
		candicate[1]= make_float2(copysignf(1,moveVec.x),-1);
		candicate[2] = make_float2(copysignf(1,moveVec.x),1);

	}
	else if (absTangleValue>tanf(PI*3/4/2))
	{
		
		candicate[0]= make_float2(0,copysignf(1,moveVec.y));
		candicate[1] = make_float2(-1,copysignf(1,moveVec.y));
		candicate[2] = make_float2(1,copysignf(1,moveVec.y));
	}
	else 
	{
		candicate[0]= make_float2(copysignf(1,moveVec.x),copysignf(1,moveVec.y));
		candicate[1] = make_float2(copysignf(1,moveVec.x),0);
		candicate[2] = make_float2(0,copysignf(1,moveVec.y));
	}
	candicate[0]+=currentUv;
	candicate[1]+=currentUv;
	candicate[2]+=currentUv;

}
enum planeType
{
	originType,
	validSearchType,
	invalidType,
	e_noReflectType,
};
#define PROJECTTHRES 0.02
class Plane
{
public:
	planeType m_type;
	float3 m_worldPos;
	float3 m_worldNormal;
	float m_dis;
	float3 m_reflectPos;
	float3 m_mirrorPos;
	float2 m_tc;
	int m_orginID;

	__device__ Plane()
	{
	}
	__device__ Plane(float2 tc)
	{
		m_tc = tc;
		float4 WorldTmp;
		WorldTmp = tex2D(cuda_WorldPos_Tex,tc.x,tc.y);
		m_orginID = (int)WorldTmp.w;
		m_worldPos = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		WorldTmp = tex2D(cuda_WorlNormal_Tex,tc.x,tc.y);
		m_worldNormal = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		m_dis = tex2D(cuda_Reflect_Tex,tc.x,tc.y).w;
	
	}
	__device__ Plane(float2 tc,int *pIsReflectd)
	{
		m_tc = tc;
		float4 WorldTmp;
		WorldTmp = tex2D(cuda_WorldPos_Tex,tc.x,tc.y);
		m_orginID = (int)WorldTmp.w;
		m_worldPos = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		WorldTmp = tex2D(cuda_WorlNormal_Tex,tc.x,tc.y);
		if(WorldTmp.w <0.01)
		{
			*pIsReflectd = 0;
		}
		m_worldNormal = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		m_dis = tex2D(cuda_Reflect_Tex,tc.x,tc.y).w;
	
	}
	__device__ float2 getTc()
	{
		return m_tc;
	}
	__device__ int getID()
	{
		return m_orginID;
	}
	__device__ int isNoReflect()
	{
		return (m_type== e_noReflectType);
	}
	__device__ int isInValid()
	{
		return (m_type== invalidType);
	}
#ifdef DIFFNORMAL
	__device__ float2 changeMoveVec(float2 moveVec,float2 * newProjectPos, int* isLocal)
	{
		//printf("origin Move:(%f,%f)\n",moveVec.x,moveVec.y);
		float2 nextStep;
		if(isInBox(m_tc+moveVec,m_tc))
		{
			nextStep = moveVec;
		}
		else
		{
			nextStep = normalizeVec(moveVec);
		}
		//printf("nextStep:(%f,%f)\n",nextStep.x,nextStep.y);
		//printf("local pos:(%f,%f,%f)",m_worldPos.x,m_worldPos.y,m_worldPos.z);
		float2 targetVec = m_tc+nextStep;
		float4 value = tex2D(cuda_Diff_Tex,m_tc.x,m_tc.y);
		float ddx = value.x,ddy = value.y,dxdy = value.z;
		//printf("ddx:%f,ddy:%f,dxdy:%f\n",ddx,ddy,dxdy);
		float3 predictPos = getIntersectPosition(m_worldPos,m_worldNormal,targetVec/make_float2(rastWidth,rastHeight));
		float f0 = getDepthRep(predictPos);
		float pridictF = f0 + 1/2.0*ddx*nextStep.x*nextStep.x+ 1/2.0*ddy*nextStep.y*nextStep.y+ dxdy*nextStep.x*nextStep.y;
		float3 worldPos = getWorldPos(targetVec/make_float2(rastWidth,rastHeight),pridictF);
		//printf("predict pos:(%f,%f,%f)\n",worldPos.x,worldPos.y,worldPos.z);
		float3 projectPlace = transFormToNdc(worldPos);
		*newProjectPos = make_float2(projectPlace.x,projectPlace.y);
		//printf("projectPlace:(%f,%f)\n",projectPlace.x,projectPlace.y);
		float2 newMove = make_float2(projectPlace.x,projectPlace.y)-m_tc;
		if(isInBox(*newProjectPos,m_tc))
		{
			*isLocal = 1;
		}
		else
		{
			*isLocal = 0;
		}
		//printf("newMove:(%f,%f)\n",newMove.x,newMove.y);
		return newMove;
	}
#endif


#define NORMALDIFF 0.001
#define DISDIFF 0.002
	__device__ bool isEqual(Plane& anotherPlane)
	{
		//printf("normal diff:%f, dis diff:%f\n",length(m_worldNormal-anotherPlane.m_worldNormal),dot(m_worldPos-anotherPlane.m_worldPos,m_worldNormal));
		if(length(m_worldNormal-anotherPlane.m_worldNormal)>NORMALDIFF)
		{
			//printf("normal diff,dis:%f,%d\n",length(m_worldNormal-anotherPlane.m_worldNormal),length(m_worldNormal-anotherPlane.m_worldNormal)>NORMALDIFF);
			///printf("(%f,%f,%f)\n",m_worldNormal.x,m_worldNormal.y,m_worldNormal.z);					
			//printf("(%f,%f,%f)\n",anotherPlane.m_worldNormal.x,anotherPlane.m_worldNormal.y,anotherPlane.m_worldNormal.z);
			return false;
		}
		if(dot(m_worldPos-anotherPlane.m_worldPos,m_worldNormal)>DISDIFF)
		{
			//printf("distance too mush:%f\n",dot(m_worldPos-anotherPlane.m_worldPos,m_worldNormal));
			return false;
		}
		return true;
	}
	// 查询是否是投影点 返回1 可以
	__device__ int isAbleFastProj(float2 reProjectTc)
	{
		 
		/*float4 WorldTmp = tex2D(cuda_WorlNormal_Tex,reProjectTc.x,reProjectTc.y);
		int reprjectId  = (int)WorldTmp.w;
		float3 m_worldNormal = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		WorldTmp = tex2D(cuda_WorldPos_Tex,reProjectTc.x,reProjectTc.y);
		float3 worldPos = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		*/
		
		Plane pixelPlane(reProjectTc);
		if(pixelPlane.isInValid())
		{
			//printf("is notVaid");
			return 0;
		}
		if(!isEqual(pixelPlane))
		{
			//printf("is NotEqual");
			return 0;
		}
		return 1;  
		// no need to do futher check; nearest sampling
		pixelPlane.setReflectedPos(*this);
		float DisPoint2Line;
		DisPoint2Line = pixelPlane.getDisToPath();
		if(PROJECTTHRES>DisPoint2Line)
		{
			//printf("within threshold");
			return 1;
		}
		//printf("finally");
		return 0;
	
	}
	
	__device__ void setPreviousPos( float d)
	{
		m_dis = d;
	
	}
	__device__ float3 getReflectedPos()
	{
		return m_reflectPos;
	}
	__device__ float getMirrorDepth()
	{
		float4 ProPos = cuda_Mat_Mut3(Cuda_MvpMat,m_mirrorPos);
		return ProPos.z/ProPos.w;
	}
	__device__ void setReflectedPos(Plane p)
	{
		m_reflectPos = p.getReflectedPos();
		if(m_orginID == p.getID())
		{
			m_type = validSearchType;
		}		
		else 
		{
			m_type = invalidType;
		}
	}
	__device__ void setReflectedPos()
	{
		float3 inComeDirection = m_worldPos - d_refCameraPos;
		float3 LookVec = normalize(inComeDirection);
		//计算反射光线方向
		float3 ReflectVec = normalize(reflect(LookVec,m_worldNormal));
		m_type = originType;
		m_reflectPos = m_worldPos + ReflectVec * m_dis;	
	}
	__device__ float3 getMirrorPos()
	{
		float3 reflectPos = m_reflectPos;
		m_mirrorPos = abs(dot(m_worldPos - reflectPos,m_worldNormal)) * (-2) * m_worldNormal + reflectPos;
		return m_mirrorPos;
	}
	__device__  float3 beneathPos(float3 sourcePos, float dis)
	{
		float3 inComeDirection = m_worldPos - sourcePos;
		float3 LookVec = normalize(inComeDirection);
		float3 ReflectVec = normalize(reflect(LookVec,m_worldNormal));

		float CosReCorner = dot(ReflectVec,m_worldNormal);
		float3 reflectedPos = m_worldPos + ReflectVec * dis;
		//镜像点的坐标
		float3 ReMirrorPos = dis * CosReCorner * 2 * (-1) * (m_worldNormal) + reflectedPos;

		//相机到反射面的距离
		return ReMirrorPos;
	}
	//得出以该点为平面的反射物体的镜像点
	__device__ float3 intersectPos()
	{
		getMirrorPos();
		float3 VecEye2Ref = normalize(m_mirrorPos - d_newCameraPos);
		return dot( m_worldPos -d_newCameraPos ,m_worldNormal)/ dot(VecEye2Ref,m_worldNormal) * VecEye2Ref + d_newCameraPos;
	}
	
	__device__ float3 intersetVirtualPos()
	{
		float3 ReMirrorPos =  abs(dot(m_worldPos - m_reflectPos,m_worldNormal)) * (-2) * m_worldNormal + m_reflectPos;


		//outPut[index] = make_float4(ReMirrorPos.x,ReMirrorPos.y,ReMirrorPos.z,1.0);
		//return;

		//求出该点与相机连线和平面的新交点
		float DisEye2Plane = abs(dot(d_newCameraPos - m_worldPos,m_worldNormal));
		float CoseTheta2 = abs(dot(normalize(d_newCameraPos - ReMirrorPos),m_worldNormal));
		float3 VecEye2Ref = normalize(ReMirrorPos - d_newCameraPos);
		float3 FinalPos2;
		//	FinalPos2 = d_newCameraPos + VecEye2Ref * (DisEye2Plane/CoseTheta2 );
		FinalPos2 = dot(m_worldPos -d_newCameraPos ,m_worldNormal)/ dot(VecEye2Ref,m_worldNormal) * VecEye2Ref + d_newCameraPos;
		return FinalPos2;
	}
	
	__device__ int checkConverge()
	{
#define CONVERGETHRES 5
		float3 FinalPos  = intersectPos();
		float3 ndcPos =  transFormToNdc(FinalPos);
		float2 ProPosUv =make_float2(ndcPos.x,ndcPos.y);
		//printf("!tc:(%f,%f)\n",m_tc.x,m_tc.y);
		//printf("!proPosUv(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		return length(m_tc-ProPosUv)<CONVERGETHRES;
	}
	__device__ float getSpherDistance()
	{
		float3 inComeDirection = m_worldPos - m_reflectPos;
		float3 LookVec = normalize(inComeDirection);
		//计算反射光线方向
		float3 reflectVec = normalize(reflect(LookVec,m_worldNormal));
		float3 cuttingPoint = m_worldPos+reflectVec*(dot(d_newCameraPos-m_worldPos,reflectVec));
		return length(cuttingPoint-d_newCameraPos);
	}
	__device__ float getDisToPath()
	{
		if(isInValid())
		{
			return 3000.0;
		}
		//求出新的镜像点
		return getSpherDistance();
		float3 ReMirrorPos =  getMirrorPos();
		//printf("!ReMirrorPos: (%f,%f,%f)\n",ReMirrorPos.x,ReMirrorPos.y,ReMirrorPos.z);
		//求相机与镜像点组成的平面的法线
		 
		float3 Cam_Mirror_Normal = normalize(cross(d_cameraVec,normalize(d_refCameraPos - ReMirrorPos)));
		//printf("!Cam_Mirror_Normal: (%f,%f,%f)\n",Cam_Mirror_Normal.x,Cam_Mirror_Normal.y,Cam_Mirror_Normal.z);

		//求相机轨迹所在法线
		float3 Camera_Vec_Normal = normalize(cross(Cam_Mirror_Normal,d_cameraVec));
		//printf("!Camera_Vec_Normal: (%f,%f,%f)\n",Camera_Vec_Normal.x,Camera_Vec_Normal.y,Camera_Vec_Normal.z);

		//分别求三个点与该平面的交点
		float3 InsertPoint = abs(dot(Camera_Vec_Normal,d_refCameraPos - ReMirrorPos)/dot(Camera_Vec_Normal,normalize(m_worldPos-ReMirrorPos))) * normalize(m_worldPos-ReMirrorPos) + ReMirrorPos;
		//printf("!InsertPoint: (%f,%f,%f)\n",InsertPoint.x,InsertPoint.y,InsertPoint.z);

		//求点与直线的距离

		float3 toNewPlace2 = d_newCameraPos-InsertPoint;
		float DisPoint2Line = length(toNewPlace2);
		//DisPoint2Line1  =  dot(normalize(InsertPoint1 - d_refCameraPos),d_cameraVec);

		//printf("!return value: (%f)\n",DisPoint2Line1);

		return DisPoint2Line;


	}
	__device__ Plane next3(float2 moveVec,float* minDis)
	{
	    *minDis = DISMAXTHRES;
		float2 candicate[3];
		Plane minPlane;
		nextThreeStep(moveVec,candicate,m_tc);
		for(int i = 0;i<3;i++)
		{

			float2 uv = candicate[i];
			if(isOutOfRange(uv))
			{
				continue;
			}
			Plane testPlan(uv);	
			testPlan.setReflectedPos(*this);
			if(testPlan.isInValid())
			{
			//	printf("invalid\n");
				continue;
			}
			float dis = testPlan.getDisToPath();
			//printf("point1(%f,%f,%f)\n",uv.x,uv.y,dis);
			if(*minDis>dis)
			{
				//printf("min\n");
				*minDis = dis;
				minPlane = testPlan;
			}
		}
		return minPlane;
	}
	__device__ Plane next9(float * minDis)
	{
	    *minDis = DISMAXTHRES;
		Plane minPlane;
		for(int dx = -1;dx<=1;dx++)
		{
			for(int dy = -1;dy<=1;dy++)
			{
				float2 uv = m_tc+make_float2(dx,dy);
				if(dx==0&&dy==0)
					continue;
				if(isOutOfRange(uv))
				{
					
					//printf("OutOfRange\n");
					continue;
				}
				Plane testPlan(uv);
				testPlan.setReflectedPos(*this);
				
				if(testPlan.isInValid())
				{
				//	printf("invalid\n");
					continue;
				}
				float dis = testPlan.getDisToPath();
				//printf("point1(%f,%f,%f)\n",uv.x,uv.y,dis);
				if(*minDis>dis)
				{
					//printf("min\n");
					*minDis = dis;
					minPlane = testPlan;
				}
			}
		}
		return minPlane;
	}
};

__device__ int isInSamePixel(float2 moveVec)
{
	if(abs(moveVec.x)<0.5&&abs(moveVec.y)<0.5)
		return 1;
	else
		return 0;
}
__device__ int nextStep(float2 moveVec,float2* pStepVec)
{
#define PI 3.141592654f
	if(length(moveVec)<0.717)
		return 0;
	if(moveVec.x==0)
	{
		*pStepVec = make_float2(0,copysignf(1,moveVec.y));
		return 1;
	}
	float absTangleValue = abs(moveVec.y/moveVec.x);
	if(absTangleValue<tanf(PI/4/2))
	{
		
		*pStepVec = make_float2(copysignf(1,moveVec.x),0);

	}
	else if (absTangleValue>tanf(PI*3/4/2))
	{
		
		*pStepVec = make_float2(0,copysignf(1,moveVec.y));
	}
	else 
	{
		*pStepVec = make_float2(copysignf(1,moveVec.x),copysignf(1,moveVec.y));
	}
	return 1;
}
__device__ int nextThreeStep(float2 moveVec,float2* candicate1,float2* candicate2,float2* candicate3,float2 currentUv)
{
	#define PI 3.141592654f
/*	if(length(moveVec)<0.717)
	{

		return 0;
	}*/
	if(moveVec.x==0)
	{
		*candicate1 = make_float2(0,copysignf(1,moveVec.y));
		*candicate2 = make_float2(-1,copysignf(1,moveVec.y));
		*candicate3 = make_float2(1,copysignf(1,moveVec.y));
		*candicate1+=currentUv;
		*candicate2+=currentUv;
		*candicate3+=currentUv;
		return 1;
	}
	float absTangleValue = abs(moveVec.y/moveVec.x);
	if(absTangleValue<tanf(PI/4/2))
	{
		
		*candicate1 = make_float2(copysignf(1,moveVec.x),0);
		*candicate2 = make_float2(copysignf(1,moveVec.x),-1);
		*candicate3 = make_float2(copysignf(1,moveVec.x),1);

	}
	else if (absTangleValue>tanf(PI*3/4/2))
	{
		
		*candicate1 = make_float2(0,copysignf(1,moveVec.y));
		*candicate2 = make_float2(-1,copysignf(1,moveVec.y));
		*candicate3 = make_float2(1,copysignf(1,moveVec.y));
	}
	else 
	{
		*candicate1 = make_float2(copysignf(1,moveVec.x),copysignf(1,moveVec.y));
		*candicate2 = make_float2(copysignf(1,moveVec.x),0);
		*candicate3 = make_float2(0,copysignf(1,moveVec.y));
	}
	*candicate1+=currentUv;
	*candicate2+=currentUv;
	*candicate3+=currentUv;
	return 1;
}
#define CONVERGE 5
#define OUTRANGE 2
#define OUTOBJECT 3
#define FASTPROJT 4
__device__ int Floor2Int(float f)
{
	return (int)f+0.5;
}


__device__ int threePointSearch(float2 currentPlace)
{

#define STEPNUMBER 25
	int x =Floor2Int(currentPlace.x-0.5);
	int y =Floor2Int(currentPlace.y-0.5);
	int index = y * rasterWidth + x;
	float2 currentUv = make_float2(currentPlace.x,currentPlace.y);
	
	int isReflectedPixel = 1;
	Plane fittingPlane(currentUv,&isReflectedPixel);
	if(0==isReflectedPixel)
	{
		d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,1,OUTRANGE);
		return;
	}
	//if(x!=502||y!=583)
	// return;
	
	
	
	//printf("1Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	//�����������ľ���
	//= d_newCameraPos + DisEye2Plane/abs(dot(VecEye2Ref,WorldNormal))* VecEye2Ref;
	fittingPlane.setReflectedPos();
	float3 FinalPos  = fittingPlane.intersectPos();
	float3 ndcPos =  transFormToNdc(FinalPos);
	float2 ProPosUv =make_float2(ndcPos.x,ndcPos.y);
	float rejectDepth = fittingPlane.getMirrorDepth();
	

	/*
	float3 worldPos = fittingPlane.m_worldPos;
	float3 worldNormal = fittingPlane.m_worldNormal;
	float2 step = make_float2(1,1);neiboghoodConcave(currentUv,step,tex2D(cuda_WorldPos_Tex,currentUv.x,currentUv.y), tex2D(cuda_WorlNormal_Tex,currentUv.x,currentUv.y));
	return;
	float3 upPosPredict = getIntersectPosition(worldPos,worldNormal,(currentUv+make_float2(0,step.y))/make_float2(1024.0,1024.0));
	float4 posValue;
	posValue = tex2D(cuda_WorldPos_Tex,currentUv.x,currentUv.y);
		
	printf("printdict up pos:(%f,%f,%f)\n",upPosPredict.x,upPosPredict.y,upPosPredict.z);
	float upPredictValue = getDepthRep(upPosPredict);
	float upValue = neiboghoodPixel(currentUv,make_float2(0,step.y),posValue);
	float	ddy = 2*(upValue-upPredictValue);
	printf("normal:(%f,%f,%f)",worldNormal.x,worldNormal.y,worldNormal.z);
	
	float  curvature = neiboghoodConcave(currentUv,step,tex2D(cuda_WorldPos_Tex,currentUv.x,currentUv.y), tex2D(cuda_WorlNormal_Tex,currentUv.x,currentUv.y));
	

	printf("ddy:%f",ddy);
	return;
	
	printf("three x,y:%d,%d\n",x,y);
	printf("pos:(%f,%f,%f)\n",worldPos.x,worldPos.y,worldPos.z);
	printf("normal:(%f,%f,%f)\n",worldNormal.x,worldNormal.y,worldNormal.z);
	printf("reflected pos:(%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	printf("intersect pos:(%f,%f,%f)\n",FinalPos.x,FinalPos.y,FinalPos.z);
	printf("refCamera:(%f,%f,%f)\n",d_refCameraPos.x,d_refCameraPos.y,d_refCameraPos.z);
	printf("newCameraPos:(%f,%f,%f)\n",d_newCameraPos.x,d_newCameraPos.y,d_newCameraPos.z);
	printf("project Pos:(%f,%f),depth:%f\n",ProPosUv.x,ProPosUv.y,rejectDepth);
	*/
		
	float2 MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
	
	//printf("ProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
	//printf("Move(%f,%f)\n",MoveVec.x,MoveVec.y);
	
	int isLocal;
	float formerDis = fittingPlane.getDisToPath();
	if(fittingPlane.isAbleFastProj(ProPosUv))
	{
		d_cudaPboBuffer[index] =   make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,FASTPROJT);
		return FASTPROJT;
	}
	int IterTime = 0;
#ifdef DIFFNORMAL
#endif
	/*
	printf("ProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
	printf("Move(%f,%f)\n",MoveVec.x,MoveVec.y);
	printf("dis:%f\n",formerDis);
	*/
	while(IterTime<STEPNUMBER/1024.0*rasterWidth)
	{
		//printf("Item:%d currentUv:(%f,%f)\n",IterTime,currentUv.x,currentUv.y);
		float minDis;
		Plane minPlane = fittingPlane.next3(MoveVec,&minDis);
		
		//printf("result uv(%f,%f) nextuv(%f,%f),dis:%f,formerDis:%f\n",currentUv.x,currentUv.y, minPlane.getTc().x, minPlane.getTc().y,minDis,formerDis);
		/*if(isLocal)
		{
					//printf("local Converge\n");
				d_cudaPboBuffer[index] =  make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
				return CONVERGE;		
		}*/
		if(minDis>formerDis)
		{
			if(length(MoveVec)<15/1024.0*rasterWidth&&formerDis<5)
			{
				
				d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
				//printf("non local Converge\n");
				return CONVERGE;
			}
			else
			{
				d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,-0.1,OUTOBJECT);
				//printf("not right minDis\n");	
				return OUTOBJECT;
			}
		}
		//TmpUv = TmpUv1;
		
		currentUv = minPlane.getTc();
		formerDis = minDis;
		fittingPlane = minPlane;

		FinalPos  = fittingPlane.intersectPos();
		ndcPos =  transFormToNdc(FinalPos);
	    ProPosUv =make_float2(ndcPos.x,ndcPos.y);
		rejectDepth = fittingPlane.getMirrorDepth();
		/*
		printf("ProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		printf("Move(%f,%f)\n",MoveVec.x,MoveVec.y);
		*/
		MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);


#ifdef DIFFNORMAL
	#endif
		/*
		printf("ProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		printf("Move(%f,%f)\n",MoveVec.x,MoveVec.y);
		printf("worldPos(%f,%f,%f),normal:(%f,%f,%f)\n",fittingPlane.m_worldPos.x,fittingPlane.m_worldPos.y,fittingPlane.m_worldPos.z,fittingPlane.m_worldNormal.x,fittingPlane.m_worldNormal.y,fittingPlane.m_worldNormal.z);
		*/
		IterTime++;
	}
	d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTRANGE);
				
	//printf("！x:%d y:%d: %f,%f,%f,%f\n",x,y,d_cudaPboBuffer[index] .x,d_cudaPboBuffer[index] .y,d_cudaPboBuffer[index] .z,d_cudaPboBuffer[index] .w);
	
	
	return OUTRANGE;
	
}
__device__ int hybridPointSearch(float2 currentPlace)
{
	#define STEPNUMBER 25
	int x =Floor2Int(currentPlace.x-0.5);
	int y =Floor2Int(currentPlace.y-0.5);
	int index = y * rasterWidth + x;
	float2 currentUv = make_float2(currentPlace.x,currentPlace.y);

	int isReflectedPixel = 1;
	Plane fittingPlane(currentUv,&isReflectedPixel);
	if(0==isReflectedPixel)
	{
		d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,1,OUTRANGE);
		return;
	}
	fittingPlane.setReflectedPos();
	float3 FinalPos  = fittingPlane.intersectPos();
	float3 ndcPos =  transFormToNdc(FinalPos);
	float2 ProPosUv =make_float2(ndcPos.x,ndcPos.y);
	float rejectDepth = fittingPlane.getMirrorDepth();
	float2 MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
	float formerDis = fittingPlane.getDisToPath();

	
	if(fittingPlane.isAbleFastProj(ProPosUv))
	{
		if(fabs(ProPosUv.x-currentUv.x)<0.5&&(fabs(ProPosUv.y-currentUv.y)<0.5))
		{
			d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
			return CONVERGE;
		}
				
		d_cudaPboBuffer[index] =   make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,FASTPROJT);
		return FASTPROJT;
	}
	int IterTime = 0;
	while(IterTime<STEPNUMBER/1024.0*rasterWidth)
	{
		float minDis = DISMAXTHRES;
		float2 minUv;
		Plane minPlane; 
		if(isConcave(fittingPlane.getTc()))
		{
			minPlane = fittingPlane.next9(&minDis);
			
		}
		else
			minPlane = fittingPlane.next3(MoveVec,&minDis);
		if(minDis == DISMAXTHRES)
		{
			d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,-0.1,OUTOBJECT);
			return OUTOBJECT;
		}
		if(minDis>formerDis)
		{
			if(isInBox(currentUv,ProPosUv))
			{
					//printf("here\n");
				d_cudaPboBuffer[index] =  make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
					return CONVERGE;
			}
			else{
				d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
				return CONVERGE;
			}
			
		}
		
		currentUv = minPlane.getTc();
		formerDis = minDis;
		fittingPlane = minPlane;

		FinalPos  = fittingPlane.intersectPos();
		ndcPos =  transFormToNdc(FinalPos);
	    ProPosUv =make_float2(ndcPos.x,ndcPos.y);
		rejectDepth = fittingPlane.getMirrorDepth();
		MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
		/*
		printf("ProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		printf("Move(%f,%f)\n",MoveVec.x,MoveVec.y);
		printf("worldPos(%f,%f,%f),normal:(%f,%f,%f)\n",fittingPlane.m_worldPos.x,fittingPlane.m_worldPos.y,fittingPlane.m_worldPos.z,fittingPlane.m_worldNormal.x,fittingPlane.m_worldNormal.y,fittingPlane.m_worldNormal.z);
		*/
		IterTime++;
	}
	//printf("out range\n");
			
	d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTRANGE);		
	return OUTRANGE;
}
__device__ int postIndex(float2 tc)
{
	int x =Floor2Int(tc.x-0.5);
	int y =Floor2Int(tc.y-0.5);
	return  y * rasterWidth + x;
}
__device__ int ninePointPostSerach(float2 currentPlace)
{

	#define STEPNUMBER 25
	int x =Floor2Int(currentPlace.x-0.5);
	int y =Floor2Int(currentPlace.y-0.5);
	int index = y * rasterWidth + x;
	
	float2 currentUv = make_float2(currentPlace.x,currentPlace.y);
	float2 initUv = currentUv;
	int isReflectedPixel = 1;
	Plane fittingPlane(currentUv,&isReflectedPixel);
	if(0==isReflectedPixel)
	{
		d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,1,OUTRANGE);
		return;
	}
	/*
	if(x!=502||y!=583)
	 return;
	*/
	
	//printf("1Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	//�����������ľ���
	//= d_newCameraPos + DisEye2Plane/abs(dot(VecEye2Ref,WorldNormal))* VecEye2Ref;
	fittingPlane.setReflectedPos();
	float3 FinalPos  = fittingPlane.intersectPos();
	float3 ndcPos =  transFormToNdc(FinalPos);
	float2 ProPosUv =make_float2(ndcPos.x,ndcPos.y);
	float rejectDepth = fittingPlane.getMirrorDepth();
	float2 MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
	float formerDis = fittingPlane.getDisToPath();

	if(fittingPlane.isAbleFastProj(ProPosUv))
	{
	//	printf("fitting\n");
		if(fabs(ProPosUv.x-currentUv.x)<0.5&&(fabs(ProPosUv.y-currentUv.y)<0.5))
		{
			d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
			return CONVERGE;
		}
				
		d_cudaPboBuffer[index] =   make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,FASTPROJT);
		return FASTPROJT;
	}
	int IterTime = 0;
	while(IterTime<STEPNUMBER/1024.0*rasterWidth)
	{
		float minDis = DISMAXTHRES;
		float2 minUv;
		Plane minPlane = fittingPlane.next9(&minDis);
		if(minDis == DISMAXTHRES)
		{
		//	printf("here\n");
			d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,-0.1,OUTOBJECT);
			return OUTOBJECT;
		}
		if(minDis>formerDis)
		{
			if(isInBox(currentUv,ProPosUv))
			{
					//printf("here\n");
				d_cudaPboBuffer[index] =  make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
					return CONVERGE;
			}
			else{
				d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
				return CONVERGE;
			}
			
		}
		
		currentUv = minPlane.getTc();
		formerDis = minDis;
		fittingPlane = minPlane;

		FinalPos  = fittingPlane.intersectPos();
		ndcPos =  transFormToNdc(FinalPos);
	    ProPosUv =make_float2(ndcPos.x,ndcPos.y);
		rejectDepth = fittingPlane.getMirrorDepth();
		MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
	
		IterTime++;
	}		
	d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTRANGE);		
	return OUTRANGE;
	
}

__device__ int ninePointSearch(float2 currentPlace)
{
#define STEPNUMBER 25
	int x =Floor2Int(currentPlace.x-0.5);
	int y =Floor2Int(currentPlace.y-0.5);
	int index = y * rasterWidth + x;
	float2 currentUv = make_float2(currentPlace.x,currentPlace.y);

	int isReflectedPixel = 1;
	Plane fittingPlane(currentUv,&isReflectedPixel);
	if(0==isReflectedPixel)
	{
		d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,1,OUTRANGE);
		return;
	}
	/*
	if(x!=502||y!=583)
	 return;
	*/
	
	//printf("1Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	//�����������ľ���
	//= d_newCameraPos + DisEye2Plane/abs(dot(VecEye2Ref,WorldNormal))* VecEye2Ref;
	fittingPlane.setReflectedPos();
	float3 FinalPos  = fittingPlane.intersectPos();
	float3 ndcPos =  transFormToNdc(FinalPos);
	float2 ProPosUv =make_float2(ndcPos.x,ndcPos.y);
	float rejectDepth = fittingPlane.getMirrorDepth();
	float2 MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
	float formerDis = fittingPlane.getDisToPath();

	/*
	float3 worldPos = fittingPlane.m_worldPos;
	float3 worldNormal = fittingPlane.m_worldNormal;
	
	
	printf("nine search: x,y:%d,%d\n",x,y);
	printf("pos:(%f,%f,%f)\n",worldPos.x,worldPos.y,worldPos.z);
	printf("normal:(%f,%f,%f)\n",worldNormal.x,worldNormal.y,worldNormal.z);
	printf("reflected pos:(%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	printf("intersect pos:(%f,%f,%f)\n",FinalPos.x,FinalPos.y,FinalPos.z);
	printf("refCamera:(%f,%f,%f)\n",d_refCameraPos.x,d_refCameraPos.y,d_refCameraPos.z);
	printf("newCameraPos:(%f,%f,%f)\n",d_newCameraPos.x,d_newCameraPos.y,d_newCameraPos.z);
	printf("project Pos:(%f,%f),depth:%f\n",ProPosUv.x,ProPosUv.y,rejectDepth);
	*/
	if(fittingPlane.isAbleFastProj(ProPosUv))
	{
	//	printf("fitting\n");
		if(fabs(ProPosUv.x-currentUv.x)<0.5&&(fabs(ProPosUv.y-currentUv.y)<0.5))
		{
			d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
			return CONVERGE;
		}
				
		d_cudaPboBuffer[index] =   make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,FASTPROJT);
		return FASTPROJT;
	}
	int IterTime = 0;
	while(IterTime<STEPNUMBER/1024.0*rasterWidth)
	{
		float minDis = DISMAXTHRES;
		float2 minUv;
		Plane minPlane = fittingPlane.next9(&minDis);
		if(minDis == DISMAXTHRES)
		{
		//	printf("here\n");
			d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,-0.1,OUTOBJECT);
			return OUTOBJECT;
		}
		if(minDis>formerDis)
		{
			if(isInBox(currentUv,ProPosUv))
			{
					//printf("here\n");
				d_cudaPboBuffer[index] =  make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
					return CONVERGE;
			}
			else{
				d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
				return CONVERGE;
			}
			
		}
		
		currentUv = minPlane.getTc();
		formerDis = minDis;
		fittingPlane = minPlane;

		FinalPos  = fittingPlane.intersectPos();
		ndcPos =  transFormToNdc(FinalPos);
	    ProPosUv =make_float2(ndcPos.x,ndcPos.y);
		rejectDepth = fittingPlane.getMirrorDepth();
		MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
		/*
		printf("ProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		printf("Move(%f,%f)\n",MoveVec.x,MoveVec.y);
		printf("worldPos(%f,%f,%f),normal:(%f,%f,%f)\n",fittingPlane.m_worldPos.x,fittingPlane.m_worldPos.y,fittingPlane.m_worldPos.z,fittingPlane.m_worldNormal.x,fittingPlane.m_worldNormal.y,fittingPlane.m_worldNormal.z);
		*/
		IterTime++;
	}
	//printf("out range\n");
			
	d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTRANGE);		
	return OUTRANGE;
	
}

__global__ void MyNewKernel(int width,int height)
{
	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;
	if( x >width || y> height)
			return;
	//if(x!=91||y!=623)
	//	return;
	
	//printf("x:%d,y:%d\n",x,y);
	/*if(x==1024)
	{
		printf("x:%d,y:%d\n",x,y);
	}*/
	int index = y * width + x;
	float2 currentUv = make_float2(x+0.5,y+0.5);
	//d_cudaPboBuffer[index] =   make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,-0.1,FASTPROJT);
	//return;		
	ninePointSearch(currentUv);
	//threePointSearch(currentUv);
	//hybridPointSearch(currentUv);

}
extern "C" void MappingContantPeremeter(int* w,int* h)
{


	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_width),w,sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_height),h,sizeof(int)));


}
/*
extern "C" void cudaSetRefCamera(CCamera * prefCamera)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_MvpMat),prefCamera->getMvpMat(),sizeof(float)*16,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_CameraViewMat),prefCamera->getModelViewMat();,sizeof(float)*16,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_refCameraPos),(float*)prefCamera->getCameraPos(),sizeof(float3),0,hipMemcpyHostToDevice));

	nv::matrix4f modelViewInv = inverse(nv::matrix4f((float*)prefCamera->getModelViewMat()));
	nv::matrix4f mvpInv = inverse(nv::matrix4f((float*)prefCamera->getMvpMat()));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_MvpMatInv),mvpInv.get_value(),sizeof(float)*16,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_CameraViewMatInv),modelViewInv.get_value(),sizeof(float)*16,0,hipMemcpyHostToDevice));
	

	
}
extern "C" void cudaSetCurrrentCamera(CCamera * pCurrentCamera)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_newCameraPos),pCurrentCamera->getCameraPos();,sizeof(float3),0,hipMemcpyHostToDevice));
	
}
extern "C" void cudaSetRefAndCurrentCamera(CCamera * prefCamera,CCamera * pCurrentCamera)
{
	cudaSetRefCamera(prefCamera);
	cudaSetCurrrentCamera(pCurrentCamera);
	float3 CameraVec = normalize(*pCurrentCamera->getCameraPos() - *prefCamera->getCameraPos());
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cameraVec),&CameraVec,sizeof(float3),0,hipMemcpyHostToDevice));

	
}*/
extern "C" void cudaSetImageFrame(nv::vec2f bbmin,nv::vec2f bbmax)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_imageBBmin),&bbmin,sizeof(nv::vec2f),0,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_imageBBmax),&bbmax,sizeof(nv::vec2f),0,hipMemcpyHostToDevice));

}
extern "C" void TransConstData(const float *MvpMat,const float *ViewMat,float3 *pRefCamera,float3 *pNewCameraPos)
{

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_MvpMat),MvpMat,sizeof(float)*16,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_CameraViewMat),ViewMat,sizeof(float)*16,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_refCameraPos),pRefCamera,sizeof(float3),0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_newCameraPos),pNewCameraPos,sizeof(float3),0,hipMemcpyHostToDevice));
	nv::matrix4f modelViewInv = inverse(nv::matrix4f((float*)ViewMat));
	nv::matrix4f mvpInv = inverse(nv::matrix4f((float*)MvpMat));
	float3 CameraVec = normalize(*pNewCameraPos - *pRefCamera);
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_MvpMatInv),mvpInv.get_value(),sizeof(float)*16,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_CameraViewMatInv),modelViewInv.get_value(),sizeof(float)*16,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cameraVec),&CameraVec,sizeof(float3),0,hipMemcpyHostToDevice));

}


__device__ bool colorEqual(float3 soourceColor, float3 targetColor)
{
	return length(soourceColor-targetColor)<0.1;
}
__global__ void MyFirstPassKernel(int width,int height,uint *PixelState,float4 * cuda_PBO_Buffer)
{

	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;
	if(x > width || y > height)
		//if(x!=224||y!=267)
			return;

	//if(x !=224 || y !=267)
	//	if(x !=117 || y !=712)
	//	return;

	int index = y * width + x;

	//printf("$%d,%d\n",x,y);

	float4 StateValue =tex2D(cuda_RePro_Pixel,x+0.5,y+0.5);
	float3 reflectColor = make_float3(StateValue.x,StateValue.y,StateValue.z);
	if(colorEqual(reflectColor,make_float3(1,0,0))||colorEqual(reflectColor,make_float3(0,1,0))||reflectColor.x>1.01||reflectColor.y>1.01)  // fetch the color in additional tex
	{
		d_state[index] = 1;
	}
	else
	{
		//printf("ddddddd\n");
		d_state[index] = 0;
	}

}
extern "C" void cudaPredict(int width,int height)
{
	hipEvent_t begin_t,end_t;
	checkCudaErrors( hipEventCreate(&begin_t) );
    checkCudaErrors( hipEventCreate(&end_t)  );

	hipEventRecord(begin_t,0);
	dim3 blockSize(16,16,1);
	dim3 gridSize(width/blockSize.x,height/blockSize.y,1);
	MyNewKernel<<<gridSize,blockSize>>>(width,height);
	hipEventRecord(end_t,0);
	hipEventSynchronize(end_t); 
	float costtime;
	checkCudaErrors( hipEventElapsedTime(&costtime, begin_t, end_t) );

	printf("cuda measura time:%f\n",costtime);
	checkCudaErrors( hipEventDestroy(begin_t) );
    checkCudaErrors( hipEventDestroy(end_t)  );
}

void mapThustResourse()
{
	thrust::device_ptr<uint> dp = &thrustStateVec[0];
	uint * raw_ptr = thrust :: raw_pointer_cast ( dp );  
}
__global__ void MyThridPass(uint *PixePos,uint *PixelState,uint *PixelSums);

extern "C" void RunFirstPass(dim3 BlockSize,dim3 GridSize,int width,int height)
{
	thrust::device_ptr<uint> dp = &thrustStateVec[0];
	uint * raw_ptr = thrust :: raw_pointer_cast ( dp );  
	MyFirstPassKernel<<<GridSize,BlockSize>>>(width,height,raw_ptr,cuda_PBO_Buffer);
	


}


__global__ void MyThridPass(uint *PixePos,uint *PixelState,uint *PixelSums)
{

	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;

	if(x > rastWidth || y > rastHeight)
		return;
	uint index = y * rasterWidth + x;
	if(PixelState[index] == 1)
	{
		uint Pos1 = PixelSums[index];
		PixePos[Pos1] = index;

	}
}
void InitThrust()
{	
	thrustSumVec = thrust::device_vector<uint>(rastWidth*rastHeight); 
	thrustStateVec = thrust::device_vector<uint>(rastWidth*rastHeight);
	thrustPosVec = thrust::device_vector<uint>(rastWidth*rastHeight);

	thrust::device_ptr<uint> d_pstate = &thrustStateVec[0];
	g_PixelState = thrust :: raw_pointer_cast ( d_pstate );  
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_state), &g_PixelState,sizeof(int*)));

	thrust::device_ptr<uint> d_psum = &thrustSumVec[0];
	g_PixelSums = thrust :: raw_pointer_cast ( d_psum );
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_sum), &g_PixelSums,sizeof(int*)));

	thrust::device_ptr<uint> dp = &thrustPosVec[0];
	g_PixelPos= thrust :: raw_pointer_cast ( dp );
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_pos), &g_PixelPos,sizeof(int*)));

}



extern "C" void RunThridPass(dim3 BlockSize ,dim3 GridSize)
{	  	
	thrust::device_ptr<uint> dp = &thrustPosVec[0];
	uint * raw_ptr_pos = thrust :: raw_pointer_cast ( dp );  

	thrust::device_ptr<uint> d_pstate = &thrustStateVec[0];
	uint * raw_ptr_state = thrust :: raw_pointer_cast ( d_pstate );  

	thrust::device_ptr<uint> d_psum = &thrustSumVec[0];
	uint * raw_ptr_sum = thrust :: raw_pointer_cast ( d_psum );
	MyThridPass<<<GridSize,BlockSize>>>(raw_ptr_pos,raw_ptr_state,raw_ptr_sum);

	//uint* h_state = thrust::raw_pointer_cast(d_pstate);
	//thrust::host_vector <uint> h_pos =thrustSumVec;
	//thrust::host_vector <uint> h_pos =thrustPosVec;
	/* 
	for(int y = 0;y<1024;y++)
	{
		for(int x = 0;x<1024;x++)
		{
			int index = 1024*y+x;
			if(h_state[index] == 1)
			{

			int stateValue = h_state[index];
			int sumValue = h_sum[index];
			printf("index(%d,%d,%d) v:(%d,%d)\n",x,y,index,stateValue,sumValue);
			}
		}
	}
	*/
	/*for(int i =totalNum*10/11.0;i<totalNum;i++)
	{

		int value = h_pos[i];
		int	y	 =  value/1024;
 		int x  = value%1024;
		printf("(%d,%d,pos:%d，index:%d)\n",x,y,h_pos[i],i);
	}*/

	
}
int thrustReduction(int width,int height)
{

	dim3 blockSize(16,16,1);
	dim3 GridSize(width/blockSize.x,height/blockSize.y,1);
	RunFirstPass(blockSize,GridSize,width,height);

	thrust::inclusive_scan(thrustStateVec.begin(), thrustStateVec.end() , thrustSumVec.begin());
	//thrust::host_vector<uint> hSum = thrustSumVec;
	

	
	int totalNum = thrustSumVec[width*height-1]+thrustStateVec[width*height-1];
	
	RunThridPass(blockSize,GridSize);
	//return totalNum;

	return totalNum;
}
extern "C"  void cudaRelateTex(CudaTexResourse * pResouce)
{

	hipArray *tmpcudaArray;
	hipGraphicsResource ** pCudaTex = pResouce->getResPoint(); 
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&tmpcudaArray,*pCudaTex,0,0));
	if(worldPosRef_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_WorldPos_Tex,tmpcudaArray,channelDesc));
		cuda_WorldPos_Tex.filterMode = hipFilterModeLinear;
	}
	else if(worldNormalRef_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_WorlNormal_Tex,tmpcudaArray,channelDesc));
		cuda_WorlNormal_Tex.filterMode = hipFilterModePoint;
	}
	else if(reflecionRef_t ==  pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_Reflect_Tex,tmpcudaArray,channelDesc));
		cuda_Reflect_Tex.filterMode = hipFilterModePoint;
	}
	else if(finalEffect_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_RePro_Pixel,tmpcudaArray,channelDesc));
	}
#ifdef DIFFNORMAL
	else if(diffRef_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_Diff_Tex,tmpcudaArray,channelDesc));
	}
#endif

}
extern "C" void cudaRelateArray(CudaPboResource * pResource)
{
	size_t numBytes;
	hipGraphicsResource ** pCudaTex = pResource->getResPoint(); 
	/*
	moveVector_t,
	lastVector_t,
	*/
	if(float4_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_PBO_Buffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaPboBuffer), &cuda_PBO_Buffer,sizeof(float4*)));
	}
	else if(float2_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_LastVec_Buffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaLastBuffer), &cuda_LastVec_Buffer,sizeof(float2*)));

	}
	else if(test_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_test_Buffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaTestBuffer), &cuda_test_Buffer,sizeof(float4*)));

	}
	else if(diff_normal_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_diffNormBuffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaTestBuffer), &cuda_diffNormBuffer,sizeof(float4*)));

	}
}