#include "hip/hip_runtime.h"
﻿//#include<stdio.h>
#include"hip/hip_runtime.h"
#include "macro.h"
//#include<cudpp.h>
#include <hip/hip_vector_types.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#define UseNine 0
//#include <cuda_gl_interop.h>
//#include <cutil_gl_inline.h>
//#include <helper_cuda_gl.h>
#define UseTrial   1
__constant__  float Cuda_MvpMat[16];
__constant__  float Cuda_CameraViewMat[16]; 
__constant__ float3 d_refCameraPos;
__constant__ float3 d_newCameraPos;
__constant__ float3 d_cameraVec;
__constant__ int Cuda_FramMent;
__constant__ int d_width,d_height;


//__host__ __device__  uint rastWidth = 1024,rastHeight=1024;
#define rastWidth rasterWidth
#define rastHeight rasterHeight
unsigned int  *g_PixelState;//in device address
unsigned int  *g_PixelSums;//in device address
unsigned int  *g_PixelPos; //in device address

hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

texture<float4,2,hipReadModeElementType> cuda_WorldPos_Tex;
texture<float4,2,hipReadModeElementType> cuda_WorlNormal_Tex;
texture<float4,2,hipReadModeElementType> cuda_Reflect_Tex;
texture<float4,2,hipReadModeElementType> cuda_RePro_Pixel;

#ifdef DIFFNORMAL
   texture<float4,2,hipReadModeElementType> cuda_Diff_Tex;
#endif

float4 *cuda_PBO_Buffer;
float2 *cuda_LastVec_Buffer;
float4 *cuda_test_Buffer;
float4 *cuda_diffNormBuffer;
__device__ float4* d_cudaPboBuffer;
__device__ float4* d_cudaTestBuffer;
__device__ float2* d_cudaLastBuffer;
__device__ float4* d_cudaDiffNorlBuffer;
//float4 *cuda_RePos_Buffer;
/*CUDPPHandle theCudpp;
CUDPPHandle scanplan;
*/
thrust::device_vector<uint> thrustSumVec;
thrust::device_vector<uint> thrustStateVec;
thrust::device_vector<uint> thrustPosVec;

__device__ int* d_sum;
__device__ int* d_state;
__device__ int* d_pos;

__device__ float2 toNDC(float4 temp)
{
	temp.x = temp.x / temp.w * 0.5 + 0.5;
	temp.y = temp.y / temp.w * 0.5 + 0.5;
	return make_float2(temp.x,temp.y)* make_float2(rastWidth,rastHeight);
}


__device__ float4 cuda_Mat_Mut3(const float Mat4[16],float3 vec)
{
	float result_F[4];

	for(int i =0;i<4;i++)
		result_F[i] = (Mat4[i] * vec.x + Mat4[i+4] * vec.y +
		Mat4[i+8] * vec.z + Mat4[i+12] );
	float4 result_V = make_float4(result_F[0],result_F[1],result_F[2],result_F[3]);
	return result_V;
}

__device__ float4 Cude_Mul_Mat4(const float Mat4[16],float TmpVec4[4]){

	float result_F[4];
	for(int i =0;i<4;i++)
		result_F[i] = (Mat4[i] * TmpVec4[0] + Mat4[i+4] * TmpVec4[1] +
		Mat4[i+8] * TmpVec4[2] + Mat4[i+12] * TmpVec4[3]);
	float4 result_V = make_float4(result_F[0],result_F[1],result_F[2],result_F[3]);

	return result_V;


}
__device__ float3 transFormToNdc(float3 temp)
{
	float4 ProPos = cuda_Mat_Mut3(Cuda_MvpMat,temp);
	float2 ndcTc=  toNDC(ProPos);
	return make_float3(ndcTc.x,ndcTc.y,ProPos.z/ProPos.w);
}
__device__ float Cuda_Compute_Cos(float2 TmpUV,float3 ReflectPos,int orignId)
{
	float3 WorldPos,WorldNormal;
	float4 Tmpfloat4;
	Tmpfloat4 = tex2D(cuda_WorlNormal_Tex,TmpUV.x,TmpUV.y);
	int NowId = (int)Tmpfloat4.w;
	if(NowId != orignId)
		return -100;
	WorldNormal = make_float3(Tmpfloat4.x,Tmpfloat4.y,Tmpfloat4.z);

	Tmpfloat4 = tex2D(cuda_WorldPos_Tex,TmpUV.x,TmpUV.y);
	WorldPos = make_float3(Tmpfloat4.x,Tmpfloat4.y,Tmpfloat4.z);

	float3 TmpReMirrorPos,TmpCam_Mirror_Normal,TmpCamera_Vec_Normal,TmpInsertPoint;

	TmpReMirrorPos =  abs(dot(WorldPos - ReflectPos,WorldNormal)) * (-2) * WorldNormal + ReflectPos;
	TmpCam_Mirror_Normal = normalize(cross(d_cameraVec,normalize(d_refCameraPos - TmpReMirrorPos)));
	TmpCamera_Vec_Normal = normalize(cross(TmpCam_Mirror_Normal,d_cameraVec));
	TmpInsertPoint = abs(dot(TmpCamera_Vec_Normal,d_refCameraPos - TmpReMirrorPos)/dot(TmpCamera_Vec_Normal,normalize(WorldPos-TmpReMirrorPos))) * normalize(WorldPos-TmpReMirrorPos) + TmpReMirrorPos;
	//TmpDis1 = dot(TmpInsertPoint - d_refCameraPos,d_cameraVec);
	//TmpDis2 = dot(TmpInsertPoint - d_refCameraPos,TmpInsertPoint - d_refCameraPos);
	//return  TmpDis2 - TmpDis1*TmpDis1 ;
	//return  dot(d_cameraVec,TmpInsertPoint - d_refCameraPos);
	return  dot(d_cameraVec,normalize(TmpInsertPoint - d_refCameraPos));
}
enum planeType
{
	originType,
	validSearchType,
	invalidType,
	e_noReflectType,
};
#define PROJECTTHRES 0.02
class Plane
{
public:
	planeType m_type;
	float3 m_worldPos;
	float3 m_worldNormal;
	float m_dis;
	float3 m_reflectPos;
	float3 m_mirrorPos;
	float2 m_tc;
	int m_orginID;

	__device__ Plane()
	{
	}
	__device__ Plane(float2 tc)
	{
		m_tc = tc;
		float4 WorldTmp;
		WorldTmp = tex2D(cuda_WorldPos_Tex,tc.x,tc.y);
		m_worldPos = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		WorldTmp = tex2D(cuda_WorlNormal_Tex,tc.x,tc.y);
		m_orginID = (int)WorldTmp.w;
		m_worldNormal = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		m_dis = tex2D(cuda_Reflect_Tex,tc.x,tc.y).w;
	
	}
	__device__ Plane(float2 tc,int *pIsReflectd)
	{
		m_tc = tc;
		float4 WorldTmp;
		WorldTmp = tex2D(cuda_WorldPos_Tex,tc.x,tc.y);
		m_orginID = (int)WorldTmp.w;
		m_worldPos = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		WorldTmp = tex2D(cuda_WorlNormal_Tex,tc.x,tc.y);
		if(WorldTmp.w <0.01)
		{
			*pIsReflectd = 0;
		}
		m_worldNormal = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		m_dis = tex2D(cuda_Reflect_Tex,tc.x,tc.y).w;
	
	}
	__device__ float2 getTc()
	{
		return m_tc;
	}
	__device__ int getID()
	{
		return m_orginID;
	}
	__device__ int isNoReflect()
	{
		return (m_type== e_noReflectType);
	}
	__device__ int isInValid()
	{
		return (m_type== invalidType);
	}
#define NORMALDIFF 0.001
#define DISDIFF 0.002
	__device__ bool isEqual(Plane& anotherPlane)
	{
		//printf("normal diff:%f, dis diff:%f\n",length(m_worldNormal-anotherPlane.m_worldNormal),dot(m_worldPos-anotherPlane.m_worldPos,m_worldNormal));
		if(length(m_worldNormal-anotherPlane.m_worldNormal)>NORMALDIFF)
		{
			//printf("normal diff,dis:%f,%d\n",length(m_worldNormal-anotherPlane.m_worldNormal),length(m_worldNormal-anotherPlane.m_worldNormal)>NORMALDIFF);
			///printf("(%f,%f,%f)\n",m_worldNormal.x,m_worldNormal.y,m_worldNormal.z);					
			//printf("(%f,%f,%f)\n",anotherPlane.m_worldNormal.x,anotherPlane.m_worldNormal.y,anotherPlane.m_worldNormal.z);
			return false;
		}
		if(dot(m_worldPos-anotherPlane.m_worldPos,m_worldNormal)>DISDIFF)
		{
			//printf("distance too mush:%f\n",dot(m_worldPos-anotherPlane.m_worldPos,m_worldNormal));
			return false;
		}
		return true;
	}
	// 查询是否是投影点 返回1 可以
	__device__ int isAbleFastProj(float2 reProjectTc)
	{
		 
		/*float4 WorldTmp = tex2D(cuda_WorlNormal_Tex,reProjectTc.x,reProjectTc.y);
		int reprjectId  = (int)WorldTmp.w;
		float3 m_worldNormal = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		WorldTmp = tex2D(cuda_WorldPos_Tex,reProjectTc.x,reProjectTc.y);
		float3 worldPos = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		*/
		
		Plane pixelPlane(reProjectTc);
		if(pixelPlane.isInValid())
		{
			//printf("is notVaid");
			return 0;
		}
		if(!isEqual(pixelPlane))
		{
			//printf("is NotEqual");
			return 0;
		}
		return 1;  
		// no need to do futher check; nearest sampling
		pixelPlane.setReflectedPos(*this);
		float DisPoint2Line;
		DisPoint2Line = pixelPlane.getDisToPath();
		if(PROJECTTHRES>DisPoint2Line)
		{
			//printf("within threshold");
			return 1;
		}
		//printf("finally");
		return 0;
	
	}
	
	
	
	__device__ void setPreviousPos( float d)
	{
		m_dis = d;
	
	}
	__device__ float3 getReflectedPos()
	{
		return m_reflectPos;
	}
	__device__ float getMirrorDepth()
	{
		float4 ProPos = cuda_Mat_Mut3(Cuda_MvpMat,m_mirrorPos);
		return ProPos.z/ProPos.w;
	}
	__device__ void setReflectedPos(Plane p)
	{
		m_reflectPos = p.getReflectedPos();
		if(m_orginID == p.getID())
		{
			m_type = validSearchType;
		}		
		else 
		{
			m_type = invalidType;
		}
	}
	__device__ void setReflectedPos()
	{
		float3 inComeDirection = m_worldPos - d_refCameraPos;
		float3 LookVec = normalize(inComeDirection);
		//计算反射光线方向
		float3 ReflectVec = normalize(reflect(LookVec,m_worldNormal));
		m_type = originType;
		m_reflectPos = m_worldPos + ReflectVec * m_dis;	
	}
	__device__ float3 getMirrorPos()
	{
		float3 reflectPos = m_reflectPos;
		m_mirrorPos = abs(dot(m_worldPos - reflectPos,m_worldNormal)) * (-2) * m_worldNormal + reflectPos;
		return m_mirrorPos;
	}
	__device__  float3 beneathPos(float3 sourcePos, float dis)
	{
		float3 inComeDirection = m_worldPos - sourcePos;
		float3 LookVec = normalize(inComeDirection);
		float3 ReflectVec = normalize(reflect(LookVec,m_worldNormal));

		float CosReCorner = dot(ReflectVec,m_worldNormal);
		float3 reflectedPos = m_worldPos + ReflectVec * dis;
		//镜像点的坐标
		float3 ReMirrorPos = dis * CosReCorner * 2 * (-1) * (m_worldNormal) + reflectedPos;

		//相机到反射面的距离
		return ReMirrorPos;
	}
	//得出以该点为平面的反射物体的镜像点
	__device__ float3 intersectPos()
	{
		getMirrorPos();
		float3 VecEye2Ref = normalize(m_mirrorPos - d_newCameraPos);
		return dot( m_worldPos -d_newCameraPos ,m_worldNormal)/ dot(VecEye2Ref,m_worldNormal) * VecEye2Ref + d_newCameraPos;
	}
	
	__device__ float3 intersetVirtualPos()
	{
		float3 ReMirrorPos =  abs(dot(m_worldPos - m_reflectPos,m_worldNormal)) * (-2) * m_worldNormal + m_reflectPos;


		//outPut[index] = make_float4(ReMirrorPos.x,ReMirrorPos.y,ReMirrorPos.z,1.0);
		//return;

		//求出该点与相机连线和平面的新交点
		float DisEye2Plane = abs(dot(d_newCameraPos - m_worldPos,m_worldNormal));
		float CoseTheta2 = abs(dot(normalize(d_newCameraPos - ReMirrorPos),m_worldNormal));
		float3 VecEye2Ref = normalize(ReMirrorPos - d_newCameraPos);
		float3 FinalPos2;
		//	FinalPos2 = d_newCameraPos + VecEye2Ref * (DisEye2Plane/CoseTheta2 );
		FinalPos2 = dot(m_worldPos -d_newCameraPos ,m_worldNormal)/ dot(VecEye2Ref,m_worldNormal) * VecEye2Ref + d_newCameraPos;
		return FinalPos2;
	}
	
	__device__ int checkConverge()
	{
#define CONVERGETHRES 5
		float3 FinalPos  = intersectPos();
		float3 ndcPos =  transFormToNdc(FinalPos);
		float2 ProPosUv =make_float2(ndcPos.x,ndcPos.y);
		//printf("!tc:(%f,%f)\n",m_tc.x,m_tc.y);
		//printf("!proPosUv(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		return length(m_tc-ProPosUv)<CONVERGETHRES;
	}
	__device__ float getSpherDistance()
	{
		float3 inComeDirection = m_worldPos - m_reflectPos;
		float3 LookVec = normalize(inComeDirection);
		//计算反射光线方向
		float3 reflectVec = normalize(reflect(LookVec,m_worldNormal));
		float3 cuttingPoint = m_worldPos+reflectVec*(dot(d_newCameraPos-m_worldPos,reflectVec));
		return length(cuttingPoint-d_newCameraPos);
	}
	__device__ float getDisToPath()
	{
		if(isInValid())
		{
			return 3000.0;
		}
		//求出新的镜像点
		return getSpherDistance();
		float3 ReMirrorPos =  getMirrorPos();
		//printf("!ReMirrorPos: (%f,%f,%f)\n",ReMirrorPos.x,ReMirrorPos.y,ReMirrorPos.z);
		//求相机与镜像点组成的平面的法线
		 
		float3 Cam_Mirror_Normal = normalize(cross(d_cameraVec,normalize(d_refCameraPos - ReMirrorPos)));
		//printf("!Cam_Mirror_Normal: (%f,%f,%f)\n",Cam_Mirror_Normal.x,Cam_Mirror_Normal.y,Cam_Mirror_Normal.z);

		//求相机轨迹所在法线
		float3 Camera_Vec_Normal = normalize(cross(Cam_Mirror_Normal,d_cameraVec));
		//printf("!Camera_Vec_Normal: (%f,%f,%f)\n",Camera_Vec_Normal.x,Camera_Vec_Normal.y,Camera_Vec_Normal.z);

		//分别求三个点与该平面的交点
		float3 InsertPoint = abs(dot(Camera_Vec_Normal,d_refCameraPos - ReMirrorPos)/dot(Camera_Vec_Normal,normalize(m_worldPos-ReMirrorPos))) * normalize(m_worldPos-ReMirrorPos) + ReMirrorPos;
		//printf("!InsertPoint: (%f,%f,%f)\n",InsertPoint.x,InsertPoint.y,InsertPoint.z);

		//求点与直线的距离

		float3 toNewPlace2 = d_newCameraPos-InsertPoint;
		float DisPoint2Line = length(toNewPlace2);
		//DisPoint2Line1  =  dot(normalize(InsertPoint1 - d_refCameraPos),d_cameraVec);

		//printf("!return value: (%f)\n",DisPoint2Line1);

		return DisPoint2Line;


	}
};
/*
__global__ void lineSearchKernel(int width,int height)
{


	//return;



	//return;
	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;

	if( x >width || y> height)
		//if(x !=696 || y !=640)
			//if(x !=373 || y !=139)
				return;
	int index = y * width + x;

	//提取数据
	float3 WorldPos,WorldNormal;
	float ReflectDis;
	float4 WorldTmp;

	WorldTmp = tex2D(cuda_WorldPos_Tex,x+0.5,y+0.5);

	WorldPos = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
	float4 WorldNormalTmp = tex2D(cuda_WorlNormal_Tex,x+0.5,y+0.5);

	int Origin_Id = (int)WorldNormalTmp.w;
	WorldNormal = normalize(make_float3(WorldNormalTmp.x,WorldNormalTmp.y,WorldNormalTmp.z));

	ReflectDis = tex2D(cuda_Reflect_Tex,x,y).w;

	Plane fittingPlane(&WorldPos,&WorldNormal);
	fittingPlane.setPreviousPos(&d_refCameraPos,ReflectDis,&d_newCameraPos);

	float3 ReflectPos =fittingPlane.reflectCamera();
	//printf("origin pos:(%f,%f,%f)\n",ReflectPos.x,ReflectPos.y,ReflectPos.z);
	//printf("0Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);

	float RefelctInCameraDepth;
	RefelctInCameraDepth = ReflectDis /10000.0;

	//RefelctInCameraDepth  = TMpReflecPosInCamera.z / TMpReflecPosInCamera.w;
	//镜像点的坐标
	float3 ReMirrorPos =fittingPlane.mirrorPos();

	//printf("1Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	//相机到反射面的距离
	float DisEye2Plane = abs(dot(d_newCameraPos - WorldPos,WorldNormal));
	//镜像点与相机的向量
	float3 VecEye2Ref = normalize(ReMirrorPos - d_newCameraPos);

	//camera2_pos = new camera position
	//camera1_pos = last camera position
	//镜像点与相机连线和反射平面的交点
	float3 FinalPos ;//= d_newCameraPos + DisEye2Plane/abs(dot(VecEye2Ref,WorldNormal))* VecEye2Ref;

	FinalPos = fittingPlane.intersectPos();

	float4 ProPos = cuda_Mat_Mut3(Cuda_MvpMat,FinalPos);

	//Pro 位置
	float2 ProPosUv;

	float2 TmpUv;

	ProPosUv = toNDC(ProPos)* make_float2(width,height);


	float2 MoveVec = ProPosUv - make_float2(x*1.0+0.5,y*1.0+0.5);
	TmpUv = make_float2(x*1.0,y*1.0);
	int IterTime = 0;

	float LastCosLength = 0;

	float ChengShu = 1.0f;

	float2 TmpMov = make_float2(0,0);

	float2 TheComputVU = make_float2(-1.0f,-1.0f);
	if(MoveVec.x >=0)
		TheComputVU.x =1;
	if(MoveVec.y >=0)
		TheComputVU.y =1;

	float Tmpfloat = Cuda_Compute_Cos(TheComputVU + TmpUv,ReflectPos,Origin_Id);
	if(Tmpfloat>-50 && Tmpfloat < 0)
		ChengShu = -1;

	float LastCoslength;
	//printf("2Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);

	while(IterTime<5 && dot(MoveVec,MoveVec) >=2.5)
	{
		//printf("3Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);

		float2 TheComputVU = make_float2(-1.0f,-1.0f);
		if(MoveVec.x >=0)
			TheComputVU.x =1;
		if(MoveVec.y >=0)
			TheComputVU.y =1;
		//TheComputVU = TheComputVU + TmpUv;


		float2 TmpUv1,TmpUv2,TmpUv3;
		// 		if(ChengShu >0){
		// 		float Tmpfloat = Cuda_Compute_Cos(TheComputVU + TmpUv,ReflectPos,Origin_Id);
		// 		if(Tmpfloat < -50 && Tmpfloat<0 )
		// 			ChengShu = -1;
		// 		else
		// 			ChengShu = 1;
		// 		}

		//int2 TmpUv1,TmpUv2,TmpUv3; 

		TmpUv1 = TmpUv;
		TmpUv2 = TmpUv;
		TmpUv3 = TmpUv;

		//MoveVec = ChengShu * MoveVec;

#if UseNine
		float2 TmpUvs[8];
		int NowSum = 0;
		for(int xi = -1;xi<=1;xi++)
			for(int yi = -1;yi<=1;yi++)
			{
				if((xi != 0 || yi !=0) )//&& length(TmpMov + make_float2(xi,yi)) > 0.5)
					TmpUvs[NowSum++] = TmpUv + make_float2(xi*1.0f,yi*1.0f);
				//TmpUvs[tmpIndex++] =make_float2((x+50)*1.0f,(y+50)*1.0f);
			}
			//outPut[index] = make_float4((TmpUvs[0].x+50)/1024.0,(TmpUvs[7].y+50)/1024.0,Origin_Id,RefelctInCameraDepth);
			//return;

			//取出每个采样点对应的世界坐标
			float3 WorldPosEs[8],WorldNormals[8];
			float4 TmpFloat;
			int ModelIds[8];
			for(int i =0;i<8;i++)
			{
				TmpFloat = tex2D(cuda_WorldPos_Tex,TmpUvs[i].x,TmpUvs[i].y);
				WorldPosEs[i] = make_float3(TmpFloat.x,TmpFloat.y,TmpFloat.z);
				TmpFloat = tex2D(cuda_WorlNormal_Tex,TmpUvs[i].x,TmpUvs[i].y);
				WorldNormals[i] = make_float3(TmpFloat.x,TmpFloat.y,TmpFloat.z);
				ModelIds[i] = (int) TmpFloat.w;

			}
			//outPut[index] = make_float4(TmpUv.x/1024.0,TmpUv.y/1024.0,Origin_Id,RefelctInCameraDepth);


#else
		if(MoveVec.x > 0){
			TmpUv1.x += 1.0;
			TmpUv3.x += 1.0;
		}
		else{
			TmpUv1.x -= 1.0;
			TmpUv3.x -= 1.0;
		}

		if(MoveVec.y >0){
			TmpUv2.y += 1.0;
			TmpUv3.y += 1.0;

		}
		else{
			TmpUv2.y -= 1.0;
			TmpUv3.y -= 1.0;
		}
		//取出每个采样点对应的世界坐标
		float4 TmpFloat4;
		float3 WorldPos1,WorldPos2,WorldPos3;
		float3 WorldNormal1,WorldNormal2,WorldNormal3;
		int Now_ModelId1,Now_ModelId2,Now_ModelId3;

		TmpFloat4 = tex2D(cuda_WorldPos_Tex,TmpUv1.x,TmpUv1.y);
		WorldPos1 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);

		TmpFloat4 = tex2D(cuda_WorldPos_Tex,TmpUv2.x,TmpUv2.y);
		WorldPos2 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);

		TmpFloat4 = tex2D(cuda_WorldPos_Tex,TmpUv3.x,TmpUv3.y);
		WorldPos3 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);

		TmpFloat4 = tex2D(cuda_WorlNormal_Tex,TmpUv1.x,TmpUv1.y);
		WorldNormal1 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);
		Now_ModelId1 = (int)TmpFloat4.w;

		TmpFloat4 = tex2D(cuda_WorlNormal_Tex,TmpUv3.x,TmpUv3.y);
		WorldNormal3 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);
		Now_ModelId3 = (int)TmpFloat4.w;

		TmpFloat4 = tex2D(cuda_WorlNormal_Tex,TmpUv2.x,TmpUv2.y);
		WorldNormal2 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);
		Now_ModelId2 = (int)TmpFloat4.w;

#endif



		//求出新的镜像点
		float DisPoint2Line1,DisPoint2Line2,DisPoint2Line3;
		DisPoint2Line1 = fittingPlane.getDisToPath(WorldPos1,WorldNormal1);
		DisPoint2Line2 = fittingPlane.getDisToPath(WorldPos2,WorldNormal2);
		DisPoint2Line3 = fittingPlane.getDisToPath(WorldPos3,WorldNormal3);

		//
		//printf("DisPoint2Line1:%f\n",DisPoint2Line1);
		float TmpDis;
		if(DisPoint2Line2<DisPoint2Line1)
		{
			TmpUv = TmpUv1;
			TmpUv1 = TmpUv2;
			TmpUv2 = TmpUv;
			WorldPos = WorldPos1;
			WorldPos1 = WorldPos2;
			WorldPos2 = WorldPos;

			TmpDis = DisPoint2Line1;
			DisPoint2Line1 = DisPoint2Line2;
			DisPoint2Line2 = TmpDis;

			TmpDis = Now_ModelId1;
			Now_ModelId1 = Now_ModelId2;
			Now_ModelId2 = TmpDis;

		}

		if(DisPoint2Line3< DisPoint2Line1){
			TmpUv = TmpUv1;
			TmpUv1 = TmpUv3;
			TmpUv3 = TmpUv;
			WorldPos = WorldPos1;
			WorldPos1 = WorldPos3;
			WorldPos3 = WorldPos;

			TmpDis = DisPoint2Line1;
			DisPoint2Line1 = DisPoint2Line2;
			DisPoint2Line2 = TmpDis;

			TmpDis = Now_ModelId1;
			Now_ModelId1 = Now_ModelId2;
			Now_ModelId2 = TmpDis;
		}

		if(DisPoint2Line3< DisPoint2Line2){
			TmpUv = TmpUv2;
			TmpUv2 = TmpUv3;
			TmpUv3 = TmpUv;
			WorldPos = WorldPos2;
			WorldPos2 = WorldPos3;
			WorldPos3 = WorldPos;

			TmpDis = DisPoint2Line1;
			DisPoint2Line1 = DisPoint2Line2;
			DisPoint2Line2 = TmpDis;

			TmpDis = Now_ModelId1;
			Now_ModelId1 = Now_ModelId2;
			Now_ModelId2 = TmpDis;
		}


		//TmpUv = TmpUv1;

		if(Now_ModelId1 == Origin_Id){
			TmpUv = TmpUv1;
			WorldPos = WorldPos1;
		}
		// 		else{
		// 		outPut[index] = make_float4(-10.0f,-10.0f,Origin_Id,RefelctInCameraDepth);
		// 			return;
		// 		}

		else if(Now_ModelId2 == Origin_Id){
			TmpUv = TmpUv2;
			WorldPos = WorldPos2;
		}
		else if(Now_ModelId3 == Origin_Id){
			TmpUv = TmpUv3;
			WorldPos = WorldPos3;
		}
		else{
			d_cudaPboBuffer[index] = make_float4(-10.0f,-10.0f,Origin_Id,RefelctInCameraDepth);
			return;
		}
		TmpUv = TmpUv+TheComputVU;

		//LastCoslength = Cuda_Compute_Cos(TmpUv,ReflectPos,Origin_Id);
		//LastCoslength = 0;
		//if(LastCoslength > length(Cuda_CameraPos1[1] - Cuda_CameraPos2[1])-0.5)
		//break;



		//得出该点法线

		float4 TmpFloat41= tex2D(cuda_WorlNormal_Tex,TmpUv.x,TmpUv.y);
		WorldNormal  = make_float3(TmpFloat41.x,TmpFloat41.y,TmpFloat41.z);


		float NowModelId = TmpFloat41.w;
		fittingPlane = Plane(&WorldPos,&WorldNormal);



		float3 FinalPos2 =fittingPlane.intersetPos2();
		//printf("!WorldNormal: (%f,%f,%f)\n",WorldNormal.x,WorldNormal.y,WorldNormal.z);
		//printf("!WorldPos: (%f,%f,%f)\n",WorldPos.x,WorldPos.y,WorldPos.z);
		//printf("!FinalPos2: (%f,%f,%f)\n",FinalPos2.x,FinalPos2.y,FinalPos2.z);

		float4 ProPos2;
		ProPos2 = cuda_Mat_Mut3(Cuda_MvpMat,FinalPos2);
		ProPosUv = toNDC(ProPos2)* make_float2(width,height);


		MoveVec = ProPosUv - TmpUv;
		//printf("!ProPosUv: (%f,%f)\n",ProPosUv.x,ProPosUv.y);
		//printf("!TmpUv: (%f,%f)\n",TmpUv.x,TmpUv.y);
		//printf("!moveVec:(%f,%f)\n",MoveVec.x,MoveVec.y);
		IterTime++;

	}

	//if(length(TmpUv - make_float2(x,y))> 30){
	//outPut[index] = make_float4(-10.0,-10.0,Origin_Id,RefelctInCameraDepth);
	//return;

	//}

	//if(dot(MoveVec,MoveVec) >=20){
	//outPut[index] = make_float4(-10.0f,-10.0f,Origin_Id,RefelctInCameraDepth);
	//return;

	//}


	//outPut[index] = ProPos;
	//if(TmpUv.x>=0 && TmpUv.x<width && TmpUv.y>=0 && TmpUv.y <= height)
	//	outPut[(int)(TmpUv.y * width + TmpUv.x)] = tex2D(cuda_Reflect_Tex,x+0.5y+0.5);
	//WorlPosInCameraDepth = 10;
	//if(dot(MoveVec,MoveVec)>200)
	//WorlPosInCameraDepth = -10;
	//outPut[index] = make_float4((TmpUv.x - x)/1024.0,(TmpUv.y - y)/1024.0,Origin_Id,RefelctInCameraDepth);
	d_cudaPboBuffer[index] = make_float4(TmpUv.x/(float)width,TmpUv.y/(float)height,Origin_Id,RefelctInCameraDepth);

}*/

__device__ int isInSamePixel(float2 moveVec)
{
	if(abs(moveVec.x)<0.5&&abs(moveVec.y)<0.5)
		return 1;
	else
		return 0;
}
__device__ int nextStep(float2 moveVec,float2* pStepVec)
{
#define PI 3.141592654f
	if(length(moveVec)<0.717)
		return 0;
	if(moveVec.x==0)
	{
		*pStepVec = make_float2(0,copysignf(1,moveVec.y));
		return 1;
	}
	float absTangleValue = abs(moveVec.y/moveVec.x);
	if(absTangleValue<tanf(PI/4/2))
	{
		
		*pStepVec = make_float2(copysignf(1,moveVec.x),0);

	}
	else if (absTangleValue>tanf(PI*3/4/2))
	{
		
		*pStepVec = make_float2(0,copysignf(1,moveVec.y));
	}
	else 
	{
		*pStepVec = make_float2(copysignf(1,moveVec.x),copysignf(1,moveVec.y));
	}
	return 1;
}
__device__ int nextThreeStep(float2 moveVec,float2* candicate1,float2* candicate2,float2* candicate3,float2 currentUv)
{
	#define PI 3.141592654f
/*	if(length(moveVec)<0.717)
	{

		return 0;
	}*/
	if(moveVec.x==0)
	{
		*candicate1 = make_float2(0,copysignf(1,moveVec.y));
		*candicate2 = make_float2(-1,copysignf(1,moveVec.y));
		*candicate3 = make_float2(1,copysignf(1,moveVec.y));
		*candicate1+=currentUv;
		*candicate2+=currentUv;
		*candicate3+=currentUv;
		return 1;
	}
	float absTangleValue = abs(moveVec.y/moveVec.x);
	if(absTangleValue<tanf(PI/4/2))
	{
		
		*candicate1 = make_float2(copysignf(1,moveVec.x),0);
		*candicate2 = make_float2(copysignf(1,moveVec.x),-1);
		*candicate3 = make_float2(copysignf(1,moveVec.x),1);

	}
	else if (absTangleValue>tanf(PI*3/4/2))
	{
		
		*candicate1 = make_float2(0,copysignf(1,moveVec.y));
		*candicate2 = make_float2(-1,copysignf(1,moveVec.y));
		*candicate3 = make_float2(1,copysignf(1,moveVec.y));
	}
	else 
	{
		*candicate1 = make_float2(copysignf(1,moveVec.x),copysignf(1,moveVec.y));
		*candicate2 = make_float2(copysignf(1,moveVec.x),0);
		*candicate3 = make_float2(0,copysignf(1,moveVec.y));
	}
	*candicate1+=currentUv;
	*candicate2+=currentUv;
	*candicate3+=currentUv;
	return 1;
}
#define CONVERGE 5
#define OUTRANGE 2
#define OUTOBJECT 3
#define FASTPROJT 4
__device__ int Floor2Int(float f)
{
	return (int)f+0.5;
}
__device__ int isOutOfRange(float2 tc)
{
	return tc.x>rasterWidth-0.5||tc.y>rasterHeight-0.5||tc.x<0||tc.y<0;
}
__device__ int threePointSearch(float2 currentPlace,float2* moveToVec)
{

#define STEPNUMBER 25
	int x =Floor2Int(currentPlace.x-0.5);
	int y =Floor2Int(currentPlace.y-0.5);
	int index = y * rasterWidth + x;
	float2 currentUv = make_float2(currentPlace.x,currentPlace.y);
	
	int isReflectedPixel = 1;
	Plane fittingPlane(currentUv,&isReflectedPixel);
	if(0==isReflectedPixel)
	{
		d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,1,OUTRANGE);
		return;
	}
	if(x!=482||y!=585)  //convave point
	  return;
	
	//printf("1Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	//�����������ľ���
	//= d_newCameraPos + DisEye2Plane/abs(dot(VecEye2Ref,WorldNormal))* VecEye2Ref;
	fittingPlane.setReflectedPos();
	float3 FinalPos  = fittingPlane.intersectPos();
	float3 ndcPos =  transFormToNdc(FinalPos);
	float2 ProPosUv =make_float2(ndcPos.x,ndcPos.y);
	float rejectDepth = fittingPlane.getMirrorDepth();
	float2 MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
	float formerDis = fittingPlane.getDisToPath();

	
	float3 worldPos = fittingPlane.m_worldPos;
	float3 worldNormal = fittingPlane.m_worldNormal;
	
	
	printf("three x,y:%d,%d\n",x,y);
	printf("pos:(%f,%f,%f)\n",worldPos.x,worldPos.y,worldPos.z);
	printf("normal:(%f,%f,%f)\n",worldNormal.x,worldNormal.y,worldNormal.z);
	printf("reflected pos:(%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	printf("intersect pos:(%f,%f,%f)\n",FinalPos.x,FinalPos.y,FinalPos.z);
	printf("refCamera:(%f,%f,%f)\n",d_refCameraPos.x,d_refCameraPos.y,d_refCameraPos.z);
	printf("newCameraPos:(%f,%f,%f)\n",d_newCameraPos.x,d_newCameraPos.y,d_newCameraPos.z);
	printf("project Pos:(%f,%f),depth:%f\n",ProPosUv.x,ProPosUv.y,rejectDepth);
	
	if(fittingPlane.isAbleFastProj(ProPosUv))
	{
		if(fabs(ProPosUv.x-currentUv.x)<0.5&&(fabs(ProPosUv.y-currentUv.y)<0.5))
		{
			d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
			return CONVERGE;
		}
		d_cudaPboBuffer[index] =   make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,FASTPROJT);
		return FASTPROJT;
	}
	int IterTime = 0;
	while(IterTime<STEPNUMBER/1024.0*rasterWidth)
	{
		//printf("Item:%d currentUv:(%f,%f)\n",IterTime,currentUv.x,currentUv.y);
		
		float2 TmpUv1,TmpUv2,TmpUv3,TmpUv;
		/*
		TmpUv1 = currentUv;
		TmpUv2 = currentUv;
		TmpUv3 = currentUv;

		//MoveVec = ChengShu * MoveVec;


		if(MoveVec.x > 0)
		{
			TmpUv1.x += 1.0;
			TmpUv3.x += 1.0;
		}
		else{
			TmpUv1.x -= 1.0;
			TmpUv3.x -= 1.0;
		}

		if(MoveVec.y >0){
			TmpUv2.y += 1.0;
			TmpUv3.y += 1.0;

		}
		else{
			TmpUv2.y -= 1.0;
			TmpUv3.y -= 1.0;
		}
		*/
		//取出每个采样点对应的世界坐标
		nextThreeStep(MoveVec,&TmpUv1,&TmpUv2,&TmpUv3,currentUv);
			
		if(isOutOfRange(TmpUv1)||isOutOfRange(TmpUv2)||isOutOfRange(TmpUv3))
		{
			//printf("(%f,%f),(%f,%f)，(%f,%f)\n",TmpUv1.x,TmpUv1.y,TmpUv2.x,TmpUv2.y,TmpUv3.x,TmpUv3.y);
			// printf("@out of range\n");
	
			d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTOBJECT);
		  
			return OUTOBJECT;
		}
		Plane pixelPlane1(TmpUv1),pixelPlane2(TmpUv2),pixelPlane3(TmpUv3);
		pixelPlane1.setReflectedPos(fittingPlane);
		pixelPlane2.setReflectedPos(fittingPlane);
		pixelPlane3.setReflectedPos(fittingPlane);

		float DisPoint2Line1,DisPoint2Line2,DisPoint2Line3;
		DisPoint2Line1 = pixelPlane1.getDisToPath();
		DisPoint2Line2 = pixelPlane2.getDisToPath();
		DisPoint2Line3 = pixelPlane3.getDisToPath();
		
	/*	for(int dx = -1;dx<=1;dx++)
		{
			for(int dy = -1;dy<=1;dy++)
			{
				float2 uv = currentUv+make_float2(dx,dy);
				Plane testPlan(uv);
				testPlan.setReflectedPos(fittingPlane);
				float dis = testPlan.getDisToPath();
				//printf("point(%f,%f,%f)\n",uv.x,uv.y,dis);

			}
		}*/
		//printf("currentProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		//printf("currentMove(%f,%f)\n",MoveVec.x,MoveVec.y);
		printf("point1(%f,%f,%f),point2(%f,%f,%f),point3(%f,%f,%f)\n",TmpUv1.x,TmpUv1.y,DisPoint2Line1,TmpUv2.x,TmpUv2.y,DisPoint2Line2,TmpUv3.x,TmpUv3.y,DisPoint2Line3);
		
		Plane minPlane;
		float minDis;
		//
		//printf("DisPoint2Line1:%f\n",DisPoint2Line1);
		if(pixelPlane1.isInValid()&&pixelPlane2.isInValid()&&pixelPlane2.isInValid())
		{	
			//printf("invalid\n");
			
			d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTOBJECT);
	    	  printf("@x:%d y:%d: %f,%f,%f,%f\n",x,y,d_cudaPboBuffer[index] .x,d_cudaPboBuffer[index] .y,d_cudaPboBuffer[index] .z,d_cudaPboBuffer[index] .w);
	
			return OUTOBJECT;
		}
		if(DisPoint2Line2<DisPoint2Line1)
		{
			if(DisPoint2Line3< DisPoint2Line2)
			{
				minPlane = pixelPlane3;
				minDis = DisPoint2Line3;
				// min = 3
			}	
			else
			{
				minPlane = pixelPlane2;
				minDis = DisPoint2Line2;
				//min  = 2
			}
		}
		else
		{
			if(DisPoint2Line3< DisPoint2Line1)
			{
				minPlane = pixelPlane3;
				minDis = DisPoint2Line3;
				// min = 3
			}	
			else
			{
				minPlane = pixelPlane1;		
				minDis = DisPoint2Line1;
				//min  = 1
			}
		}
		printf("result uv(%f,%f) nextuv(%f,%f),dis:%f,formerDis:%f\n",currentUv.x,currentUv.y, minPlane.getTc().x, minPlane.getTc().y,minDis,formerDis);
		if(minDis>formerDis)
		{
			*moveToVec = currentUv;
			if(length(MoveVec)<15/1024.0*rasterWidth&&formerDis<5)
			{
				d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
			//	printf("Converge\n");
				return CONVERGE;
			}
			else
			{
				d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,-0.1,OUTOBJECT);
			//	printf("not right minDis\n");	
				return OUTOBJECT;
			}
		}
		//TmpUv = TmpUv1;
		
		currentUv = minPlane.getTc();
		formerDis = minDis;
		fittingPlane = minPlane;

		FinalPos  = fittingPlane.intersectPos();
		ndcPos =  transFormToNdc(FinalPos);
	    ProPosUv =make_float2(ndcPos.x,ndcPos.y);
		rejectDepth = fittingPlane.getMirrorDepth();
		MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
		
		
		printf("ProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		printf("Move(%f,%f)\n",MoveVec.x,MoveVec.y);
		printf("worldPos(%f,%f,%f),normal:(%f,%f,%f)\n",fittingPlane.m_worldPos.x,fittingPlane.m_worldPos.y,fittingPlane.m_worldPos.z,fittingPlane.m_worldNormal.x,fittingPlane.m_worldNormal.y,fittingPlane.m_worldNormal.z);
		
		IterTime++;
	}
	d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTRANGE);
				
	printf("！x:%d y:%d: %f,%f,%f,%f\n",x,y,d_cudaPboBuffer[index] .x,d_cudaPboBuffer[index] .y,d_cudaPboBuffer[index] .z,d_cudaPboBuffer[index] .w);
	
	
	return OUTRANGE;
	
}
__device__ bool isInBox(float2 centerUv,float2 projectUv)
{
	if(fabs(centerUv.x-projectUv.x)<0.5&&fabs(centerUv.y-projectUv.y)<0.5)
		return true;
	else 
		return false;
}

__device__ int ninePointSearch(float2 currentPlace,float2* moveToVec)
{
#define STEPNUMBER 25
	int x =Floor2Int(currentPlace.x-0.5);
	int y =Floor2Int(currentPlace.y-0.5);
	int index = y * rasterWidth + x;
	float2 currentUv = make_float2(currentPlace.x,currentPlace.y);

	int isReflectedPixel = 1;
	Plane fittingPlane(currentUv,&isReflectedPixel);
	if(0==isReflectedPixel)
	{
		d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,1,OUTRANGE);
		return;
	}
	
	//if(x!=482||y!=585)
	//  return;
	
	//printf("1Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	//�����������ľ���
	//= d_newCameraPos + DisEye2Plane/abs(dot(VecEye2Ref,WorldNormal))* VecEye2Ref;
	fittingPlane.setReflectedPos();
	float3 FinalPos  = fittingPlane.intersectPos();
	float3 ndcPos =  transFormToNdc(FinalPos);
	float2 ProPosUv =make_float2(ndcPos.x,ndcPos.y);
	float rejectDepth = fittingPlane.getMirrorDepth();
	float2 MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
	float formerDis = fittingPlane.getDisToPath();

	
	float3 worldPos = fittingPlane.m_worldPos;
	float3 worldNormal = fittingPlane.m_worldNormal;
	
	
	/*printf("nine search: x,y:%d,%d\n",x,y);
	printf("pos:(%f,%f,%f)\n",worldPos.x,worldPos.y,worldPos.z);
	printf("normal:(%f,%f,%f)\n",worldNormal.x,worldNormal.y,worldNormal.z);
	printf("reflected pos:(%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	printf("intersect pos:(%f,%f,%f)\n",FinalPos.x,FinalPos.y,FinalPos.z);
	printf("refCamera:(%f,%f,%f)\n",d_refCameraPos.x,d_refCameraPos.y,d_refCameraPos.z);
	printf("newCameraPos:(%f,%f,%f)\n",d_newCameraPos.x,d_newCameraPos.y,d_newCameraPos.z);
	printf("project Pos:(%f,%f),depth:%f\n",ProPosUv.x,ProPosUv.y,rejectDepth);
	*/
	if(fittingPlane.isAbleFastProj(ProPosUv))
	{
		//printf("fitting\n");
		if(fabs(ProPosUv.x-currentUv.x)<0.5&&(fabs(ProPosUv.y-currentUv.y)<0.5))
		{
			d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
			return CONVERGE;
		}
				
		d_cudaPboBuffer[index] =   make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,FASTPROJT);
		return FASTPROJT;
	}
	int IterTime = 0;
	while(IterTime<STEPNUMBER/1024.0*rasterWidth)
	{
		float minDis = 3000;
		float2 minUv;
		Plane minPlane;
		for(int dx = -1;dx<=1;dx++)
		{
			for(int dy = -1;dy<=1;dy++)
			{
				float2 uv = currentUv+make_float2(dx,dy);
				if(dx==0&&dy==0)
					continue;
				if(isOutOfRange(uv))
					continue;
				Plane testPlan(uv);
				testPlan.setReflectedPos(fittingPlane);
				
				if(testPlan.isInValid())
					continue;
				float dis = testPlan.getDisToPath();
				//printf("point1(%f,%f,%f)\n",uv.x,uv.y,dis);
				if(minDis>dis)
				{
					//printf("min\n");
					minDis = dis;
					minUv = uv;
					minPlane = testPlan;
				}

			}
		}
			//
		//printf("current uv(%f,%f) nextuv(%f,%f),dis:%f,formerDis:%f\n",currentUv.x,currentUv.y, minPlane.getTc().x, minPlane.getTc().y,minDis,formerDis);
		if(minDis == 3000)
		{
			d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,-0.1,OUTOBJECT);
			return OUTOBJECT;
		}
		if(minDis>formerDis)
		{
			*moveToVec = currentUv;
			if(length(MoveVec)<15/1024.0*rasterWidth&&minDis<5)
			{
			//	printf("convenge result uv(%f,%f) nextuv(%f,%f),dis:%f,formerDis:%f\n",currentUv.x,currentUv.y, minPlane.getTc().x, minPlane.getTc().y,minDis,formerDis);
				if(isInBox(currentUv,ProPosUv))
				{
					d_cudaPboBuffer[index] =  make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
					return CONVERGE;
				}
				else{
					d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
					return CONVERGE;
				}
			}
			else
			{
			//	printf("out id\n");
				d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,-0.1,OUTOBJECT);
				return OUTOBJECT;
			}
		}
		
		currentUv = minPlane.getTc();
		formerDis = minDis;
		fittingPlane = minPlane;

		FinalPos  = fittingPlane.intersectPos();
		ndcPos =  transFormToNdc(FinalPos);
	    ProPosUv =make_float2(ndcPos.x,ndcPos.y);
		rejectDepth = fittingPlane.getMirrorDepth();
		MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
		/*
		printf("ProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		printf("Move(%f,%f)\n",MoveVec.x,MoveVec.y);
		printf("worldPos(%f,%f,%f),normal:(%f,%f,%f)\n",fittingPlane.m_worldPos.x,fittingPlane.m_worldPos.y,fittingPlane.m_worldPos.z,fittingPlane.m_worldNormal.x,fittingPlane.m_worldNormal.y,fittingPlane.m_worldNormal.z);
		*/
		IterTime++;
	}
	//printf("out range\n");
			
	d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTRANGE);		
	return OUTRANGE;
	
}

__global__ void MyNewKernel(int width,int height)
{
	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;
	if( x >width || y> height)
			return;
	//if(x!=91||y!=623)
	//	return;
	
	//printf("x:%d,y:%d\n",x,y);
	/*if(x==1024)
	{
		printf("x:%d,y:%d\n",x,y);
	}*/
	int index = y * width + x;
	float2 currentUv = make_float2(x+0.5,y+0.5);
	
	//d_cudaPboBuffer[index] =   make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,-0.1,FASTPROJT);
	//return;		
	float2 resultValue;
	ninePointSearch(currentUv,&resultValue);
	

}
extern "C" void MappingContantPeremeter(int* w,int* h)
{


	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_width),w,sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_height),h,sizeof(int)));


}
extern "C" void TransConstData(const float *MvpMat,const float *ViewMat,float3 *pRefCamera,float3 *pNewCameraPos)
{

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_MvpMat),MvpMat,sizeof(float)*16,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_CameraViewMat),ViewMat,sizeof(float)*16,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_refCameraPos),pRefCamera,sizeof(float3),0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_newCameraPos),pNewCameraPos,sizeof(float3),0,hipMemcpyHostToDevice));

	float3 CameraVec = normalize(*pNewCameraPos - *pRefCamera);
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cameraVec),&CameraVec,sizeof(float3),0,hipMemcpyHostToDevice));

}


__device__ bool colorEqual(float3 soourceColor, float3 targetColor)
{
	return length(soourceColor-targetColor)<0.1;
}
__global__ void MyFirstPassKernel(int width,int height,uint *PixelState,float4 * cuda_PBO_Buffer)
{

	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;
	if(x > width || y > height)
		//if(x!=224||y!=267)
			return;

	//if(x !=224 || y !=267)
	//	if(x !=117 || y !=712)
	//	return;

	int index = y * width + x;

	//printf("$%d,%d\n",x,y);

	float4 StateValue =tex2D(cuda_RePro_Pixel,x+0.5,y+0.5);
	float3 reflectColor = make_float3(StateValue.x,StateValue.y,StateValue.z);

	if(colorEqual(reflectColor,make_float3(1,0,0))||colorEqual(reflectColor,make_float3(0,1,0))||reflectColor.x>1.01||reflectColor.y>1.01)  // fetch the color in additional tex
	{
		d_state[index] = 1;
	}
	else
	{
		//printf("ddddddd\n");
		d_state[index] = 0;
	}

}
extern "C" void cudaPredict(int width,int height)
{
	hipEvent_t begin_t,end_t;
	checkCudaErrors( hipEventCreate(&begin_t) );
    checkCudaErrors( hipEventCreate(&end_t)  );

	hipEventRecord(begin_t,0);
	dim3 blockSize(16,16,1);
	dim3 gridSize(width/blockSize.x,height/blockSize.y,1);
	MyNewKernel<<<gridSize,blockSize>>>(width,height);
	hipEventRecord(end_t,0);
	hipEventSynchronize(end_t); 
	float costtime;
	checkCudaErrors( hipEventElapsedTime(&costtime, begin_t, end_t) );

	printf("cuda measura time:%f\n",costtime);
	checkCudaErrors( hipEventDestroy(begin_t) );
    checkCudaErrors( hipEventDestroy(end_t)  );
}

void mapThustResourse()
{
	thrust::device_ptr<uint> dp = &thrustStateVec[0];
	uint * raw_ptr = thrust :: raw_pointer_cast ( dp );  
}
__global__ void MyThridPass(uint *PixePos,uint *PixelState,uint *PixelSums);

extern "C" void RunFirstPass(dim3 BlockSize,dim3 GridSize,int width,int height)
{
	thrust::device_ptr<uint> dp = &thrustStateVec[0];
	uint * raw_ptr = thrust :: raw_pointer_cast ( dp );  
	MyFirstPassKernel<<<GridSize,BlockSize>>>(width,height,raw_ptr,cuda_PBO_Buffer);
	


}


__global__ void MyThridPass(uint *PixePos,uint *PixelState,uint *PixelSums)
{

	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;

	if(x > rastWidth || y > rastHeight)
		return;
	uint index = y * rasterWidth + x;
	if(PixelState[index] == 1)
	{
		uint Pos1 = PixelSums[index];
		PixePos[Pos1] = index;

	}
}
void InitThrust()
{	
	thrustSumVec = thrust::device_vector<uint>(rastWidth*rastHeight); 
	thrustStateVec = thrust::device_vector<uint>(rastWidth*rastHeight);
	thrustPosVec = thrust::device_vector<uint>(rastWidth*rastHeight);

	thrust::device_ptr<uint> d_pstate = &thrustStateVec[0];
	g_PixelState = thrust :: raw_pointer_cast ( d_pstate );  
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_state), &g_PixelState,sizeof(int*)));

	thrust::device_ptr<uint> d_psum = &thrustSumVec[0];
	g_PixelSums = thrust :: raw_pointer_cast ( d_psum );
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_sum), &g_PixelSums,sizeof(int*)));

	thrust::device_ptr<uint> dp = &thrustPosVec[0];
	g_PixelPos= thrust :: raw_pointer_cast ( dp );
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_pos), &g_PixelPos,sizeof(int*)));

}



extern "C" void RunThridPass(dim3 BlockSize ,dim3 GridSize)
{	  	
	thrust::device_ptr<uint> dp = &thrustPosVec[0];
	uint * raw_ptr_pos = thrust :: raw_pointer_cast ( dp );  

	thrust::device_ptr<uint> d_pstate = &thrustStateVec[0];
	uint * raw_ptr_state = thrust :: raw_pointer_cast ( d_pstate );  

	thrust::device_ptr<uint> d_psum = &thrustSumVec[0];
	uint * raw_ptr_sum = thrust :: raw_pointer_cast ( d_psum );
	MyThridPass<<<GridSize,BlockSize>>>(raw_ptr_pos,raw_ptr_state,raw_ptr_sum);

	//uint* h_state = thrust::raw_pointer_cast(d_pstate);
	//thrust::host_vector <uint> h_pos =thrustSumVec;
	//thrust::host_vector <uint> h_pos =thrustPosVec;
	/* 
	for(int y = 0;y<1024;y++)
	{
		for(int x = 0;x<1024;x++)
		{
			int index = 1024*y+x;
			if(h_state[index] == 1)
			{

			int stateValue = h_state[index];
			int sumValue = h_sum[index];
			printf("index(%d,%d,%d) v:(%d,%d)\n",x,y,index,stateValue,sumValue);
			}
		}
	}
	*/
	/*for(int i =totalNum*10/11.0;i<totalNum;i++)
	{

		int value = h_pos[i];
		int	y	 =  value/1024;
 		int x  = value%1024;
		printf("(%d,%d,pos:%d，index:%d)\n",x,y,h_pos[i],i);
	}*/

	
}
int thrustReduction(int width,int height)
{

	dim3 blockSize(16,16,1);
	dim3 GridSize(width/blockSize.x,height/blockSize.y,1);
	RunFirstPass(blockSize,GridSize,width,height);

	thrust::inclusive_scan(thrustStateVec.begin(), thrustStateVec.end() , thrustSumVec.begin());
	//thrust::host_vector<uint> hSum = thrustSumVec;
	

	
	int totalNum = thrustSumVec[width*height-1]+thrustStateVec[width*height-1];
	
	RunThridPass(blockSize,GridSize);
	//return totalNum;

	return totalNum;
}
extern "C"  void cudaRelateTex(CudaTexResourse * pResouce)
{

	hipArray *tmpcudaArray;
	hipGraphicsResource ** pCudaTex = pResouce->getResPoint(); 
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&tmpcudaArray,*pCudaTex,0,0));
	if(worldPosRef_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_WorldPos_Tex,tmpcudaArray,channelDesc));
		cuda_WorldPos_Tex.filterMode = hipFilterModeLinear;
	}
	else if(worldNormalRef_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_WorlNormal_Tex,tmpcudaArray,channelDesc));
		cuda_WorlNormal_Tex.filterMode = hipFilterModePoint;
	}
	else if(reflecionRef_t ==  pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_Reflect_Tex,tmpcudaArray,channelDesc));
		cuda_Reflect_Tex.filterMode = hipFilterModePoint;
	}
	else if(finalEffect_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_RePro_Pixel,tmpcudaArray,channelDesc));
	}
#ifdef DIFFNORMAL
	else if(diffRef_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_Diff_Tex,tmpcudaArray,channelDesc));
	}
#endif

}
extern "C" void cudaRelateArray(CudaPboResource * pResource)
{
	size_t numBytes;
	hipGraphicsResource ** pCudaTex = pResource->getResPoint(); 
	/*
	moveVector_t,
	lastVector_t,
	*/
	if(float4_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_PBO_Buffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaPboBuffer), &cuda_PBO_Buffer,sizeof(float4*)));
	}
	else if(float2_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_LastVec_Buffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaLastBuffer), &cuda_LastVec_Buffer,sizeof(float2*)));

	}
	else if(test_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_test_Buffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaTestBuffer), &cuda_test_Buffer,sizeof(float4*)));

	}
	else if(diff_normal_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_diffNormBuffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaTestBuffer), &cuda_diffNormBuffer,sizeof(float4*)));

	}
}