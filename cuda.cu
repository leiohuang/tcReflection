#include "hip/hip_runtime.h"
﻿//#include<stdio.h>
#include"hip/hip_runtime.h"
#include "macro.h"
//#include<cudpp.h>
#include <hip/hip_vector_types.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#define UseNine 0
//#include <cuda_gl_interop.h>
//#include <cutil_gl_inline.h>
//#include <helper_cuda_gl.h>
#define UseTrial   1
__constant__  float Cuda_MvpMat[16];
__constant__  float Cuda_CameraViewMat[16]; 
__constant__ float3 d_refCameraPos;
__constant__ float3 d_newCameraPos;
__constant__ float3 d_cameraVec;
__constant__ int Cuda_FramMent;
__constant__ int d_width,d_height;


//__host__ __device__  uint rastWidth = 1024,rastHeight=1024;
#define rastWidth rasterWidth
#define rastHeight rasterHeight
unsigned int  *g_PixelState;//in device address
unsigned int  *g_PixelSums;//in device address
unsigned int  *g_PixelPos; //in device address

hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

texture<float4,2,hipReadModeElementType> cuda_WorldPos_Tex;
texture<float4,2,hipReadModeElementType> cuda_WorlNormal_Tex;
texture<float4,2,hipReadModeElementType> cuda_Reflect_Tex;
texture<float4,2,hipReadModeElementType> cuda_RePro_Pixel;

float4 *cuda_PBO_Buffer;
float2 *cuda_LastVec_Buffer;
float4 *cuda_test_Buffer;
float4 *cuda_diffNormBuffer;
__device__ float4* d_cudaPboBuffer;
__device__ float4* d_cudaTestBuffer;
__device__ float2* d_cudaLastBuffer;
__device__ float4* d_cudaDiffNorlBuffer;
//float4 *cuda_RePos_Buffer;
/*CUDPPHandle theCudpp;
CUDPPHandle scanplan;
*/
thrust::device_vector<uint> thrustSumVec;
thrust::device_vector<uint> thrustStateVec;
thrust::device_vector<uint> thrustPosVec;

__device__ int* d_sum;
__device__ int* d_state;
__device__ int* d_pos;

__device__ float2 toNDC(float4 temp)
{
	temp.x = temp.x / temp.w * 0.5 + 0.5;
	temp.y = temp.y / temp.w * 0.5 + 0.5;
	return make_float2(temp.x,temp.y)* make_float2(rastWidth,rastHeight);
}


__device__ float4 cuda_Mat_Mut3(const float Mat4[16],float3 vec)
{
	float result_F[4];

	for(int i =0;i<4;i++)
		result_F[i] = (Mat4[i] * vec.x + Mat4[i+4] * vec.y +
		Mat4[i+8] * vec.z + Mat4[i+12] );
	float4 result_V = make_float4(result_F[0],result_F[1],result_F[2],result_F[3]);
	return result_V;
}

__device__ float4 Cude_Mul_Mat4(const float Mat4[16],float TmpVec4[4]){

	float result_F[4];
	for(int i =0;i<4;i++)
		result_F[i] = (Mat4[i] * TmpVec4[0] + Mat4[i+4] * TmpVec4[1] +
		Mat4[i+8] * TmpVec4[2] + Mat4[i+12] * TmpVec4[3]);
	float4 result_V = make_float4(result_F[0],result_F[1],result_F[2],result_F[3]);

	return result_V;


}
__device__ float3 transFormToNdc(float3 temp)
{
	float4 ProPos = cuda_Mat_Mut3(Cuda_MvpMat,temp);
	float2 ndcTc=  toNDC(ProPos);
	return make_float3(ndcTc.x,ndcTc.y,ProPos.z/ProPos.w);
}
__device__ float Cuda_Compute_Cos(float2 TmpUV,float3 ReflectPos,int orignId)
{
	float3 WorldPos,WorldNormal;
	float4 Tmpfloat4;
	Tmpfloat4 = tex2D(cuda_WorlNormal_Tex,TmpUV.x,TmpUV.y);
	int NowId = (int)Tmpfloat4.w;
	if(NowId != orignId)
		return -100;
	WorldNormal = make_float3(Tmpfloat4.x,Tmpfloat4.y,Tmpfloat4.z);

	Tmpfloat4 = tex2D(cuda_WorldPos_Tex,TmpUV.x,TmpUV.y);
	WorldPos = make_float3(Tmpfloat4.x,Tmpfloat4.y,Tmpfloat4.z);

	float3 TmpReMirrorPos,TmpCam_Mirror_Normal,TmpCamera_Vec_Normal,TmpInsertPoint;

	TmpReMirrorPos =  abs(dot(WorldPos - ReflectPos,WorldNormal)) * (-2) * WorldNormal + ReflectPos;
	TmpCam_Mirror_Normal = normalize(cross(d_cameraVec,normalize(d_refCameraPos - TmpReMirrorPos)));
	TmpCamera_Vec_Normal = normalize(cross(TmpCam_Mirror_Normal,d_cameraVec));
	TmpInsertPoint = abs(dot(TmpCamera_Vec_Normal,d_refCameraPos - TmpReMirrorPos)/dot(TmpCamera_Vec_Normal,normalize(WorldPos-TmpReMirrorPos))) * normalize(WorldPos-TmpReMirrorPos) + TmpReMirrorPos;
	//TmpDis1 = dot(TmpInsertPoint - d_refCameraPos,d_cameraVec);
	//TmpDis2 = dot(TmpInsertPoint - d_refCameraPos,TmpInsertPoint - d_refCameraPos);
	//return  TmpDis2 - TmpDis1*TmpDis1 ;
	//return  dot(d_cameraVec,TmpInsertPoint - d_refCameraPos);
	return  dot(d_cameraVec,normalize(TmpInsertPoint - d_refCameraPos));
}
enum planeType
{
	originType,
	validSearchType,
	invalidType,
};
#define PROJECTTHRES 0.02
class Plane
{
public:
	planeType m_type;
	float3 m_worldPos;
	float3 m_worldNormal;
	float m_dis;
	float3 m_reflectPos;
	float3 m_mirrorPos;
	float2 m_tc;
	int m_orginID;
	__device__ Plane()
	{
	}
	__device__ float2 getTc()
	{
		return m_tc;
	}
	__device__ int getID()
	{
		return m_orginID;
	}
	__device__ int isInValid()
	{
		return (m_type== invalidType);
	}
#define NORMALDIFF 0.001
#define DISDIFF 0.002
	__device__ bool isEqual(Plane& anotherPlane)
	{
		//printf("normal diff:%f, dis diff:%f\n",length(m_worldNormal-anotherPlane.m_worldNormal),dot(m_worldPos-anotherPlane.m_worldPos,m_worldNormal));
		if(length(m_worldNormal-anotherPlane.m_worldNormal)>NORMALDIFF)
		{
			//printf("normal diff,dis:%f,%d\n",length(m_worldNormal-anotherPlane.m_worldNormal),length(m_worldNormal-anotherPlane.m_worldNormal)>NORMALDIFF);
			///printf("(%f,%f,%f)\n",m_worldNormal.x,m_worldNormal.y,m_worldNormal.z);					
			//printf("(%f,%f,%f)\n",anotherPlane.m_worldNormal.x,anotherPlane.m_worldNormal.y,anotherPlane.m_worldNormal.z);
			return false;
		}
		if(dot(m_worldPos-anotherPlane.m_worldPos,m_worldNormal)>DISDIFF)
		{
			//printf("distance too mush:%f\n",dot(m_worldPos-anotherPlane.m_worldPos,m_worldNormal));
			return false;
		}
		return true;
	}
	// 查询是否是投影点 返回1 可以
	__device__ int isAbleFastProj(float2 reProjectTc)
	{
		 
		/*float4 WorldTmp = tex2D(cuda_WorlNormal_Tex,reProjectTc.x,reProjectTc.y);
		int reprjectId  = (int)WorldTmp.w;
		float3 m_worldNormal = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		WorldTmp = tex2D(cuda_WorldPos_Tex,reProjectTc.x,reProjectTc.y);
		float3 worldPos = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		*/
		
		Plane pixelPlane(reProjectTc);
		if(pixelPlane.isInValid())
		{
			//printf("is notVaid");
			return 0;
		}
		if(!isEqual(pixelPlane))
		{
			//printf("is NotEqual");
			return 0;
		}
		return 1;  
		// no need to do futher check; nearest sampling
		pixelPlane.setReflectedPos(*this);
		float DisPoint2Line;
		DisPoint2Line = pixelPlane.getDisToPath();
		if(PROJECTTHRES>DisPoint2Line)
		{
			//printf("within threshold");
			return 1;
		}
		//printf("finally");
		return 0;
	
	}
	__device__ Plane(float2 tc)
	{
		m_tc = tc;
		float4 WorldTmp;
		WorldTmp = tex2D(cuda_WorldPos_Tex,tc.x,tc.y);
		m_worldPos = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		WorldTmp = tex2D(cuda_WorlNormal_Tex,tc.x,tc.y);
		m_orginID = (int)WorldTmp.w;
		m_worldNormal = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
		m_dis = tex2D(cuda_Reflect_Tex,tc.x,tc.y).w;
	
	}
	
	
	__device__ void setPreviousPos( float d)
	{
		m_dis = d;
	
	}
	__device__ float3 getReflectedPos()
	{
		return m_reflectPos;
	}
	__device__ float getMirrorDepth()
	{
		float4 ProPos = cuda_Mat_Mut3(Cuda_MvpMat,m_mirrorPos);
		return ProPos.z/ProPos.w;
	}
	__device__ void setReflectedPos(Plane p)
	{
		m_reflectPos = p.getReflectedPos();
		if(m_orginID == p.getID())
		{
			m_type = validSearchType;
		}		
		else 
		{
			m_type = invalidType;
		}
	}
	__device__ void setReflectedPos()
	{
		float3 inComeDirection = m_worldPos - d_refCameraPos;
		float3 LookVec = normalize(inComeDirection);
		//计算反射光线方向
		float3 ReflectVec = normalize(reflect(LookVec,m_worldNormal));
		m_type = originType;
		m_reflectPos = m_worldPos + ReflectVec * m_dis;	
	}
	__device__ float3 getMirrorPos()
	{
		float3 reflectPos = m_reflectPos;
		m_mirrorPos = abs(dot(m_worldPos - reflectPos,m_worldNormal)) * (-2) * m_worldNormal + reflectPos;
		return m_mirrorPos;
	}
	__device__  float3 beneathPos(float3 sourcePos, float dis)
	{
		float3 inComeDirection = m_worldPos - sourcePos;
		float3 LookVec = normalize(inComeDirection);
		float3 ReflectVec = normalize(reflect(LookVec,m_worldNormal));

		float CosReCorner = dot(ReflectVec,m_worldNormal);
		float3 reflectedPos = m_worldPos + ReflectVec * dis;
		//镜像点的坐标
		float3 ReMirrorPos = dis * CosReCorner * 2 * (-1) * (m_worldNormal) + reflectedPos;

		//相机到反射面的距离
		return ReMirrorPos;
	}
	//得出以该点为平面的反射物体的镜像点
	__device__ float3 intersectPos()
	{
		getMirrorPos();
		float3 VecEye2Ref = normalize(m_mirrorPos - d_newCameraPos);
		return dot( m_worldPos -d_newCameraPos ,m_worldNormal)/ dot(VecEye2Ref,m_worldNormal) * VecEye2Ref + d_newCameraPos;
	}
	
	__device__ float3 intersetVirtualPos()
	{
		float3 ReMirrorPos =  abs(dot(m_worldPos - m_reflectPos,m_worldNormal)) * (-2) * m_worldNormal + m_reflectPos;


		//outPut[index] = make_float4(ReMirrorPos.x,ReMirrorPos.y,ReMirrorPos.z,1.0);
		//return;

		//求出该点与相机连线和平面的新交点
		float DisEye2Plane = abs(dot(d_newCameraPos - m_worldPos,m_worldNormal));
		float CoseTheta2 = abs(dot(normalize(d_newCameraPos - ReMirrorPos),m_worldNormal));
		float3 VecEye2Ref = normalize(ReMirrorPos - d_newCameraPos);
		float3 FinalPos2;
		//	FinalPos2 = d_newCameraPos + VecEye2Ref * (DisEye2Plane/CoseTheta2 );
		FinalPos2 = dot(m_worldPos -d_newCameraPos ,m_worldNormal)/ dot(VecEye2Ref,m_worldNormal) * VecEye2Ref + d_newCameraPos;
		return FinalPos2;
	}
	
	__device__ int checkConverge()
	{
#define CONVERGETHRES 5
		float3 FinalPos  = intersectPos();
		float3 ndcPos =  transFormToNdc(FinalPos);
		float2 ProPosUv =make_float2(ndcPos.x,ndcPos.y);
		//printf("!tc:(%f,%f)\n",m_tc.x,m_tc.y);
		//printf("!proPosUv(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		return length(m_tc-ProPosUv)<CONVERGETHRES;
	}
	__device__ float getSpherDistance()
	{
		float3 inComeDirection = m_worldPos - m_reflectPos;
		float3 LookVec = normalize(inComeDirection);
		//计算反射光线方向
		float3 reflectVec = normalize(reflect(LookVec,m_worldNormal));
		float3 cuttingPoint = m_worldPos+reflectVec*(dot(d_newCameraPos-m_worldPos,reflectVec));
		return length(cuttingPoint-d_newCameraPos);
	}
	__device__ float getDisToPath()
	{
		if(isInValid())
		{
			return 3000.0;
		}
		//求出新的镜像点
		return getSpherDistance();
		float3 ReMirrorPos =  getMirrorPos();
		//printf("!ReMirrorPos: (%f,%f,%f)\n",ReMirrorPos.x,ReMirrorPos.y,ReMirrorPos.z);
		//求相机与镜像点组成的平面的法线
		 
		float3 Cam_Mirror_Normal = normalize(cross(d_cameraVec,normalize(d_refCameraPos - ReMirrorPos)));
		//printf("!Cam_Mirror_Normal: (%f,%f,%f)\n",Cam_Mirror_Normal.x,Cam_Mirror_Normal.y,Cam_Mirror_Normal.z);

		//求相机轨迹所在法线
		float3 Camera_Vec_Normal = normalize(cross(Cam_Mirror_Normal,d_cameraVec));
		//printf("!Camera_Vec_Normal: (%f,%f,%f)\n",Camera_Vec_Normal.x,Camera_Vec_Normal.y,Camera_Vec_Normal.z);

		//分别求三个点与该平面的交点
		float3 InsertPoint = abs(dot(Camera_Vec_Normal,d_refCameraPos - ReMirrorPos)/dot(Camera_Vec_Normal,normalize(m_worldPos-ReMirrorPos))) * normalize(m_worldPos-ReMirrorPos) + ReMirrorPos;
		//printf("!InsertPoint: (%f,%f,%f)\n",InsertPoint.x,InsertPoint.y,InsertPoint.z);

		//求点与直线的距离

		float3 toNewPlace2 = d_newCameraPos-InsertPoint;
		float DisPoint2Line = length(toNewPlace2);
		//DisPoint2Line1  =  dot(normalize(InsertPoint1 - d_refCameraPos),d_cameraVec);

		//printf("!return value: (%f)\n",DisPoint2Line1);

		return DisPoint2Line;


	}
};
/*
__global__ void lineSearchKernel(int width,int height)
{


	//return;



	//return;
	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;

	if( x >width || y> height)
		//if(x !=696 || y !=640)
			//if(x !=373 || y !=139)
				return;
	int index = y * width + x;

	//提取数据
	float3 WorldPos,WorldNormal;
	float ReflectDis;
	float4 WorldTmp;

	WorldTmp = tex2D(cuda_WorldPos_Tex,x+0.5,y+0.5);

	WorldPos = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
	float4 WorldNormalTmp = tex2D(cuda_WorlNormal_Tex,x+0.5,y+0.5);

	int Origin_Id = (int)WorldNormalTmp.w;
	WorldNormal = normalize(make_float3(WorldNormalTmp.x,WorldNormalTmp.y,WorldNormalTmp.z));

	ReflectDis = tex2D(cuda_Reflect_Tex,x,y).w;

	Plane fittingPlane(&WorldPos,&WorldNormal);
	fittingPlane.setPreviousPos(&d_refCameraPos,ReflectDis,&d_newCameraPos);

	float3 ReflectPos =fittingPlane.reflectCamera();
	//printf("origin pos:(%f,%f,%f)\n",ReflectPos.x,ReflectPos.y,ReflectPos.z);
	//printf("0Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);

	float RefelctInCameraDepth;
	RefelctInCameraDepth = ReflectDis /10000.0;

	//RefelctInCameraDepth  = TMpReflecPosInCamera.z / TMpReflecPosInCamera.w;
	//镜像点的坐标
	float3 ReMirrorPos =fittingPlane.mirrorPos();

	//printf("1Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	//相机到反射面的距离
	float DisEye2Plane = abs(dot(d_newCameraPos - WorldPos,WorldNormal));
	//镜像点与相机的向量
	float3 VecEye2Ref = normalize(ReMirrorPos - d_newCameraPos);

	//camera2_pos = new camera position
	//camera1_pos = last camera position
	//镜像点与相机连线和反射平面的交点
	float3 FinalPos ;//= d_newCameraPos + DisEye2Plane/abs(dot(VecEye2Ref,WorldNormal))* VecEye2Ref;

	FinalPos = fittingPlane.intersectPos();

	float4 ProPos = cuda_Mat_Mut3(Cuda_MvpMat,FinalPos);

	//Pro 位置
	float2 ProPosUv;

	float2 TmpUv;

	ProPosUv = toNDC(ProPos)* make_float2(width,height);


	float2 MoveVec = ProPosUv - make_float2(x*1.0+0.5,y*1.0+0.5);
	TmpUv = make_float2(x*1.0,y*1.0);
	int IterTime = 0;

	float LastCosLength = 0;

	float ChengShu = 1.0f;

	float2 TmpMov = make_float2(0,0);

	float2 TheComputVU = make_float2(-1.0f,-1.0f);
	if(MoveVec.x >=0)
		TheComputVU.x =1;
	if(MoveVec.y >=0)
		TheComputVU.y =1;

	float Tmpfloat = Cuda_Compute_Cos(TheComputVU + TmpUv,ReflectPos,Origin_Id);
	if(Tmpfloat>-50 && Tmpfloat < 0)
		ChengShu = -1;

	float LastCoslength;
	//printf("2Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);

	while(IterTime<5 && dot(MoveVec,MoveVec) >=2.5)
	{
		//printf("3Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);

		float2 TheComputVU = make_float2(-1.0f,-1.0f);
		if(MoveVec.x >=0)
			TheComputVU.x =1;
		if(MoveVec.y >=0)
			TheComputVU.y =1;
		//TheComputVU = TheComputVU + TmpUv;


		float2 TmpUv1,TmpUv2,TmpUv3;
		// 		if(ChengShu >0){
		// 		float Tmpfloat = Cuda_Compute_Cos(TheComputVU + TmpUv,ReflectPos,Origin_Id);
		// 		if(Tmpfloat < -50 && Tmpfloat<0 )
		// 			ChengShu = -1;
		// 		else
		// 			ChengShu = 1;
		// 		}

		//int2 TmpUv1,TmpUv2,TmpUv3; 

		TmpUv1 = TmpUv;
		TmpUv2 = TmpUv;
		TmpUv3 = TmpUv;

		//MoveVec = ChengShu * MoveVec;

#if UseNine
		float2 TmpUvs[8];
		int NowSum = 0;
		for(int xi = -1;xi<=1;xi++)
			for(int yi = -1;yi<=1;yi++)
			{
				if((xi != 0 || yi !=0) )//&& length(TmpMov + make_float2(xi,yi)) > 0.5)
					TmpUvs[NowSum++] = TmpUv + make_float2(xi*1.0f,yi*1.0f);
				//TmpUvs[tmpIndex++] =make_float2((x+50)*1.0f,(y+50)*1.0f);
			}
			//outPut[index] = make_float4((TmpUvs[0].x+50)/1024.0,(TmpUvs[7].y+50)/1024.0,Origin_Id,RefelctInCameraDepth);
			//return;

			//取出每个采样点对应的世界坐标
			float3 WorldPosEs[8],WorldNormals[8];
			float4 TmpFloat;
			int ModelIds[8];
			for(int i =0;i<8;i++)
			{
				TmpFloat = tex2D(cuda_WorldPos_Tex,TmpUvs[i].x,TmpUvs[i].y);
				WorldPosEs[i] = make_float3(TmpFloat.x,TmpFloat.y,TmpFloat.z);
				TmpFloat = tex2D(cuda_WorlNormal_Tex,TmpUvs[i].x,TmpUvs[i].y);
				WorldNormals[i] = make_float3(TmpFloat.x,TmpFloat.y,TmpFloat.z);
				ModelIds[i] = (int) TmpFloat.w;

			}
			//outPut[index] = make_float4(TmpUv.x/1024.0,TmpUv.y/1024.0,Origin_Id,RefelctInCameraDepth);


#else
		if(MoveVec.x > 0){
			TmpUv1.x += 1.0;
			TmpUv3.x += 1.0;
		}
		else{
			TmpUv1.x -= 1.0;
			TmpUv3.x -= 1.0;
		}

		if(MoveVec.y >0){
			TmpUv2.y += 1.0;
			TmpUv3.y += 1.0;

		}
		else{
			TmpUv2.y -= 1.0;
			TmpUv3.y -= 1.0;
		}
		//取出每个采样点对应的世界坐标
		float4 TmpFloat4;
		float3 WorldPos1,WorldPos2,WorldPos3;
		float3 WorldNormal1,WorldNormal2,WorldNormal3;
		int Now_ModelId1,Now_ModelId2,Now_ModelId3;

		TmpFloat4 = tex2D(cuda_WorldPos_Tex,TmpUv1.x,TmpUv1.y);
		WorldPos1 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);

		TmpFloat4 = tex2D(cuda_WorldPos_Tex,TmpUv2.x,TmpUv2.y);
		WorldPos2 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);

		TmpFloat4 = tex2D(cuda_WorldPos_Tex,TmpUv3.x,TmpUv3.y);
		WorldPos3 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);

		TmpFloat4 = tex2D(cuda_WorlNormal_Tex,TmpUv1.x,TmpUv1.y);
		WorldNormal1 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);
		Now_ModelId1 = (int)TmpFloat4.w;

		TmpFloat4 = tex2D(cuda_WorlNormal_Tex,TmpUv3.x,TmpUv3.y);
		WorldNormal3 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);
		Now_ModelId3 = (int)TmpFloat4.w;

		TmpFloat4 = tex2D(cuda_WorlNormal_Tex,TmpUv2.x,TmpUv2.y);
		WorldNormal2 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);
		Now_ModelId2 = (int)TmpFloat4.w;

#endif



		//求出新的镜像点
		float DisPoint2Line1,DisPoint2Line2,DisPoint2Line3;
		DisPoint2Line1 = fittingPlane.getDisToPath(WorldPos1,WorldNormal1);
		DisPoint2Line2 = fittingPlane.getDisToPath(WorldPos2,WorldNormal2);
		DisPoint2Line3 = fittingPlane.getDisToPath(WorldPos3,WorldNormal3);

		//
		//printf("DisPoint2Line1:%f\n",DisPoint2Line1);
		float TmpDis;
		if(DisPoint2Line2<DisPoint2Line1)
		{
			TmpUv = TmpUv1;
			TmpUv1 = TmpUv2;
			TmpUv2 = TmpUv;
			WorldPos = WorldPos1;
			WorldPos1 = WorldPos2;
			WorldPos2 = WorldPos;

			TmpDis = DisPoint2Line1;
			DisPoint2Line1 = DisPoint2Line2;
			DisPoint2Line2 = TmpDis;

			TmpDis = Now_ModelId1;
			Now_ModelId1 = Now_ModelId2;
			Now_ModelId2 = TmpDis;

		}

		if(DisPoint2Line3< DisPoint2Line1){
			TmpUv = TmpUv1;
			TmpUv1 = TmpUv3;
			TmpUv3 = TmpUv;
			WorldPos = WorldPos1;
			WorldPos1 = WorldPos3;
			WorldPos3 = WorldPos;

			TmpDis = DisPoint2Line1;
			DisPoint2Line1 = DisPoint2Line2;
			DisPoint2Line2 = TmpDis;

			TmpDis = Now_ModelId1;
			Now_ModelId1 = Now_ModelId2;
			Now_ModelId2 = TmpDis;
		}

		if(DisPoint2Line3< DisPoint2Line2){
			TmpUv = TmpUv2;
			TmpUv2 = TmpUv3;
			TmpUv3 = TmpUv;
			WorldPos = WorldPos2;
			WorldPos2 = WorldPos3;
			WorldPos3 = WorldPos;

			TmpDis = DisPoint2Line1;
			DisPoint2Line1 = DisPoint2Line2;
			DisPoint2Line2 = TmpDis;

			TmpDis = Now_ModelId1;
			Now_ModelId1 = Now_ModelId2;
			Now_ModelId2 = TmpDis;
		}


		//TmpUv = TmpUv1;

		if(Now_ModelId1 == Origin_Id){
			TmpUv = TmpUv1;
			WorldPos = WorldPos1;
		}
		// 		else{
		// 		outPut[index] = make_float4(-10.0f,-10.0f,Origin_Id,RefelctInCameraDepth);
		// 			return;
		// 		}

		else if(Now_ModelId2 == Origin_Id){
			TmpUv = TmpUv2;
			WorldPos = WorldPos2;
		}
		else if(Now_ModelId3 == Origin_Id){
			TmpUv = TmpUv3;
			WorldPos = WorldPos3;
		}
		else{
			d_cudaPboBuffer[index] = make_float4(-10.0f,-10.0f,Origin_Id,RefelctInCameraDepth);
			return;
		}
		TmpUv = TmpUv+TheComputVU;

		//LastCoslength = Cuda_Compute_Cos(TmpUv,ReflectPos,Origin_Id);
		//LastCoslength = 0;
		//if(LastCoslength > length(Cuda_CameraPos1[1] - Cuda_CameraPos2[1])-0.5)
		//break;



		//得出该点法线

		float4 TmpFloat41= tex2D(cuda_WorlNormal_Tex,TmpUv.x,TmpUv.y);
		WorldNormal  = make_float3(TmpFloat41.x,TmpFloat41.y,TmpFloat41.z);


		float NowModelId = TmpFloat41.w;
		fittingPlane = Plane(&WorldPos,&WorldNormal);



		float3 FinalPos2 =fittingPlane.intersetPos2();
		//printf("!WorldNormal: (%f,%f,%f)\n",WorldNormal.x,WorldNormal.y,WorldNormal.z);
		//printf("!WorldPos: (%f,%f,%f)\n",WorldPos.x,WorldPos.y,WorldPos.z);
		//printf("!FinalPos2: (%f,%f,%f)\n",FinalPos2.x,FinalPos2.y,FinalPos2.z);

		float4 ProPos2;
		ProPos2 = cuda_Mat_Mut3(Cuda_MvpMat,FinalPos2);
		ProPosUv = toNDC(ProPos2)* make_float2(width,height);


		MoveVec = ProPosUv - TmpUv;
		//printf("!ProPosUv: (%f,%f)\n",ProPosUv.x,ProPosUv.y);
		//printf("!TmpUv: (%f,%f)\n",TmpUv.x,TmpUv.y);
		//printf("!moveVec:(%f,%f)\n",MoveVec.x,MoveVec.y);
		IterTime++;

	}

	//if(length(TmpUv - make_float2(x,y))> 30){
	//outPut[index] = make_float4(-10.0,-10.0,Origin_Id,RefelctInCameraDepth);
	//return;

	//}

	//if(dot(MoveVec,MoveVec) >=20){
	//outPut[index] = make_float4(-10.0f,-10.0f,Origin_Id,RefelctInCameraDepth);
	//return;

	//}


	//outPut[index] = ProPos;
	//if(TmpUv.x>=0 && TmpUv.x<width && TmpUv.y>=0 && TmpUv.y <= height)
	//	outPut[(int)(TmpUv.y * width + TmpUv.x)] = tex2D(cuda_Reflect_Tex,x+0.5y+0.5);
	//WorlPosInCameraDepth = 10;
	//if(dot(MoveVec,MoveVec)>200)
	//WorlPosInCameraDepth = -10;
	//outPut[index] = make_float4((TmpUv.x - x)/1024.0,(TmpUv.y - y)/1024.0,Origin_Id,RefelctInCameraDepth);
	d_cudaPboBuffer[index] = make_float4(TmpUv.x/(float)width,TmpUv.y/(float)height,Origin_Id,RefelctInCameraDepth);

}*/

__device__ int isInSamePixel(float2 moveVec)
{
	if(abs(moveVec.x)<0.5&&abs(moveVec.y)<0.5)
		return 1;
	else
		return 0;
}
__device__ int nextStep(float2 moveVec,float2* pStepVec)
{
#define PI 3.141592654f
	if(length(moveVec)<0.717)
		return 0;
	if(moveVec.x==0)
	{
		*pStepVec = make_float2(0,copysignf(1,moveVec.y));
		return 1;
	}
	float absTangleValue = abs(moveVec.y/moveVec.x);
	if(absTangleValue<tanf(PI/4/2))
	{
		
		*pStepVec = make_float2(copysignf(1,moveVec.x),0);

	}
	else if (absTangleValue>tanf(PI*3/4/2))
	{
		
		*pStepVec = make_float2(0,copysignf(1,moveVec.y));
	}
	else 
	{
		*pStepVec = make_float2(copysignf(1,moveVec.x),copysignf(1,moveVec.y));
	}
	return 1;
}
__device__ int nextThreeStep(float2 moveVec,float2* candicate1,float2* candicate2,float2* candicate3)
{
	#define PI 3.141592654f
	if(length(moveVec)<0.717)
		return 0;
	if(moveVec.x==0)
	{
		*candicate1 = make_float2(0,copysignf(1,moveVec.y));
		*candicate2 = make_float2(-1,copysignf(1,moveVec.y));
		*candicate3 = make_float2(1,copysignf(1,moveVec.y));
		return 1;
	}
	float absTangleValue = abs(moveVec.y/moveVec.x);
	if(absTangleValue<tanf(PI/4/2))
	{
		
		*candicate1 = make_float2(copysignf(1,moveVec.x),0);
		*candicate2 = make_float2(copysignf(1,moveVec.x),-1);
		*candicate3 = make_float2(copysignf(1,moveVec.x),1);

	}
	else if (absTangleValue>tanf(PI*3/4/2))
	{
		
		*candicate1 = make_float2(0,copysignf(1,moveVec.y));
		*candicate2 = make_float2(-1,copysignf(1,moveVec.y));
		*candicate3 = make_float2(1,copysignf(1,moveVec.y));
	}
	else 
	{
		*candicate1 = make_float2(copysignf(1,moveVec.x),copysignf(1,moveVec.y));
		*candicate2 = make_float2(copysignf(1,moveVec.x),0);
		*candicate3 = make_float2(0,copysignf(1,moveVec.y));
	}
	return 1;
}
#define CONVERGE 5
#define OUTRANGE 2
#define OUTOBJECT 3
#define FASTPROJT 4
__device__ int Floor2Int(float f)
{
	return (int)f+0.5;
}
__device__ int isOutOfRange(float2 tc)
{
	return tc.x>rasterWidth-0.5||tc.y>rasterHeight-0.5||tc.x<0||tc.y<0;
}
__device__ int threePointSearch(float2 currentPlace,float2* moveToVec)
{
#define STEPNUMBER 25
	int x =Floor2Int(currentPlace.x-0.5);
	int y =Floor2Int(currentPlace.y-0.5);
	int index = y * rasterWidth + x;
	float2 currentUv = make_float2(currentPlace.x,currentPlace.y);

	Plane fittingPlane(currentUv);
	
	//if(x!=399||y!=703)
	//   return;
	
	//printf("1Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	//�����������ľ���
	//= d_newCameraPos + DisEye2Plane/abs(dot(VecEye2Ref,WorldNormal))* VecEye2Ref;
	fittingPlane.setReflectedPos();
	float3 FinalPos  = fittingPlane.intersectPos();
	float3 ndcPos =  transFormToNdc(FinalPos);
	float2 ProPosUv =make_float2(ndcPos.x,ndcPos.y);
	float rejectDepth = fittingPlane.getMirrorDepth();
	float2 MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
	float formerDis = fittingPlane.getDisToPath();

	
	float3 worldPos = fittingPlane.m_worldPos;
	float3 worldNormal = fittingPlane.m_worldNormal;
	
	/*
	printf("x,y:%d,%d\n",x,y);
	printf("pos:(%f,%f,%f)\n",worldPos.x,worldPos.y,worldPos.z);
	printf("normal:(%f,%f,%f)\n",worldNormal.x,worldNormal.y,worldNormal.z);
	printf("reflected pos:(%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	printf("intersect pos:(%f,%f,%f)\n",FinalPos.x,FinalPos.y,FinalPos.z);
	printf("refCamera:(%f,%f,%f)\n",d_refCameraPos.x,d_refCameraPos.y,d_refCameraPos.z);
	printf("newCameraPos:(%f,%f,%f)\n",d_newCameraPos.x,d_newCameraPos.y,d_newCameraPos.z);
	printf("reject Pos:(%f,%f),depth:%f\n",ProPosUv.x,ProPosUv.y,rejectDepth);
	*/
	if(fittingPlane.isAbleFastProj(ProPosUv))
	{
		if(fabs(ProPosUv.x-currentUv.x)<0.5&&(fabs(ProPosUv.y-currentUv.y)<0.5))
		{
			d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
			return CONVERGE;
		}
		d_cudaPboBuffer[index] =   make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,FASTPROJT);
		return FASTPROJT;
	}
	int IterTime = 0;
	while(IterTime<STEPNUMBER/1024.0*rasterWidth)
	{
		//printf("Item:%d currentUv:(%f,%f)\n",IterTime,currentUv.x,currentUv.y);
		
		float2 TmpUv1,TmpUv2,TmpUv3,TmpUv;
		
		TmpUv1 = currentUv;
		TmpUv2 = currentUv;
		TmpUv3 = currentUv;

		//MoveVec = ChengShu * MoveVec;


		if(MoveVec.x > 0)
		{
			TmpUv1.x += 1.0;
			TmpUv3.x += 1.0;
		}
		else{
			TmpUv1.x -= 1.0;
			TmpUv3.x -= 1.0;
		}

		if(MoveVec.y >0){
			TmpUv2.y += 1.0;
			TmpUv3.y += 1.0;

		}
		else{
			TmpUv2.y -= 1.0;
			TmpUv3.y -= 1.0;
		}
		//取出每个采样点对应的世界坐标
		//nextThreeStep(MoveVec,&TmpUv1,&TmpUv2,&TmpUv3);
		/*TmpUv1+=currentUv;
		TmpUv2+=currentUv;
		TmpUv3+=currentUv;*/
		if(isOutOfRange(TmpUv1)||isOutOfRange(TmpUv2)||isOutOfRange(TmpUv3))
		{

			d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTOBJECT);
		  
			return OUTOBJECT;
		}
		Plane pixelPlane1(TmpUv1),pixelPlane2(TmpUv2),pixelPlane3(TmpUv3);
		pixelPlane1.setReflectedPos(fittingPlane);
		pixelPlane2.setReflectedPos(fittingPlane);
		pixelPlane3.setReflectedPos(fittingPlane);

		float DisPoint2Line1,DisPoint2Line2,DisPoint2Line3;
		DisPoint2Line1 = pixelPlane1.getDisToPath();
		DisPoint2Line2 = pixelPlane2.getDisToPath();
		DisPoint2Line3 = pixelPlane3.getDisToPath();
		
		for(int dx = -1;dx<=1;dx++)
		{
			for(int dy = -1;dy<=1;dy++)
			{
				float2 uv = currentUv+make_float2(dx,dy);
				Plane testPlan(uv);
				testPlan.setReflectedPos(fittingPlane);
				float dis = testPlan.getDisToPath();
				//printf("point(%f,%f,%f)\n",uv.x,uv.y,dis);

			}
		}
		//printf("currentProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		//printf("currentMove(%f,%f)\n",MoveVec.x,MoveVec.y);
		//printf("point1(%f,%f,%f),point2(%f,%f,%f),point3(%f,%f,%f)\n",TmpUv1.x,TmpUv1.y,DisPoint2Line1,TmpUv2.x,TmpUv2.y,DisPoint2Line2,TmpUv3.x,TmpUv3.y,DisPoint2Line3);
		
		Plane minPlane;
		float minDis;
		//
		//printf("DisPoint2Line1:%f\n",DisPoint2Line1);
		if(pixelPlane1.isInValid()&&pixelPlane2.isInValid()&&pixelPlane2.isInValid())
		{	
			//printf("invalid\n");
			
			d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTOBJECT);
		   //printf("@x:%d y:%d: %f,%f,%f,%f\n",x,y,d_cudaPboBuffer[index] .x,d_cudaPboBuffer[index] .y,d_cudaPboBuffer[index] .z,d_cudaPboBuffer[index] .w);
	
			return OUTOBJECT;
		}
		if(DisPoint2Line2<DisPoint2Line1)
		{
			if(DisPoint2Line3< DisPoint2Line2)
			{
				minPlane = pixelPlane3;
				minDis = DisPoint2Line3;
				// min = 3
			}	
			else
			{
				minPlane = pixelPlane2;
				minDis = DisPoint2Line2;
				//min  = 2
			}
		}
		else
		{
			if(DisPoint2Line3< DisPoint2Line1)
			{
				minPlane = pixelPlane3;
				minDis = DisPoint2Line3;
				// min = 3
			}	
			else
			{
				minPlane = pixelPlane1;		
				minDis = DisPoint2Line1;
				//min  = 1
			}
		}
		if(minDis>formerDis)
		{
			*moveToVec = currentUv;
			if(length(MoveVec)<15/1024.0*rasterWidth&&minDis<5)
			{
				d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
				//printf("minDis:%f,formerDis:%f,Converge\n",minDis,formerDis);
				return CONVERGE;
			}
			else
			{
				d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,-0.1,OUTOBJECT);
				//printf("not right\n");
				
				return OUTOBJECT;
			
			}
		}
		//TmpUv = TmpUv1;
		
		currentUv = minPlane.getTc();
		formerDis = minDis;
		fittingPlane = minPlane;

		FinalPos  = fittingPlane.intersectPos();
		ndcPos =  transFormToNdc(FinalPos);
	    ProPosUv =make_float2(ndcPos.x,ndcPos.y);
		rejectDepth = fittingPlane.getMirrorDepth();
		MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
		/*
		printf("nextUv minDis:(%f,%f),currentFormerDis:%f\n",currentUv.x,currentUv.y,formerDis);
		printf("ProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		printf("Move(%f,%f)\n",MoveVec.x,MoveVec.y);
		printf("worldPos(%f,%f,%f),normal:(%f,%f,%f):%f\n",fittingPlane.m_worldPos.x,fittingPlane.m_worldPos.y,fittingPlane.m_worldPos.z,fittingPlane.m_worldNormal.x,fittingPlane.m_worldNormal.y,fittingPlane.m_worldNormal.z);
		*/
		IterTime++;
	}
	d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTRANGE);
				
//	printf("！x:%d y:%d: %f,%f,%f,%f\n",x,y,d_cudaPboBuffer[index] .x,d_cudaPboBuffer[index] .y,d_cudaPboBuffer[index] .z,d_cudaPboBuffer[index] .w);
	
	
	return OUTRANGE;
	
}

__device__ int ninePointSearch(float2 currentPlace,float2* moveToVec)
{
#define STEPNUMBER 25
	int x =Floor2Int(currentPlace.x-0.5);
	int y =Floor2Int(currentPlace.y-0.5);
	int index = y * rasterWidth + x;
	float2 currentUv = make_float2(currentPlace.x,currentPlace.y);

	Plane fittingPlane(currentUv);
	
	//if(x!=140||y!=294)
	 //  return;
	
	//printf("1Class: (%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	//�����������ľ���
	//= d_newCameraPos + DisEye2Plane/abs(dot(VecEye2Ref,WorldNormal))* VecEye2Ref;
	fittingPlane.setReflectedPos();
	float3 FinalPos  = fittingPlane.intersectPos();
	float3 ndcPos =  transFormToNdc(FinalPos);
	float2 ProPosUv =make_float2(ndcPos.x,ndcPos.y);
	float rejectDepth = fittingPlane.getMirrorDepth();
	float2 MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
	float formerDis = fittingPlane.getDisToPath();

	
	float3 worldPos = fittingPlane.m_worldPos;
	float3 worldNormal = fittingPlane.m_worldNormal;
	
	/*
	printf("x,y:%d,%d\n",x,y);
	printf("pos:(%f,%f,%f)\n",worldPos.x,worldPos.y,worldPos.z);
	printf("normal:(%f,%f,%f)\n",worldNormal.x,worldNormal.y,worldNormal.z);
	printf("reflected pos:(%f,%f,%f)\n",fittingPlane.m_reflectPos.x,fittingPlane.m_reflectPos.y,fittingPlane.m_reflectPos.z);
	printf("intersect pos:(%f,%f,%f)\n",FinalPos.x,FinalPos.y,FinalPos.z);
	printf("refCamera:(%f,%f,%f)\n",d_refCameraPos.x,d_refCameraPos.y,d_refCameraPos.z);
	printf("newCameraPos:(%f,%f,%f)\n",d_newCameraPos.x,d_newCameraPos.y,d_newCameraPos.z);
	printf("reject Pos:(%f,%f),depth:%f\n",ProPosUv.x,ProPosUv.y,rejectDepth);
	*/
	if(fittingPlane.isAbleFastProj(ProPosUv))
	{
		//printf("fitting\n");
		if(fabs(ProPosUv.x-currentUv.x)<0.5&&(fabs(ProPosUv.y-currentUv.y)<0.5))
		{
			d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
			return CONVERGE;
		}
				
		d_cudaPboBuffer[index] =   make_float4(ProPosUv.x/(float)rasterWidth,ProPosUv.y/(float)rasterHeight,rejectDepth,FASTPROJT);
		return FASTPROJT;
	}
	int IterTime = 0;
	while(IterTime<STEPNUMBER/1024.0*rasterWidth)
	{
		float minDis = 3000;
		float2 minUv;
		Plane minPlane;
		for(int dx = -1;dx<=1;dx++)
		{
			for(int dy = -1;dy<=1;dy++)
			{
				float2 uv = currentUv+make_float2(dx,dy);
				if(dx==0&&dy==0)
					continue;
				if(isOutOfRange(uv))
					continue;
				Plane testPlan(uv);
				testPlan.setReflectedPos(fittingPlane);
				
				if(testPlan.isInValid())
					continue;
				float dis = testPlan.getDisToPath();
				//printf("point1(%f,%f,%f)\n",uv.x,uv.y,dis);
				if(minDis>dis)
				{
					//printf("min\n");
					minDis = dis;
					minUv = uv;
					minPlane = testPlan;
				}

			}
		}
			//
		//printf("DisPoint2Line1:%f\n",DisPoint2Line1);
		if(minDis == 3000)
		{
			d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,-0.1,OUTOBJECT);
			return OUTOBJECT;
		}
		if(minDis>formerDis)
		{
			*moveToVec = currentUv;
			if(length(MoveVec)<15/1024.0*rasterWidth&&minDis<5)
			{
				//printf("convenge\n");
				d_cudaPboBuffer[index] =  make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,rejectDepth,CONVERGE);
				return CONVERGE;
			}
			else
			{
				//printf("out\n");
				
				d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,-0.1,OUTOBJECT);
				return OUTOBJECT;
			}
		}
		
		currentUv = minPlane.getTc();
		formerDis = minDis;
		fittingPlane = minPlane;

		FinalPos  = fittingPlane.intersectPos();
		ndcPos =  transFormToNdc(FinalPos);
	    ProPosUv =make_float2(ndcPos.x,ndcPos.y);
		rejectDepth = fittingPlane.getMirrorDepth();
		MoveVec = ProPosUv - make_float2(currentUv.x,currentUv.y);
		/*
		printf("nextUv minDis:(%f,%f),currentFormerDis:%f\n",currentUv.x,currentUv.y,formerDis);
		printf("ProUV(%f,%f)\n",ProPosUv.x,ProPosUv.y);
		printf("Move(%f,%f)\n",MoveVec.x,MoveVec.y);
		printf("worldPos(%f,%f,%f),normal:(%f,%f,%f):%f\n",fittingPlane.m_worldPos.x,fittingPlane.m_worldPos.y,fittingPlane.m_worldPos.z,fittingPlane.m_worldNormal.x,fittingPlane.m_worldNormal.y,fittingPlane.m_worldNormal.z);
		*/
		IterTime++;
	}
	d_cudaPboBuffer[index] =   make_float4(-10.0,-10.0,rejectDepth,OUTRANGE);		
	return OUTRANGE;
	
}

__global__ void MyNewKernel(int width,int height)
{
	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;
	if( x >width || y> height)
			return;
	//if(x!=91||y!=623)
	//	return;
	
	//printf("x:%d,y:%d\n",x,y);
	/*if(x==1024)
	{
		printf("x:%d,y:%d\n",x,y);
	}*/
	int index = y * width + x;
	float2 currentUv = make_float2(x+0.5,y+0.5);
	
//	d_cudaPboBuffer[index] =   make_float4(currentUv.x/(float)rasterWidth,currentUv.y/(float)rasterHeight,-0.1,FASTPROJT);
//	return;		
	float2 resultValue;
	ninePointSearch(currentUv,&resultValue);
	

}
__global__ void MyKernel(int width,int height)
{
	//+0.5 thresheld


	//return;
	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;

	//if( x >width || y> height)
		//int x = 321,y =169;  

	//if(x !=0 || y !=0)
	//if(x!=321||y !=169)

	//if(x!=571||y!=702)
	//if(x!=950||y!=990)
	//if( x >width || y> height)
	//	return;
	int index = y * width + x;
	
	/*if(x>=264||x<258)
	{
		d_cudaPboBuffer[index] = make_float4(-10.0f,-10.0f,1,0.1);
		return;
	}
	if(y>=100||y<90)
	{
		d_cudaPboBuffer[index] = make_float4(-10.0f,-10.0f,1,0.1);
		return;
	};*/
	if( x >width || y> height)
			return;


	//printf("%d,%d\n",x,y);
	//提取数据
	
	float3 WorldPos,WorldNormal;
	float ReflectDis;
	float4 WorldTmp;

	WorldTmp = tex2D(cuda_WorldPos_Tex,x+0.5,y+0.5);

	WorldPos = make_float3(WorldTmp.x,WorldTmp.y,WorldTmp.z);
	float4 WorldNormalTmp = tex2D(cuda_WorlNormal_Tex,x+0.5,y+0.5);

	int Origin_Id = (int)WorldNormalTmp.w;
	WorldNormal = normalize(make_float3(WorldNormalTmp.x,WorldNormalTmp.y,WorldNormalTmp.z));

	ReflectDis = tex2D(cuda_Reflect_Tex,x+0.5,y+0.5).w;


	float3 LookVec = normalize(WorldPos - d_refCameraPos);
	//计算反射光线方向
	float3 ReflectVec = normalize(reflect(LookVec,WorldNormal));
	//计算被反射物体坐标
	float3 ReflectPos = WorldPos + ReflectVec * ReflectDis;
	float RefelctInCameraDepth;

	RefelctInCameraDepth = ReflectDis /10000.0;

	float WorlPosInCameraDepth = 1000;
	//RefelctInCameraDepth  = TMpReflecPosInCamera.z / TMpReflecPosInCamera.w;


	float CosReCorner = dot(ReflectVec,WorldNormal);
	//镜像点的坐标
	float3 ReMirrorPos = ReflectDis * CosReCorner * 2 * (-1) * WorldNormal + ReflectPos;


	//相机到反射面的距离
	float DisEye2Plane = abs(dot(d_newCameraPos- WorldPos,WorldNormal));
	//镜像点与相机的向量
	float3 VecEye2Ref = normalize(ReMirrorPos - d_newCameraPos);

	//镜像点与相机连线和反射平面的交点
	float3 FinalPos ;//= d_newCameraPos + DisEye2Plane/abs(dot(VecEye2Ref,WorldNormal))* VecEye2Ref;

	FinalPos = dot(WorldPos -d_newCameraPos ,WorldNormal)/ dot(VecEye2Ref,WorldNormal) * VecEye2Ref + d_newCameraPos;

	//printf("^^^^^^^^^^^^^^^^^^^MyKernel^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^\n");
	//printf("^^^^^^^^WorldPos^^^:%f,%f,%f\n",WorldPos.x,WorldPos.y,WorldPos.z);
	//printf("^^^^^^^^WorldNormal^^^:%f,%f,%f\n",WorldNormal.x,WorldNormal.y,WorldNormal.z);
	//printf("^^^^^^^^d_newCameraPos^^^:%f,%f,%f\n",d_newCameraPos.x,d_newCameraPos.y,d_newCameraPos.z);

	//printf("^^^^^^^^ReflectPos^^^:%f,%f,%f\n",ReflectPos.x,ReflectPos.y,ReflectPos.z);
	//printf("^^^^^^^^ReMirrorPos^^^:%f,%f,%f\n",ReMirrorPos.x,ReMirrorPos.y,ReMirrorPos.z);
	//printf("^^^^^^^^VecEye2Ref^^^:%f,%f,%f\n",VecEye2Ref.x,VecEye2Ref.y,VecEye2Ref.z);

	//printf("^^^^^^^^FinalPos_F^^^:%f,%f,%f\n",FinalPos.x,FinalPos.y,FinalPos.z);
	//printf("^^^^^^^^MirrorPos：%f,%f,%f\n",ReMirrorPos.x,ReMirrorPos.y,ReMirrorPos.z);
	
	//FinalPos = abs(dot(Camera_Vec_Normal3,d_refCameraPos - ReMirrorPos3)/dot(Camera_Vec_Normal3,normalize(WorldPos3-ReMirrorPos3))) * normalize(WorldPos3-ReMirrorPos3) + d_newCameraPos;

	float FinalPos_F[4];
	FinalPos_F[0] = FinalPos.x;
	FinalPos_F[1] = FinalPos.y;
	FinalPos_F[2] = FinalPos.z;
	FinalPos_F[3] = 1.0;
	float4 ProPos = Cude_Mul_Mat4(Cuda_MvpMat,FinalPos_F);
	//printf("^^^^^^^^ProPos^^^:%f,%f,%f,%f\n",ProPos.x,ProPos.y,ProPos.z,ProPos.w);

	//outPut[index] = make_float4(ProPos.x,ProPos.y,ProPos.z,1.0);

	//return;

	//outPut[index] = WorldTmp;
	//return ;
	//Pro 位置
	ProPos.x = ProPos.x / ProPos.w;
	ProPos.y = ProPos.y / ProPos.w;

	//ProPos.x = (ProPos.x * 0.5 +0.5 ) * width;
	//ProPos.y = (ProPos.y * 0.5 +0.5 ) * height;

	//float2 TmpUv ,ProPosUv;
	float2 ProPosUv;

	float2 cuurrentUv = make_float2((x+0.5)*1.0,(y+0.5)*1.0);;
	//int2 cuurrentUv;

	ProPosUv.x = (ProPos.x * 0.5 +0.5 ) * width;
	ProPosUv.y = (ProPos.y * 0.5 +0.5 ) * height;
	//float2 writenPlace = make_float2(ProPosUv.x-cuurrentUv.x,ProPosUv.y-cuurrentUv.y);
	//d_cudaPboBuffer[index] =  make_float4(ProPosUv.x/(float)width,ProPosUv.y/(float)height,Origin_Id,RefelctInCameraDepth);
	//printf("refCam:%f,%f,%f\n",d_refCameraPos.x,d_refCameraPos.y,d_refCameraPos.z);
	//printf("newCam:%f,%f,%f\n",d_newCameraPos.x,d_newCameraPos.y,d_newCameraPos.z);

	//printf("result(%f,%f)\n",ProPosUv.x,ProPosUv.y);

	//printf("ProPosUV:%f,%f\n",ProPosUv.x,ProPosUv.y);
	
	//outPut[index] = make_float4(WorldNormal.x,WorldNormal.y,WorldNormal.z,1.0);
	//return;
	float4 TmpFloat4;

	TmpFloat4 = tex2D(cuda_WorldPos_Tex,ProPosUv.x,ProPosUv.y);
	float3	test = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);
	//printf("value:(%f,%f,%f)\n",test.x,test.y,test.z);
	float2 MoveVec = ProPosUv - make_float2((x+0.5)*1.0,(y+0.5)*1.0);

	//outPut[index] = make_float4(MoveVec.x,MoveVec.y,dot(MoveVec,MoveVec),ReflectDis);
	//return;

	//if(Cuda_FramMent[0]%10 ==1)

	//else
	//cuurrentUv = LastVec[index];

	//outPut[index] = make_float4((cuurrentUv.x+50)/1024.0,(cuurrentUv.y+50)/1024.0,Origin_Id,RefelctInCameraDepth);
	//return;

	//outPut[index] = make_float4(MoveVec.x/1024.0,MoveVec.y/1024.0,Origin_Id,RefelctInCameraDepth);
	//outPut[index] = make_float4(ProPosUv.x/1024.0,ProPosUv.y/1024.0,Origin_Id,RefelctInCameraDepth);
	//return;
	//cuurrentUv = make_int2(x,y);
	//return ;
	int IterTime = 0;

	float LastCosLength = 0;

	float ChengShu = 1.0f;

	float2 TmpMov = make_float2(0,0);

	float2 TheComputVU = make_float2(-1.0f,-1.0f);
	if(MoveVec.x >=0)
		TheComputVU.x =1;
	if(MoveVec.y >=0)
		TheComputVU.y =1;

	float Tmpfloat = Cuda_Compute_Cos(TheComputVU + cuurrentUv,ReflectPos,Origin_Id);
	if(Tmpfloat>-50 && Tmpfloat < 0)
		ChengShu = -1;
	//	else
	//ChengShu = 1;



	//if(Tmpfloat >-50 &&Tmpfloat <0)
	//ChengShu = -1;
	float LastCoslength;
	//while(IterTime<Cuda_FramMent[0] * 5 && dot(MoveVec,MoveVec) >=2.5){


	//outPut[index] = make_float4(cuurrentUv.x/1024.0,cuurrentUv.y/1024.0,Origin_Id,RefelctInCameraDepth);
	//return;
	//printf("^^^^^^^^cuurrentUv^:%f,%f\n",cuurrentUv.x,cuurrentUv.y);
	//printf("^^^^^^^^MoveVec^^^:%f,%f\n",MoveVec.x,MoveVec.y);
	float3	Cam_Mirror_Normal = normalize(cross(d_cameraVec,normalize(d_refCameraPos - ReMirrorPos)));
	float3	Camera_Vec_Normal = normalize(cross(Cam_Mirror_Normal,d_cameraVec));
	float3 insertPoint = abs(dot(Camera_Vec_Normal,d_refCameraPos - ReMirrorPos)/dot(Camera_Vec_Normal,normalize(WorldPos-ReMirrorPos))) * normalize(WorldPos-ReMirrorPos) + ReMirrorPos;
	//printf("^^^^^^^^insertPoint^^^:%f,%f,%f\n",insertPoint.x,insertPoint.y,insertPoint.z);
	//printf("^^^^^^^^d_refCameraPos^^^:%f,%f,%f\n",d_refCameraPos.x,d_refCameraPos.y,d_refCameraPos.z);


	//float temp1 = dot(InsertPoint - d_refCameraPos,d_cameraVec);

	//float temp2 = dot(InsertPoint - d_refCameraPos,InsertPoint - d_refCameraPos);
	float3 toNewPlace = d_newCameraPos-insertPoint;
	//float	formerDis = dot(toNewPlace,d_cameraVec);
	float formerDis = length(toNewPlace);
	//printf("^^^^^^^^temp1,temp2,formerDis^^^:,%f\n",formerDis);
	float currentMaxDis = -10000;
		
	while(IterTime<25)
	{
		//while(1){
		//printf("in the loop  %d\n",IterTime);
		float2 TheComputVU = make_float2(-1.0f,-1.0f);
		if(MoveVec.x >=0)
			TheComputVU.x =1;
		if(MoveVec.y >=0)
			TheComputVU.y =1;
		//TheComputVU = TheComputVU + cuurrentUv;
		//printf("^^^^^^^^^^^^^^^^formerDis:%f^^^^^^^^^^\n",formerDis);
		//printf("IterTime:%d ^^^^^^^^cuurrentUv^^^:%f,%f\n",IterTime,cuurrentUv.x,cuurrentUv.y);
		
		/*if(1==nextStep(MoveVec,&TheComputVU))
		{
			printf("^^^^^^^^TheComputVU^^^:%f,%f\n",TheComputVU.x,TheComputVU.y);
			cuurrentUv += TheComputVU;
			printf("^^^^^^^^cuurrentUv^^^:%f,%f\n",cuurrentUv.x,cuurrentUv.y);
			float4 TmpFloat41= tex2D(cuda_WorlNormal_Tex,cuurrentUv.x,cuurrentUv.y);
			WorldNormal  = make_float3(TmpFloat41.x,TmpFloat41.y,TmpFloat41.z);
			TmpFloat41 = tex2D(cuda_WorldPos_Tex,cuurrentUv.x,cuurrentUv.y);
			WorldPos  = make_float3(TmpFloat41.x,TmpFloat41.y,TmpFloat41.z);
			float NowModelId = TmpFloat41.w;



			//outPut[index] = make_float4(WorldNormal.x,WorldNormal.y,WorldNormal.z,1.0);
			//return;
			//得出以该点为平面的反射物体的镜像点
			ReMirrorPos =  abs(dot(WorldPos - ReflectPos,WorldNormal)) * (-2) * WorldNormal + ReflectPos;



			//outPut[index] = make_float4(ReMirrorPos.x,ReMirrorPos.y,ReMirrorPos.z,1.0);
			//return;

			//求出该点与相机连线和平面的新交点
			DisEye2Plane = abs(dot(d_newCameraPos - WorldPos,WorldNormal));
			float CoseTheta2 = abs(dot(normalize(d_newCameraPos - ReMirrorPos),WorldNormal));
			VecEye2Ref = normalize(ReMirrorPos - d_newCameraPos);
			float3 FinalPos2;
			//	FinalPos2 = d_newCameraPos + VecEye2Ref * (DisEye2Plane/CoseTheta2 );
			FinalPos2 = dot(WorldPos -d_newCameraPos ,WorldNormal)/ dot(VecEye2Ref,WorldNormal) * VecEye2Ref + d_newCameraPos;

			//FinalPos2.y =  d_newCameraPos.y + VecEye2Ref.y *(DisEye2Plane/CoseTheta2);

			FinalPos_F[0] = FinalPos2.x;
			FinalPos_F[1] = FinalPos2.y;
			FinalPos_F[2] = FinalPos2.z;
			FinalPos_F[3] = 1.0;
			float4 ProPos2;
			ProPos2 = Cude_Mul_Mat4(Cuda_MvpMat,FinalPos_F);

			ProPosUv.x = (ProPos2.x / ProPos2.w * 0.5 + 0.5) * width;
			ProPosUv.y = (ProPos2.y / ProPos2.w * 0.5 + 0.5) * height;
			printf("now proposUv:(%f,%f)\n",ProPosUv.x,ProPosUv.y);
			//MoveVec = ProPosUv - make_float2(cuurrentUv.x,cuurrentUv.y);
			MoveVec = ProPosUv - cuurrentUv;
			printf("MoveVec:(%f,%f)\n",MoveVec.x,MoveVec.y);
			
			//outPut[index] = make_float4(ProPosUv.x,ProPosUv.y,ProPos2.z,ProPos2.w);

			//return;
			IterTime++;
		}
		else
		{
			printf("$$$$$$$$$$$$$$$$$$$$$$result (%f,%f)\n",cuurrentUv.x,cuurrentUv.y);
			d_cudaPboBuffer[index] =  make_float4(cuurrentUv.x/(float)width,cuurrentUv.y/(float)height,Origin_Id,RefelctInCameraDepth);
			return;
		}
		*/
		
		float2 TmpUv1,TmpUv2,TmpUv3;
		// 		if(ChengShu >0){
		// 		float Tmpfloat = Cuda_Compute_Cos(TheComputVU + cuurrentUv,ReflectPos,Origin_Id);
		// 		if(Tmpfloat < -50 && Tmpfloat<0 )
		// 			ChengShu = -1;
		// 		else
		// 			ChengShu = 1;
		// 		}

		//int2 TmpUv1,TmpUv2,TmpUv3; 

		TmpUv1 = cuurrentUv;
		TmpUv2 = cuurrentUv;
		TmpUv3 = cuurrentUv;

		//MoveVec = ChengShu * MoveVec;


		if(MoveVec.x > 0)
		{
			TmpUv1.x += 1.0;
			TmpUv3.x += 1.0;
		}
		else
		{
			TmpUv1.x -= 1.0;
			TmpUv3.x -= 1.0;
		}

		if(MoveVec.y >0)
		{
			TmpUv2.y += 1.0;
			TmpUv3.y += 1.0;

		}
		else
		{
			TmpUv2.y -= 1.0;
			TmpUv3.y -= 1.0;
		}
		float2 TmpUv4 = cuurrentUv;
		//TmpUv1.x = ProPosUv.x;
		//TmpUv1.y = ProPosUv.y;

		//printf("TmpUv1 :%f,%f\n",TmpUv1.x,TmpUv1.y);
		//printf("TmpUv2 :%f,%f\n",TmpUv2.x,TmpUv2.y);
		//printf("TmpUv3 :%f,%f\n",TmpUv3.x,TmpUv3.y);
		//printf("TmpUv4 :%f,%f\n",TmpUv4.x,TmpUv4.y);

		//取出每个采样点对应的世界坐标
		float4 TmpFloat4;
		float3 WorldPos1,WorldPos2,WorldPos3;
		float3 WorldNormal1,WorldNormal2,WorldNormal3;
		int Now_ModelId1,Now_ModelId2,Now_ModelId3;

		TmpFloat4 = tex2D(cuda_WorldPos_Tex,TmpUv1.x,TmpUv1.y);
		WorldPos1 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);

		TmpFloat4 = tex2D(cuda_WorldPos_Tex,TmpUv2.x,TmpUv2.y);
		WorldPos2 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);

		TmpFloat4 = tex2D(cuda_WorldPos_Tex,TmpUv3.x,TmpUv3.y);
		WorldPos3 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);

		TmpFloat4 = tex2D(cuda_WorldPos_Tex,TmpUv4.x,TmpUv4.y);
		float3 WorldPos4 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);


		TmpFloat4 = tex2D(cuda_WorlNormal_Tex,TmpUv1.x,TmpUv1.y);
		WorldNormal1 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);
		Now_ModelId1 = (int)TmpFloat4.w;

		TmpFloat4 = tex2D(cuda_WorlNormal_Tex,TmpUv3.x,TmpUv3.y);
		WorldNormal3 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);
		Now_ModelId3 = (int)TmpFloat4.w;

		TmpFloat4 = tex2D(cuda_WorlNormal_Tex,TmpUv2.x,TmpUv2.y);
		WorldNormal2 = make_float3(TmpFloat4.x,TmpFloat4.y,TmpFloat4.z);
		Now_ModelId2 = (int)TmpFloat4.w;



		//printf("WorldPos1 :%f,%f,%f\n",WorldPos1.x,WorldPos1.y,WorldPos1.z);
		//printf("WorldPos2 :%f,%f,%f\n",WorldPos2.x,WorldPos2.y,WorldPos2.z);
		//printf("WorldPos3 :%f,%f,%f\n",WorldPos3.x,WorldPos3.y,WorldPos3.z);
		//printf("WorldPos4 :%f,%f,%f\n",WorldPos4.x,WorldPos4.y,WorldPos4.z);



		//outPut[index] = make_float4(TmpUv2.x,TmpUv2.y,WorldPos1.z,1.0);
		//outPut[index] = make_float4((cuurrentUv.x - x)/1024.0,(cuurrentUv.y - y)/1024.0,Origin_Id,RefelctInCameraDepth);
		//return;


		//求出新的镜像点
		float3 ReMirrorPos1,ReMirrorPos2,ReMirrorPos3;

		ReMirrorPos1 =  abs(dot(WorldPos1 - ReflectPos,WorldNormal1)) * (-2) * WorldNormal1 + ReflectPos;

		ReMirrorPos2 =  abs(dot(WorldPos2 - ReflectPos,WorldNormal2)) * (-2) * WorldNormal2 + ReflectPos;

		ReMirrorPos3 =  abs(dot(WorldPos3 - ReflectPos,WorldNormal3)) * (-2) * WorldNormal3 + ReflectPos;

		//printf("ReMirrorPos1(%f,%f,%f)\n",ReMirrorPos1.x,ReMirrorPos1.y,ReMirrorPos1.z);
		//printf("ReMirrorPos2(%f,%f,%f)\n",ReMirrorPos2.x,ReMirrorPos2.y,ReMirrorPos2.z);
		//printf("ReMirrorPos3(%f,%f,%f)\n",ReMirrorPos3.x,ReMirrorPos3.y,ReMirrorPos3.z);

		//求相机与镜像点组成的平面的法线
		float3 Cam_Mirror_Normal1,Cam_Mirror_Normal2,Cam_Mirror_Normal3;
		Cam_Mirror_Normal1 = normalize(cross(d_cameraVec,normalize(d_refCameraPos - ReMirrorPos1)));
		Cam_Mirror_Normal2 = normalize(cross(d_cameraVec,normalize(d_refCameraPos - ReMirrorPos2)));
		Cam_Mirror_Normal3 = normalize(cross(d_cameraVec,normalize(d_refCameraPos - ReMirrorPos3)));

		//求相机轨迹所在法线
		float3 Camera_Vec_Normal1,Camera_Vec_Normal2,Camera_Vec_Normal3;
		Camera_Vec_Normal1 = normalize(cross(Cam_Mirror_Normal1,d_cameraVec));
		Camera_Vec_Normal2 = normalize(cross(Cam_Mirror_Normal2,d_cameraVec));
		Camera_Vec_Normal3 = normalize(cross(Cam_Mirror_Normal3,d_cameraVec));


		//printf("Camera_Vec_Normal1(%f,%f,%f)\n",Camera_Vec_Normal1.x,Camera_Vec_Normal1.y,Camera_Vec_Normal1.z);
		//printf("Camera_Vec_Normal2(%f,%f,%f)\n",Camera_Vec_Normal2.x,Camera_Vec_Normal2.y,Camera_Vec_Normal2.z);
		//printf("Camera_Vec_Normal3(%f,%f,%f)\n",Camera_Vec_Normal3.x,Camera_Vec_Normal3.y,Camera_Vec_Normal3.z);

		//分别求三个点与该平面的交点
		float3 InsertPoint1 = abs(dot(Camera_Vec_Normal1,d_refCameraPos - ReMirrorPos1)/dot(Camera_Vec_Normal1,normalize(WorldPos1-ReMirrorPos1))) * normalize(WorldPos1-ReMirrorPos1) + ReMirrorPos1;
		float3 InsertPoint2 = abs(dot(Camera_Vec_Normal2,d_refCameraPos - ReMirrorPos2)/dot(Camera_Vec_Normal2,normalize(WorldPos2-ReMirrorPos2))) * normalize(WorldPos2-ReMirrorPos2) + ReMirrorPos2;
		float3 InsertPoint3 = abs(dot(Camera_Vec_Normal3,d_refCameraPos - ReMirrorPos3)/dot(Camera_Vec_Normal3,normalize(WorldPos3-ReMirrorPos3))) * normalize(WorldPos3-ReMirrorPos3) + ReMirrorPos3;

		//printf("dis:(%f,%f,%f)\n",length(InsertPoint1-d_newCameraPos),length(InsertPoint2-d_newCameraPos),length(InsertPoint3-d_newCameraPos));
		//求点与直线的距离


		float3 toNewPlace1 = d_newCameraPos-InsertPoint1;
		//float	DisPoint2Line1 = dot(toNewPlace1,d_cameraVec); ;
		float DisPoint2Line1 = length(toNewPlace1);
		//DisPoint2Line1  =  dot(normalize(InsertPoint1 - d_refCameraPos),d_cameraVec);

		float3 toNewPlace2 = d_newCameraPos-InsertPoint2;
		// float	DisPoint2Line2 = dot(toNewPlace2,d_cameraVec); ;
		float DisPoint2Line2 = length(toNewPlace2);
		//DisPoint2Line2  =  dot(normalize(InsertPoint2 - d_refCameraPos),d_cameraVec);

		float3 toNewPlace3 = d_newCameraPos-InsertPoint3;
		//float	DisPoint2Line3 = dot(toNewPlace3,d_cameraVec); ;
		float DisPoint2Line3 = length(toNewPlace3);
		//DisPoint2Line3  =  dot(normalize(InsertPoint3 - d_refCameraPos),d_cameraVec);
		//printf("1:(%f,%f,%f) 2:(%f,%f,%f),3:(%f,%f,%f)\n",TmpUv1.x,TmpUv1.y,DisPoint2Line1,TmpUv2.x,TmpUv2.y,DisPoint2Line2,TmpUv3.x,TmpUv3.y,DisPoint2Line3);
		//
		if(formerDis<currentMaxDis)
		{
			//printf("$$$$$$$$$$$$$$$$$$$$$$search converge (%f,%f)\n",cuurrentUv.x,cuurrentUv.y);
			d_cudaPboBuffer[index] =  make_float4(cuurrentUv.x/(float)width,cuurrentUv.y/(float)height,1,-0.1);
	
			return;
		}
		float TmpDis;
		if(DisPoint2Line2<DisPoint2Line1)
		{
		TmpUv1 = TmpUv2;
		TmpUv2 = cuurrentUv;
		WorldPos = WorldPos1;
		WorldPos1 = WorldPos2;
		WorldPos2 = WorldPos;

		TmpDis = DisPoint2Line1;
		DisPoint2Line1 = DisPoint2Line2;
		DisPoint2Line2 = TmpDis;

		TmpDis = Now_ModelId1;
		Now_ModelId1 = Now_ModelId2;
		Now_ModelId2 = TmpDis;

		}

		if(DisPoint2Line3< DisPoint2Line1)
		{

		TmpUv1 = TmpUv3;
		TmpUv3 = cuurrentUv;
		WorldPos = WorldPos1;
		WorldPos1 = WorldPos3;
		WorldPos3 = WorldPos;

		TmpDis = DisPoint2Line1;
		DisPoint2Line1 = DisPoint2Line3;
		DisPoint2Line3 = TmpDis;

		TmpDis = Now_ModelId1;
		Now_ModelId1 = Now_ModelId3;
		Now_ModelId3 = TmpDis;
		}

		if(DisPoint2Line3< DisPoint2Line2)
		{

		TmpUv2 = TmpUv3;
		TmpUv3 = cuurrentUv;
		WorldPos = WorldPos2;
		WorldPos2 = WorldPos3;
		WorldPos3 = WorldPos;

		TmpDis = DisPoint2Line3;
		DisPoint2Line3 = DisPoint2Line2;
		DisPoint2Line2 = TmpDis;

		TmpDis = Now_ModelId3;
		Now_ModelId3 = Now_ModelId2;
		Now_ModelId2 = TmpDis;
		}


		//cuurrentUv = TmpUv1;
		if(Now_ModelId1 == Origin_Id)
		{
		currentMaxDis = DisPoint2Line1;
		cuurrentUv = TmpUv1;
		WorldPos = WorldPos1;
		}
		// 		else{
		// 		outPut[index] = make_float4(-10.0f,-10.0f,Origin_Id,RefelctInCameraDepth);
		// 			return;
		// 		}

		else if(Now_ModelId2 == Origin_Id)
		{

		//printf("equal id 2\n");

		currentMaxDis = DisPoint2Line2;
		cuurrentUv = TmpUv2;
		WorldPos = WorldPos2;
		}
		else if(Now_ModelId3 == Origin_Id)
		{

		//printf("equal id 3\n");
		currentMaxDis = DisPoint2Line3;
		cuurrentUv = TmpUv3;
		WorldPos = WorldPos3;
		}
		else
		{
		d_cudaPboBuffer[index] = make_float4(-10.0f,-10.0f,1,-0.1);
	
		return;
		}
		


		//printf("^^^^^^^^DisPoint2Line:%f,%f,%f,%f\n",DisPoint2Line1,DisPoint2Line2,DisPoint2Line3,formerDis);
		formerDis = currentMaxDis;
		//LastCoslength = Cuda_Compute_Cos(cuurrentUv,ReflectPos,Origin_Id);
		//LastCoslength = 0;
		//if(LastCoslength > length(Cuda_CameraPos1[1] - Cuda_CameraPos2[1])-0.5)
		//break;

		//printf("^^^^^^^^^^^^^^^^^^^^result(%f,%f)^^^^^\n",cuurrentUv.x,cuurrentUv.y);




		//得出该点法线
		float4 TmpFloat41= tex2D(cuda_WorlNormal_Tex,cuurrentUv.x,cuurrentUv.y);
		WorldNormal  = make_float3(TmpFloat41.x,TmpFloat41.y,TmpFloat41.z);

		float NowModelId = TmpFloat41.w;



		//outPut[index] = make_float4(WorldNormal.x,WorldNormal.y,WorldNormal.z,1.0);
		//return;
		//得出以该点为平面的反射物体的镜像点
		ReMirrorPos =  abs(dot(WorldPos - ReflectPos,WorldNormal)) * (-2) * WorldNormal + ReflectPos;



		//outPut[index] = make_float4(ReMirrorPos.x,ReMirrorPos.y,ReMirrorPos.z,1.0);
		//return;

		//求出该点与相机连线和平面的新交点
		DisEye2Plane = abs(dot(d_newCameraPos - WorldPos,WorldNormal));
		float CoseTheta2 = abs(dot(normalize(d_newCameraPos - ReMirrorPos),WorldNormal));
		VecEye2Ref = normalize(ReMirrorPos - d_newCameraPos);
		float3 FinalPos2;
		//	FinalPos2 = d_newCameraPos + VecEye2Ref * (DisEye2Plane/CoseTheta2 );
		FinalPos2 = dot(WorldPos -d_newCameraPos ,WorldNormal)/ dot(VecEye2Ref,WorldNormal) * VecEye2Ref + d_newCameraPos;

		//FinalPos2.y =  d_newCameraPos.y + VecEye2Ref.y *(DisEye2Plane/CoseTheta2);

		FinalPos_F[0] = FinalPos2.x;
		FinalPos_F[1] = FinalPos2.y;
		FinalPos_F[2] = FinalPos2.z;
		FinalPos_F[3] = 1.0;
		float4 ProPos2;
		ProPos2 = Cude_Mul_Mat4(Cuda_MvpMat,FinalPos_F);

		ProPosUv.x = (ProPos2.x / ProPos2.w * 0.5 + 0.5) * width;
		ProPosUv.y = (ProPos2.y / ProPos2.w * 0.5 + 0.5) * height;
		
		//MoveVec = ProPosUv - make_float2(cuurrentUv.x,cuurrentUv.y);
		MoveVec = ProPosUv - cuurrentUv;

		//printf("now proposUv:(%f,%f)\n",ProPosUv.x,ProPosUv.y);
	
		//outPut[index] = make_float4(ProPosUv.x,ProPosUv.y,ProPos2.z,ProPos2.w);

		//return;
		IterTime++;
		

	}

	//if(length(cuurrentUv - make_float2(x,y))> 30){
	//outPut[index] = make_float4(-10.0,-10.0,Origin_Id,RefelctInCameraDepth);
	//return;

	//}

	//if(dot(MoveVec,MoveVec) >=20){
	//outPut[index] = make_float4(-10.0f,-10.0f,Origin_Id,RefelctInCameraDepth);
	//return;

	//}


	//outPut[index] = ProPos;
	//if(cuurrentUv.x>=0 && cuurrentUv.x<width && cuurrentUv.y>=0 && cuurrentUv.y <= height)
	//	outPut[(int)(cuurrentUv.y * width + cuurrentUv.x)] = tex2D(cuda_Reflect_Tex,x,y);
	//WorlPosInCameraDepth = 10;
	//if(dot(MoveVec,MoveVec)>200)
	//WorlPosInCameraDepth = -10;
	//outPut[index] = make_float4((cuurrentUv.x - x)/1024.0,(cuurrentUv.y - y)/1024.0,Origin_Id,RefelctInCameraDepth);

	d_cudaPboBuffer[index] =  make_float4(cuurrentUv.x/(float)width,cuurrentUv.y/(float)height,1,-0.1);
	//printf("！x:%d y:%d: %f,%f,%f,%f\n",x,y,1024*d_cudaPboBuffer[index] .x,1024*d_cudaPboBuffer[index] .y,d_cudaPboBuffer[index] .z,d_cudaPboBuffer[index] .w);
	

	//float4 test = tex2D(cuda_Reflect_Tex,TmpUv.x,TmpUv.y);
	//d_cudaPboBuffer[index] =  test;

	//outPut[index] = make_float4((12)/256.0,(13)/256.0,0,1);
	//outPut[index] = make_float4(TmpUv.x,TmpUv.y,IterTime,1);


}
extern "C" void MappingContantPeremeter(int* w,int* h)
{


	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_width),w,sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_height),h,sizeof(int)));


}
extern "C" void TransConstData(const float *MvpMat,const float *ViewMat,float3 *pRefCamera,float3 *pNewCameraPos)
{

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_MvpMat),MvpMat,sizeof(float)*16,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Cuda_CameraViewMat),ViewMat,sizeof(float)*16,0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_refCameraPos),pRefCamera,sizeof(float3),0,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_newCameraPos),pNewCameraPos,sizeof(float3),0,hipMemcpyHostToDevice));

	float3 CameraVec = normalize(*pNewCameraPos - *pRefCamera);
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cameraVec),&CameraVec,sizeof(float3),0,hipMemcpyHostToDevice));

}


__device__ bool colorEqual(float3 soourceColor, float3 targetColor)
{
	return length(soourceColor-targetColor)<0.1;
}
__global__ void MyFirstPassKernel(int width,int height,uint *PixelState,float4 * cuda_PBO_Buffer)
{

	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;
	if(x > width || y > height)
		//if(x!=224||y!=267)
			return;

	//if(x !=224 || y !=267)
	//	if(x !=117 || y !=712)
	//	return;

	int index = y * width + x;

	//printf("$%d,%d\n",x,y);

	float4 StateValue =tex2D(cuda_RePro_Pixel,x+0.5,y+0.5);
	float3 reflectColor = make_float3(StateValue.x,StateValue.y,StateValue.z);

	if(colorEqual(reflectColor,make_float3(1,0,0))||colorEqual(reflectColor,make_float3(0,1,0))||reflectColor.x>1.01||reflectColor.y>1.01)  // fetch the color in additional tex
	{
		d_state[index] = 1;
	}
	else
	{
		//printf("ddddddd\n");
		d_state[index] = 0;
	}

}
extern "C" void cudaPredict(int width,int height)
{
	dim3 blockSize(16,16,1);
	dim3 gridSize(width/blockSize.x,height/blockSize.y,1);
	MyNewKernel<<<gridSize,blockSize>>>(width,height);
}

void mapThustResourse()
{
	thrust::device_ptr<uint> dp = &thrustStateVec[0];
	uint * raw_ptr = thrust :: raw_pointer_cast ( dp );  
}
__global__ void MyThridPass(uint *PixePos,uint *PixelState,uint *PixelSums);

extern "C" void RunFirstPass(dim3 BlockSize,dim3 GridSize,int width,int height)
{
	thrust::device_ptr<uint> dp = &thrustStateVec[0];
	uint * raw_ptr = thrust :: raw_pointer_cast ( dp );  
	MyFirstPassKernel<<<GridSize,BlockSize>>>(width,height,raw_ptr,cuda_PBO_Buffer);
	


}


__global__ void MyThridPass(uint *PixePos,uint *PixelState,uint *PixelSums)
{

	int x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;

	if(x > rastWidth || y > rastHeight)
		return;
	uint index = y * rasterWidth + x;
	if(PixelState[index] == 1)
	{
		uint Pos1 = PixelSums[index];
		PixePos[Pos1] = index;

	}
}
void InitThrust()
{	
	thrustSumVec = thrust::device_vector<uint>(rastWidth*rastHeight); 
	thrustStateVec = thrust::device_vector<uint>(rastWidth*rastHeight);
	thrustPosVec = thrust::device_vector<uint>(rastWidth*rastHeight);

	thrust::device_ptr<uint> d_pstate = &thrustStateVec[0];
	g_PixelState = thrust :: raw_pointer_cast ( d_pstate );  
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_state), &g_PixelState,sizeof(int*)));

	thrust::device_ptr<uint> d_psum = &thrustSumVec[0];
	g_PixelSums = thrust :: raw_pointer_cast ( d_psum );
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_sum), &g_PixelSums,sizeof(int*)));

	thrust::device_ptr<uint> dp = &thrustPosVec[0];
	g_PixelPos= thrust :: raw_pointer_cast ( dp );
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_pos), &g_PixelPos,sizeof(int*)));

}



extern "C" void RunThridPass(dim3 BlockSize ,dim3 GridSize,int totalNum)
{	  	
	thrust::device_ptr<uint> dp = &thrustPosVec[0];
	uint * raw_ptr_pos = thrust :: raw_pointer_cast ( dp );  

	thrust::device_ptr<uint> d_pstate = &thrustStateVec[0];
	uint * raw_ptr_state = thrust :: raw_pointer_cast ( d_pstate );  

	thrust::device_ptr<uint> d_psum = &thrustSumVec[0];
	uint * raw_ptr_sum = thrust :: raw_pointer_cast ( d_psum );
	MyThridPass<<<GridSize,BlockSize>>>(raw_ptr_pos,raw_ptr_state,raw_ptr_sum);

	uint* h_state = thrust::raw_pointer_cast(d_pstate);

	//thrust::host_vector <uint> h_pos =thrustSumVec;
	thrust::host_vector <uint> h_pos =thrustPosVec;
	/* 
	for(int y = 0;y<1024;y++)
	{
		for(int x = 0;x<1024;x++)
		{
			int index = 1024*y+x;
			if(h_state[index] == 1)
			{

			int stateValue = h_state[index];
			int sumValue = h_sum[index];
			printf("index(%d,%d,%d) v:(%d,%d)\n",x,y,index,stateValue,sumValue);
			}
		}
	}
	*/
	/*for(int i =totalNum*10/11.0;i<totalNum;i++)
	{

		int value = h_pos[i];
		int	y	 =  value/1024;
 		int x  = value%1024;
		printf("(%d,%d,pos:%d，index:%d)\n",x,y,h_pos[i],i);
	}*/

	
}
int thrustReduction(int width,int height)
{

	dim3 blockSize(16,16,1);
	dim3 GridSize(width/blockSize.x,height/blockSize.y,1);
	RunFirstPass(blockSize,GridSize,width,height);
	thrust::inclusive_scan(thrustStateVec.begin(), thrustStateVec.end() , thrustSumVec.begin());
	thrust::host_vector<uint> hSum = thrustSumVec;

	int totalNum = thrustSumVec[width*height-1]+thrustStateVec[width*height-1];
	RunThridPass(blockSize,GridSize,totalNum);
	//return totalNum;

	return totalNum;
}
extern "C"  void cudaRelateTex(CudaTexResourse * pResouce)
{

	hipArray *tmpcudaArray;
	hipGraphicsResource ** pCudaTex = pResouce->getResPoint(); 
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&tmpcudaArray,*pCudaTex,0,0));
	if(worldPosRef_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_WorldPos_Tex,tmpcudaArray,channelDesc));
		cuda_WorldPos_Tex.filterMode = hipFilterModeLinear;
	}
	else if(worldNormalRef_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_WorlNormal_Tex,tmpcudaArray,channelDesc));
		cuda_WorlNormal_Tex.filterMode = hipFilterModePoint;
	}
	else if(reflecionRef_t ==  pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_Reflect_Tex,tmpcudaArray,channelDesc));
		cuda_Reflect_Tex.filterMode = hipFilterModePoint;
	}
	else if(finalEffect_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cuda_RePro_Pixel,tmpcudaArray,channelDesc));
	}

}
extern "C" void cudaRelateArray(CudaPboResource * pResource)
{
	size_t numBytes;
	hipGraphicsResource ** pCudaTex = pResource->getResPoint(); 
	/*
	moveVector_t,
	lastVector_t,
	*/
	if(float4_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_PBO_Buffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaPboBuffer), &cuda_PBO_Buffer,sizeof(float4*)));
	}
	else if(float2_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_LastVec_Buffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaLastBuffer), &cuda_LastVec_Buffer,sizeof(float2*)));

	}
	else if(test_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_test_Buffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaTestBuffer), &cuda_test_Buffer,sizeof(float4*)));

	}
	else if(diff_normal_t == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_diffNormBuffer,&numBytes,*pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaTestBuffer), &cuda_diffNormBuffer,sizeof(float4*)));

	}
}